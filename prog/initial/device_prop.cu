
#include <hip/hip_runtime.h>
#include <stdio.h>


//get device count and device properties
int get_device_prop(int &deviceCount, hipDeviceProp_t* &deviceProp){
    //varaibles
    hipError_t currentError;	//current error
    hipDeviceProp_t temppd;	//temp varaible
    
    currentError=hipGetDeviceCount(&deviceCount);
    printf("dev count in function %d \n", deviceCount);
    if (currentError!=hipSuccess){
        fprintf(stderr,"Cannot get CUDA device count: %s\n", hipGetErrorString(currentError));
        return 1;
    }
    if (!deviceCount){
        fprintf(stderr, "No CUDA devices found\n");
        return 1;
    }
    deviceProp=(hipDeviceProp_t*) malloc(deviceCount*sizeof(hipDeviceProp_t));
    for (int i=0;i<deviceCount;i++){
        hipGetDeviceProperties(&deviceProp[i],i);
        hipGetDeviceProperties(&temppd,i);
        deviceProp[i]=temppd;
        printf("Device name %s \n", deviceProp[i].name);
        printf("Max Threads Dim: %d %d %d \n", deviceProp[i].maxThreadsDim[0],deviceProp[i].maxThreadsDim[1],deviceProp[i].maxThreadsDim[2]);
        printf("Max Grid Size: %d %d %d \n", deviceProp[i].maxGridSize[0], deviceProp[i].maxGridSize[1], deviceProp[i].maxGridSize[2]);
    }
    return 0;
}






