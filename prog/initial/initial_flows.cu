#include "hip/hip_runtime.h"
//
#include <stdio.h>
#include <math.h>
#include "../mcrec.h"
#include <time.h>

int initial_flows(options config, flows* &initFlows){
    float* l_x; //latice coords
    float* l_y;
    float* l_z;
    int latice;
    float laticeDelta;
    int id;
    int* check;
    int* moleculesNumber;
    int sum;    //temporally sum
    int set;    //seted current
    int curRand;
    int latice3;
    
    srand(time(0));
    //create arrays
    initFlows=(flows*) malloc(config.flowNum * sizeof(flows));  
    moleculesNumber=(int*) malloc(config.subNum * sizeof(int));
    for(int i=0; i<config.flowNum;i++){
        initFlows[i].molNum=2000;
        //get latice size 
        latice=ceil(pow(initFlows[i].molNum,1.0/3.0))+1;
        latice3=latice*latice*latice;
        l_x = (float*) malloc(latice3*sizeof(float));
        l_y = (float*) malloc(latice3*sizeof(float));
        l_z = (float*) malloc(latice3*sizeof(float));
        //define system size
        if(config.flowEns[i]==NVT){
            laticeDelta=pow(initFlows[i].molNum/(config.flowN[i]*NA/1.0e24),1.0/3.0)/latice;
            printf("latice delta %f\n", laticeDelta);
        }
        //get coordinats
        check=(int*) malloc(latice3*sizeof(int));  //check place busy
        id=0;
        for(int l_i=0; l_i<latice; l_i++){
            for(int l_j=0; l_j<latice; l_j++){
                for(int l_k=0; l_k<latice; l_k++){
                    l_x[id]=l_i*laticeDelta;
                    l_y[id]=l_j*laticeDelta;
                    l_z[id]=l_k*laticeDelta;
                    check[id]=-1;    //set place not busy
                    id++;
                }
            }
        }
        initFlows[i].xm=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].ym=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].zm=(float*) malloc(initFlows[i].molNum*sizeof(float));
        //get numbers of molecules of each substance
        sum=0;
        for(int s_i=0; s_i<config.subNum-1;s_i++){
            moleculesNumber[s_i]=ceil(config.flowX[i][s_i]*initFlows[i].molNum);
            sum+=moleculesNumber[s_i];
        }
        moleculesNumber[config.subNum-1]=initFlows[i].molNum - sum;
        //set molecules to placees
        id=0;
        for(int cur_sub=0;cur_sub<config.subNum;cur_sub++){
            set=0;
            while(set<moleculesNumber[cur_sub]){
                //get random position
                curRand = rand() % latice3;
                if(check[curRand]/=-1){
                    check[curRand]=cur_sub;
                    initFlows[i].xm[id]=l_x[curRand];
                    initFlows[i].ym[id]=l_y[curRand];
                    initFlows[i].zm[id]=l_z[curRand];
                    id++;
                    set++;
//                    printf("set %d %d\n",set,curRand);
                }
            }
        }
    }
    return 1;
}
