#include "hip/hip_runtime.h"
//
#include <stdio.h>
#include <math.h>
#include "../mcrec.h"

int initial_flows(options config, flows* &initFlows){
    float* l_x; //latice coords
    float* l_y;
    float* l_z;
    int latice;
    float laticeDelta;
    int id;
    //create arrays
    initFlows=(flows*) malloc(config.flowNum * sizeof(flows));
    for(int i=0; i<config.flowNum;i++){
        initFlows[i].molNum=2000;
        //get latice size 
        latice=ceil(pow(initFlows[i].molNum,1.0/3.0));
        l_x = (float*) malloc(latice*latice*latice*sizeof(float));
        l_y = (float*) malloc(latice*latice*latice*sizeof(float));
        l_z = (float*) malloc(latice*latice*latice*sizeof(float));
        //define system size
        if(config.flowEns[i]==NVT){
            laticeDelta=pow(initFlows[i].molNum/(config.flowN[i]*NA/1.0e24),1.0/3.0)/latice;
            printf("latice delta %f\n", laticeDelta);
        }
        //get coordinats
        id=0;
        for(int l_i=0; l_i<latice; l_i++){
            for(int l_j=0; l_j<latice; l_j++){
                for(int l_k=0; l_k<latice; l_k++){
                    l_x[id]=l_i*laticeDelta;
                    l_y[id]=l_j*laticeDelta;
                    l_z[id]=l_k*laticeDelta;
                    id++;
                }
            }
        }
        initFlows[i].xm=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].ym=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].zm=(float*) malloc(initFlows[i].molNum*sizeof(float));
    }
    return 1;
}
