#include "hip/hip_runtime.h"
//
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../mcrec.h"
#include <time.h>
//#include "../initial.h"

int initial_flows(options &config, singleBox* &initFlows,molecules* initMol, singleBox* &gpuSingleBox, int lines, potentialParam* allParams, potentialParam* gpuParams, potentialParam* hostParams, mixParam** gpuMixParams, mixParam** hostMixParams,hipDeviceProp_t* deviceProp){
    float* l_x; //latice coords
    float* l_y;
    float* l_z;
    int* l_t; //0 --- empty
    int latice;
    int latice3;
    float laticeDelta;
    int id;
    int sum;
    int moleculePerBox;
    int randMol;
    int l_type;
    int* linesList;
    int linesEnd;
    int checked;
    //int sizey;
    
    config.singleXDim=ceil(deviceProp[0].maxThreadsPerBlock);
    config.singleYDim=ceil(2000/config.singleXDim)+1;
    moleculePerBox=config.singleYDim*config.singleYDim;
    srand(time(0));
//    printf(" ydim  %d \n", config.singleYDim);
//    getchar();
    //create arrays
    initFlows=(singleBox*) malloc(config.flowNum * sizeof(singleBox));
    srand(time(0));
    //create arrays
//    moleculesNumber=(int*) malloc(config.subNum * sizeof(int));
    for(int i=0; i<config.flowNum;i++){
        initFlows[i].molNum=moleculePerBox;
        //get latice size 
        latice=ceil(pow(initFlows[i].molNum,1.0/3.0));
//        latice=ceil(pow(initFlows[i].molNum,1.0/3.0))+1;
        latice3=latice*latice*latice;
        l_x = (float*) malloc(latice3*sizeof(float));
        l_y = (float*) malloc(latice3*sizeof(float));
        l_z = (float*) malloc(latice3*sizeof(float));
        l_t = (int*) malloc(latice3*sizeof(int));  //check place busy
        //define system size
        if(config.flowEns[i]==NVT){
            laticeDelta=pow(initFlows[i].molNum/(config.flowN[i]*NA/1.0e24),1.0/3.0)/latice;
            printf("latice delta %f\n", laticeDelta);
        }
        //get coordinats
        id=0;
        for(int l_i=0; l_i<latice; l_i++){
            for(int l_j=0; l_j<latice; l_j++){
                for(int l_k=0; l_k<latice; l_k++){
                    l_x[id]=l_i*laticeDelta;
                    l_y[id]=l_j*laticeDelta;
                    l_z[id]=l_k*laticeDelta;
                    l_t[id]=0;    //set place not busy
                    id++;
                }
            }
        }
        initFlows[i].xm=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].ym=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].zm=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].typeMolNum=(int*) malloc(config.subNum*sizeof(int));
        initFlows[i].type=(int*) malloc(initFlows[i].molNum*sizeof(int));
        //get numbers of molecules of each types
        sum=0;
        for(int j=0;j<config.subNum-1;j++){
            initFlows[i].typeMolNum[j]=initFlows[i].molNum*config.flowX[i][j];
            sum+=initFlows[i].typeMolNum[j];
        }
        initFlows[i].typeMolNum[config.subNum-1]=initFlows[i].molNum-sum;
        //set molecules to places
        id=0;
        for(int j=0;j<config.subNum;j++){
            sum=0;
            while(sum<initFlows[i].typeMolNum[j]){
                randMol=rand() % latice3;
                
                if(l_t[randMol]==0){
//                    printf("random n %d \n", randMol);
                    initFlows[i].xm[id]=l_x[randMol];   //set ccordinates
                    initFlows[i].ym[id]=l_y[randMol];
                    initFlows[i].zm[id]=l_z[randMol];
                    l_t[randMol]=1; //now plase not empty
                    initFlows[i].type[id]=i;
                    id++;
                    sum++;
                }
            }
        }
        initFlows[i].xa=(float**) malloc(initFlows[i].molNum*sizeof(float*));
        initFlows[i].ya=(float**) malloc(initFlows[i].molNum*sizeof(float*));
        initFlows[i].za=(float**) malloc(initFlows[i].molNum*sizeof(float*));
        initFlows[i].aType=(int**) malloc(initFlows[i].molNum*sizeof(int*));
        initFlows[i].aNum=(int*)malloc(initFlows[i].molNum*sizeof(int));
        printf("num of mol %d\n", initFlows[i].molNum);
        //set atoms to places
        for(int j=0; j<initFlows[i].molNum;j++){    //check all molecules
            //
            //
            
            l_type=initFlows[i].type[j];  //get molecules type
            initFlows[i].aNum[j]=initMol[l_type].atomNum;
            initFlows[i].xa[j]=(float*) malloc(initMol[l_type].atomNum*sizeof(float));
            initFlows[i].ya[j]=(float*) malloc(initMol[l_type].atomNum*sizeof(float));
            initFlows[i].za[j]=(float*) malloc(initMol[l_type].atomNum*sizeof(float));
            initFlows[i].aType[j]=(int*) malloc(initMol[l_type].atomNum*sizeof(int));
            //printf("type %d numbers of atom in molecule %d\n",l_type,initMol[l_type].atomNum);
            for(int k=0;k<initMol[l_type].atomNum;k++){
                initFlows[i].xa[j][k]=initMol[l_type].x[k];
                initFlows[i].ya[j][k]=initMol[l_type].y[k];
                initFlows[i].za[j][k]=initMol[l_type].z[k];
            }
        }
    }
    //getchar();
    //data to GPU
    hipMalloc(&gpuSingleBox, config.flowNum*sizeof(singleBox));
    hipMemcpy(gpuSingleBox,initFlows,config.flowNum*sizeof(singleBox),hipMemcpyHostToDevice);
    
//    //check used atoms
//    for(int i=0;i<config.subNum;i++){
//        for(int j=0;j<initMol[i].atomNum;j++){
//            
//        }
//    }
    linesList=(int*) malloc (lines*sizeof(int));
    //check used atoms
    id=0;
    for(int i=0;i<lines;i++){
        checked=0;
        for(int sub=0;sub<config.subNum;sub++){
            for(int at=0;at<initMol[sub].atomNum;at++){
//                if(allParams[i].aName==initMol[sub].atomNum)
//                printf("|%s|%s|%d|%d\n",allParams[i].aName,initMol[sub].aName[at], strcmp(initMol[sub].aName[at],allParams[i].aName),strcmp(allParams[i].aName,initMol[sub].aName[at]));
                if(strcmp(initMol[sub].aName[at],allParams[i].aName)==0 || abs(strcmp(initMol[sub].aName[at],allParams[i].aName))==127){
                    printf("i %d sub %d a %d %s  %s \n",i,sub, at, allParams[i].aName,initMol[sub].aName[at]);
                    //add line from top file to gpu
                    checked=1;
                }
            }
        }
        if(checked==1){
            linesList[id]=i;
            id++;
        }
    }
    linesEnd=id;
    //host parameters
    hostParams=(potentialParam*)malloc(linesEnd*sizeof(potentialParam));
    for(int i=0;i<linesEnd;i++){
        hostParams[i]=allParams[linesList[i]];
        printf("host aname %d %s\n",i, hostParams[i].aName);
    }
    //add mixrule
    //mixParam hostMixParams[linesEnd][linesEnd];
    hostMixParams = (mixParam**) malloc(linesEnd*sizeof(mixParam*));
    for(int i=0;i<linesEnd;i++){
        hostMixParams[i]=(mixParam*) malloc(linesEnd*sizeof(mixParam));
        for(int j=0;j<linesEnd;j++){
            if(config.mixRule==LB){
                hostMixParams[i][j].sigma=0.5*(hostParams[i].sigma+hostParams[j].sigma);
                hostMixParams[i][j].epsilon=sqrt(hostParams[i].epsilon*hostParams[j].epsilon);
                hostMixParams[i][j].alpha=0.0;
                hostMixParams[i][j].charge=hostParams[i].charge*hostParams[j].charge;
            }
        }
    }
    //add atom types
    for(int i=0;i<config.flowNum;i++){  //flow
        for(int j=0;j<initFlows[i].molNum;j++){ //molecule
            
            for(int k=0;k<initFlows[i].aNum[j];k++){    //atom
                //get curent atom name
                //initMol[initFlows[i].type[j]].aName[k]
                for(int chk=0;chk<linesEnd;chk++){  //check from list
                    if(strcmp(initMol[initFlows[i].type[j]].aName[k],hostParams[chk].aName)==0 || abs(strcmp(initMol[initFlows[i].type[j]].aName[k],hostParams[chk].aName))==127 ){
                        
                    }
                }
                
            }
            
        }
        
    }
    
    //malloc gpu mix params
    hipMalloc(&gpuMixParams,linesEnd*linesEnd*sizeof(mixParam));
    hipMemcpy(gpuMixParams,hostMixParams,linesEnd*linesEnd*sizeof(mixParam),hipMemcpyHostToDevice);
//    for(int i=0;i<linesEnd;i++){
//        hipMalloc(&gpuMixParams[i],linesEnd*sizeof(mixParam));
//    }
    
    //malloc parameters
//    hipMalloc(&gpuParams,linesEnd*sizeof(potentialParam));
//    hipMemcpy(gpuParams,hostParams,linesEnd*sizeof(potentialParam),hipMemcpyHostToDevice);
    
//    hipMemcpy(gpuParams,)
    
//        //get numbers of molecules of each substance
//        sum=0;
//        for(int s_i=0; s_i<config.subNum-1;s_i++){
//            moleculesNumber[s_i]=ceil(config.flowX[i][s_i]*initFlows[i].molNum);
//            sum+=moleculesNumber[s_i];
//        }
//        moleculesNumber[config.subNum-1]=initFlows[i].molNum - sum;
//        //set molecules to placees
//        id=0;
//        for(int cur_sub=0;cur_sub<config.subNum;cur_sub++){
//            set=0;
//            while(set<moleculesNumber[cur_sub]){
//                //get random position
//                curRand = rand() % latice3;
//                if(check[curRand]/=-1){
//                    check[curRand]=cur_sub;
//                    initFlows[i].xm[id]=l_x[curRand];
//                    initFlows[i].ym[id]=l_y[curRand];
//                    initFlows[i].zm[id]=l_z[curRand];
//                    id++;
//                    set++;
////                    printf("set %d %d\n",set,curRand);
//                }
//            }
//        }
//    }
        
        
    return 1;
}


