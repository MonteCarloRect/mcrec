#include "hip/hip_runtime.h"
//
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../mcrec.h"

int initial_flows(options config, singleBox* &initFlows,molecules* initMol, singleBox* &gpuSingleBox){
    float* l_x; //latice coords
    float* l_y;
    float* l_z;
    int* l_t; //0 --- empty
    int latice;
    int latice3;
    float laticeDelta;
    int id;
    int moleculePerBox;
    int sum;
    int randMol;
    int l_type;
    
    moleculePerBox=2000;
    srand(time(0));
    
    //create arrays
    initFlows=(singleBox*) malloc(config.flowNum * sizeof(singleBox));
    for(int i=0; i<config.flowNum;i++){
        initFlows[i].molNum=moleculePerBox;
        //get latice size 
        latice=ceil(pow(initFlows[i].molNum,1.0/3.0));
        latice3=latice*latice*latice;
        l_x = (float*) malloc(latice3*sizeof(float));
        l_y = (float*) malloc(latice3*sizeof(float));
        l_z = (float*) malloc(latice3*sizeof(float));
        l_t = (int*) malloc(latice3*sizeof(int));
        //define system size
        if(config.flowEns[i]==NVT){
            laticeDelta=pow(initFlows[i].molNum/(config.flowN[i]*NA/1.0e24),1.0/3.0)/latice;
            printf("latice delta %f\n", laticeDelta);
        }
        //get coordinats
        id=0;
        for(int l_i=0; l_i<latice; l_i++){
            for(int l_j=0; l_j<latice; l_j++){
                for(int l_k=0; l_k<latice; l_k++){
                    l_x[id]=l_i*laticeDelta;
                    l_y[id]=l_j*laticeDelta;
                    l_z[id]=l_k*laticeDelta;
                    l_t[id]=0;
                    id++;
                }
            }
        }
        initFlows[i].xm=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].ym=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].zm=(float*) malloc(initFlows[i].molNum*sizeof(float));
        initFlows[i].typeMolNum=(int*) malloc(config.subNum*sizeof(int));
        initFlows[i].type=(int*) malloc(initFlows[i].molNum*sizeof(int));
        //get numbers of molecules of each types
        sum=0;
        for(int j=0;j<config.subNum-1;j++){
            initFlows[i].typeMolNum[j]=initFlows[i].molNum*config.flowX[i][j];
            sum+=initFlows[i].typeMolNum[j];
        }
        initFlows[i].typeMolNum[config.subNum-1]=initFlows[i].molNum-sum;
        //set molecules to places
        id=0;
        for(int j=0;j<config.subNum;j++){
            sum=0;
            while(sum<initFlows[i].typeMolNum[j]){
                randMol=rand() % latice3;
                
                if(l_t[randMol]==0){
//                    printf("random n %d \n", randMol);
                    initFlows[i].xm[id]=l_x[randMol];   //set ccordinates
                    initFlows[i].ym[id]=l_y[randMol];
                    initFlows[i].zm[id]=l_z[randMol];
                    l_t[randMol]=1; //now plase not empty
                    initFlows[i].type[id]=i;
                    id++;
                    sum++;
                }
            }
        }
        initFlows[i].xa=(float**) malloc(initFlows[i].molNum*sizeof(float*));
        initFlows[i].ya=(float**) malloc(initFlows[i].molNum*sizeof(float*));
        initFlows[i].za=(float**) malloc(initFlows[i].molNum*sizeof(float*));
        //set atoms to places
        for(int j=0; j<initFlows[i].molNum;j++){    //check all molecules
            l_type=initFlows[i].type[j];  //get molecules type
            initFlows[i].xa[j]=(float*) malloc(initMol[l_type].atomNum*sizeof(float));
            initFlows[i].ya[j]=(float*) malloc(initMol[l_type].atomNum*sizeof(float));
            initFlows[i].za[j]=(float*) malloc(initMol[l_type].atomNum*sizeof(float));
            for(int k=0;k<initMol[l_type].atomNum;k++){
                initFlows[i].xa[j][k]=initMol[l_type].x[k];
                initFlows[i].ya[j][k]=initMol[l_type].y[k];
                initFlows[i].za[j][k]=initMol[l_type].z[k];
            }
        }
    }
    //data to GPU
    hipMalloc(&gpuSingleBox, config.flowNum*sizeof(singleBox));
    hipMemcpy(gpuSingleBox,initFlows,config.flowNum*sizeof(singleBox),hipMemcpyHostToDevice);
    
    //check used atoms
    for(int i=0;i<config.subNum;i++){
        for(int j=0;j<initMol[i].atomNum;j++){
            
        }
    
    }
        
        
    return 1;
}
