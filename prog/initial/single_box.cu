#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../mcrec.h"
#include <time.h>

#include "cuPrintf.cuh"
#include "cuPrintf.cu"

__global__ void single_calc(singleBox* gpuSingleBox, potentialParam* gpuParams,int yDim){
    //calculate initial enegry, pressure and other
    //printf("ololo id %d blockIdx.x  %d blockIdx.y %d blockDim.x \n", threadIdx.x, threadIdx.y, blockDim.x);
    single_calc_totenergy(yDim,gpuParams);
    //main cycle
    
}

__device__ void single_calc_totenergy(int yDim, potentialParam* gpuParams){
    //printf("ololo 4");
    float* en;
    en=(float*)malloc(yDim*blockDim.x*sizeof(float));
    //hipMalloc(&en, yDim*blockDim.x*sizeof(float));
    
    for(int i=0;i<yDim;i++){
        int curMol=threadIdx.x+i*blockDim.x;
        //printf("id %d\n", curMol);
        for(int j=curMol++;j<yDim*blockDim.x;j++){
            //calculate energy  curMol and j molecules
            
        }
    }
}

//calculate potential
__device__ void single_calc_potential(int a, int b, singleBox* gpuSingleBox, potentialParam* gpuParams){
    
    
}




