#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"

__device__ int single_conf_change(int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox, hiprandState &devStates);
__device__ int single_one_energy(int yDim, int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox);

//
//#include <hiprand/hiprand_kernel.h>
//#include <hip/hip_runtime.h>

//

//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"

__global__ void single_calc(int yDim, gOptions gConf, gMolecula gTop, gSingleBox gBox){
    int test;
    __shared__ int randMol;
    hiprandState devStates;
    //calculate initial enegry, pressure and other
    //printf("ololo id %d blockIdx.x  %d blockIdx.y %d blockDim.x \n", threadIdx.x, threadIdx.y, blockDim.x);
    //test=single_calc_totenergy(yDim, gpuSingleBox, gpuParams, gpuMixParams);
    //main cycle
    //test=gpuSingleBox[0].molNum;
    test=1;
    //printf("deb 81 %d \n", gTop.aNum[0]);
    single_calc_totenergy(yDim, gConf, gTop, gBox);
    __syncthreads();
    hiprand_init(1234, threadIdx.x, 0, &devStates);
    //loop for change molecules position
    for(int i=0; i< 10; i++){
        //get random numbers
        if(threadIdx.x==0){
            //gBox.curMol[blockIdx.x] = (hiprand_uniform(&devStates)*gBox.molNum[blockIdx.x]);
            randMol = hiprand_uniform(&devStates)*gBox.molNum[blockIdx.x];
            printf("random %d %d  \n", i, randMol);
            //change molecul
        }
        __syncthreads();
        //------calculate old energy
        
        if(threadIdx.x==0){
            single_conf_change(randMol, gConf, gTop, gBox, devStates);
        }
        __syncthreads();
        
        //-------calculate new energy
        //calculate properties
    }
        


    //printf("test test %f  %d\n", gBox.xm[threadIdx.x], threadIdx.x);
}

__device__ int single_calc_totenergy(int yDim, gOptions gConf, gMolecula gTop, gSingleBox &gBox){
    float* en;
    int curMol;
    int curMol2;
    __shared__ int maxMol;
    float sumE;
    float sumV;
    __shared__ int reduce;
    //printf("tot energy %d\n",blockIdx.x);
    //printf("my_test id %d blockIdx.x  %d blockIdx.y %d blockDim.x \n", threadIdx.x, threadIdx.y, blockDim.x);
//    en=(float*)malloc(yDim*blockDim.x*sizeof(float));
//    //hipMalloc(&en, yDim*blockDim.x*sizeof(float));
//    
    maxMol = gBox.fMol[blockIdx.x]+blockDim.x*yDim;
    for(int i=0; i<yDim; i++){    //for several molecules per thread
        curMol=gBox.fMol[blockIdx.x]+threadIdx.x+i*blockDim.x;    //current number of molecule
        //printf("yDim %d i %d blockDim.x %d curmol %d maxmol %d fmol %d \n", yDim, i, blockDim.x, curMol, maxMol, gBox.fMol[blockIdx.x]);
        gBox.mEnergy[curMol]=0.0;
        gBox.mVirial[curMol]=0.0;
        for(int j= curMol++; j < maxMol; j++){    //curMol+1 //gBox.fMol[blockIdx.x]+blockDim.x*yDim
            //calculate energy curMol and j molecules
            //printf("test a %d b %d\n", curMol, j);
            single_calc_one_potential(curMol, j, gConf, gTop, gBox, sumE, sumV);
            //sum+=1;
        }
        //printf(" thread %d cur mol %d sum %f\n", threadIdx.x, curMol, sum);
        //gBox.mEnergy[curMol]=1.0;
        //printf("curmol %d energy %f virial %f \n", curMol, gBox.mEnergy[curMol], gBox.mVirial[curMol]);
        //printf("curmol %d energy %f virial %f \n", curMol, sumE, sumV);
    }
    __syncthreads();
//  //summm potential   //change reduse
    if(threadIdx.x == 0){
        gBox.energy[blockIdx.x]=0.0;
        gBox.virial[blockIdx.x]=0.0;
        for(int i=gBox.fMol[blockIdx.x]; i<maxMol; i++){
            gBox.energy[blockIdx.x]+=gBox.mEnergy[i];
            gBox.virial[blockIdx.x]+=gBox.mVirial[i];
        }
        printf("variant 1 total enegry %f virial %f \n", gBox.energy[blockIdx.x], gBox.virial[blockIdx.x]);
    }
    __syncthreads();
// variant 2
    reduce = blockDim.x / 2;
    gBox.energy[blockIdx.x]=0.0;
    gBox.virial[blockIdx.x]=0.0;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            curMol2=gBox.fMol[blockIdx.x]+threadIdx.x;
            for(int i=0; i< yDim; i++){
                curMol=gBox.fMol[blockIdx.x]+threadIdx.x+i*blockDim.x + reduce;
                gBox.mEnergy[curMol2]+=gBox.mEnergy[curMol];
                gBox.mVirial[curMol2]+=gBox.mVirial[curMol];
            }
        //printf(" reduce %d \n", reduce);
        }
        __syncthreads();
        reduce = reduce / 2;
        //
    }
    if(threadIdx.x == 0){
        gBox.energy[blockIdx.x] = gBox.mEnergy[0];
        gBox.virial[blockIdx.x] = gBox.mVirial[0];
        printf("variant 2 total enegry %f virial %f \n", gBox.energy[blockIdx.x], gBox.virial[blockIdx.x]);
    }

    return 0;
}

//calculate potential
__device__ int single_calc_one_potential(int a, int b, gOptions gConf, gMolecula gTop, gSingleBox &gBox, float &En, float &Vir){
    //printf("one potential a %d b %d \n", a, b);
    intra_potential(a, gConf, gTop, gBox);
    inter_potential(a, b, gConf, gTop, gBox, En, Vir);
    
    return 0;
}

__device__ int inter_potential(int a, int b, gOptions gConf, gMolecula gTop, gSingleBox &gBox, float &En, float &Vir){
    //Lennard-Jones potential
    float sumE; //energy
    float sumV;    //virial 
    float ra;   //distance beetwin atoms 
    float xa;   //
    float ya;
    float za;
    
    float dx;   //distance between molecules 
    float dy;
    float dz;
    float rcut;
    float rtest;
    
    //molecule number
    int curAtomNumA;
    int curAtomNumB;
    int id;
    
    //printf(" a %d type %d b %d typeb %d\n", a ,gBox.mType[a], b, gBox.mType[b]);
    
    curAtomNumA=gTop.aNum[gBox.mType[a]];
    curAtomNumB=gTop.aNum[gBox.mType[b]];
    sumE=0;
    sumV=0;
    rcut=0.5*gBox.boxLen[blockIdx.x];
    //printf("rcut %f\n", rcut);
    
    //molecule periodic boundary condition
    dx=(gBox.xm[b] - gBox.xm[a]);
    dy=(gBox.ym[b] - gBox.ym[a]);
    dz=(gBox.zm[b] - gBox.zm[a]);
    if(dx > rcut){  //
        dx = -rcut*2.0f + dx;
    }
    if(dy > rcut ){
        dy = -rcut*2.0f + dy;
    }
    if(dz > rcut){
        dz = -rcut*2.0f + dz;
    }
    if(dx < -rcut){ //
        dx = rcut*2.0f + dx;
    }
    if(dy < -rcut){
        dy = rcut*2.0f + dy;
    }
    if(dz < -rcut){
        dz = rcut*2.0f + dz;
    }
    //printf("test a %d b %d Na %d Nb %d  faa %d fab %d\n", a, b, curAtomNumA, curAtomNumB, gBox.fAtom[a], gBox.fAtom[b]);
    
    for(int i=0; i<curAtomNumA; i++){
        for(int j=0; j<curAtomNumB; j++){
            id = gBox.aType[gBox.fAtom[b] + j] * gConf.potNum[0] + gBox.aType[gBox.fAtom[a] + i];
            //printf("type a %d b %d testvar \n", gBox.aType[gBox.fAtom[b] + j], gBox.fAtom[a]);
            //id=0;
            
            xa = gBox.xa[gBox.fAtom[b] + j] - gBox.xa[gBox.fAtom[a] - i] + dx;
            ya = gBox.ya[gBox.fAtom[b] + j] - gBox.ya[gBox.fAtom[a] - i] + dy;
            za = gBox.za[gBox.fAtom[b] + j] - gBox.za[gBox.fAtom[a] - i] + dz;
            ra = xa * xa + ya * ya + za * za;
            if(ra > rcut*rcut){
                //rtest= sqrt(ra);
                ra = gTop.sigma[id] * gTop.sigma[id] / ra;
                ra = ra * ra * ra;
                //calculate potential
                
                sumE += gTop.epsi[id] * (ra* ra - ra);
                sumV += gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra);
            }
        }
    }
    gBox.mEnergy[a]+=sumE;
    //__syncthreads();
    gBox.mVirial[a]+=sumV;
    //printf("a %d b %d ra %f rm %f E %f V %f\n", a, b, rtest, sqrt(dx*dx+dy*dy+dz*dz), sumE, sumV);
//    En+=sumE;
//    Vir+=sumV;
    return 0;
}

__device__ int intra_potential(int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox){
    return 0;
}

__device__ int single_conf_change(int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox, hiprandState &devStates){
    float dx, dy, dz;
    //get type of change
    
    //molecule move
    dx = hiprand_uniform(&devStates);
    dy = hiprand_uniform(&devStates);
    dz = hiprand_uniform(&devStates);
    gBox.xm[a]+=dx*gBox.transMaxMove[blockIdx.x];
    gBox.ym[a]+=dy*gBox.transMaxMove[blockIdx.x];
    gBox.zm[a]+=dz*gBox.transMaxMove[blockIdx.x];
    return 0;
}

__device__ int single_one_energy(int yDim, int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox){
    __shared__ int maxMol;
    int curMol;
    float sumE;
    float sumV;
    __shared__ int reduce;
    
    maxMol = gBox.fMol[blockIdx.x]+blockDim.x*yDim;
    for(int i=0; i<yDim; i++){    //for several molecules per thread
        curMol = gBox.fMol[blockIdx.x]+threadIdx.x+i*blockDim.x;    //current number of molecule
        gBox.mEnergy[curMol]=0.0;
        gBox.mVirial[curMol]=0.0;
        if(a!=curMol){
            single_calc_one_potential(a, curMol, gConf, gTop, gBox, sumE, sumV);
        }
    }
}
