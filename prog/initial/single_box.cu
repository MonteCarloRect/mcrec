#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../mcrec.h"
#include <time.h>

#include "cuPrintf.cuh"
#include "cuPrintf.cu"

__global__ void single_calc(singleBox* gpuSingleBox, potentialParam* gpuParams,int yDim){
    //calculate initial enegry, pressure and other
    //printf("ololo id %d blockIdx.x  %d blockIdx.y %d blockDim.x \n", threadIdx.x, threadIdx.y, blockDim.x);
    single_calc_totenergy(yDim,gpuSingleBox,gpuParams);
    //main cycle
    
}

__device__ void single_calc_totenergy(int yDim, singleBox* gpuSingleBox, potentialParam* gpuParams){
    //printf("ololo 4");
    float* en;
    en=(float*)malloc(yDim*blockDim.x*sizeof(float));
    //hipMalloc(&en, yDim*blockDim.x*sizeof(float));
    
    for(int i=0;i<yDim;i++){    //for several molecules per thread
        int curMol=threadIdx.x+i*blockDim.x;
        //printf("id %d\n", curMol);
        for(int j=curMol++;j<yDim*blockDim.x;j++){
            //calculate energy curMol and j molecules
            single_calc_one_potential(curMol,j,gpuSingleBox,gpuParams);
        }
    }
}

//calculate potential
__device__ void single_calc_one_potential(int a, int b, singleBox* gpuSingleBox, potentialParam* gpuParams){
    intra_potential(a,b,gpuSingleBox,gpuParams);
    inter_potential();
}

__device__ void intra_potential(int a, int b, singleBox* gpuSingleBox, potentialParam* gpuMixParams){
    //Lennard-Jones potential
    float sumE; //energy
    float sumV;    //virial 
    float ra;   //distance beetwin atoms 
    float xa;   //
    float ya;
    float za;
    float dx;   //
    float dy;
    float dz;
    
    sumE=0;
    sumV=0;
    //check length/2
    //X
    if(gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b]>0.5*gpuSingleBox[blockIdx.x].boxLen){
        dx=gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b]-gpuSingleBox[blockIdx.x].boxLen;
    }
    else if(gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b]<-0.5*gpuSingleBox[blockIdx.x].boxLen){
        dx=(gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b])+gpuSingleBox[blockIdx.x].boxLen;
    }
    else{
        dx=gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b];
    }
    //Y
    if(gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]>0.5*gpuSingleBox[blockIdx.x].boxLen){
        dy=gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]-gpuSingleBox[blockIdx.x].boxLen;
    }
    else if(gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]<-0.5*gpuSingleBox[blockIdx.x].boxLen){
        dy=gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]+gpuSingleBox[blockIdx.x].boxLen;
    }
    else{
        dy=gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b];
    }
    //Z
    if(gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]>0.5*gpuSingleBox[blockIdx.x].boxLen){
        dz=gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]-gpuSingleBox[blockIdx.x].boxLen;
    }
    else if(gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]<-0.5*gpuSingleBox[blockIdx.x].boxLen){
        dz=gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]+gpuSingleBox[blockIdx.x].boxLen;
    }
    else{
        dz=gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b];
    }
    
    for(int i=0;i<gpuSingleBox[blockIdx.x].aNum[a];i++){
        for(int j=0;j<gpuSingleBox[blockIdx.x].aNum[b];j++){
            //check LJ potentail
            //calculate r
            xa=gpuSingleBox[blockIdx.x].xa[i]-gpuSingleBox[blockIdx.x].xa[j]-1;
            sumE+=1;
        }
    }
    
}

__device__ void inter_potential(){
    
}



