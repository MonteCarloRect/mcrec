#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../mcrec.h"
#include <time.h>

//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"

__global__ void single_calc(singleBox* gpuSingleBox, potentialParam* gpuParams,int yDim, mixParam** gpuMixParams){
    int test;
    //calculate initial enegry, pressure and other
    //printf("ololo id %d blockIdx.x  %d blockIdx.y %d blockDim.x \n", threadIdx.x, threadIdx.y, blockDim.x);
    test=single_calc_totenergy(yDim, gpuSingleBox, gpuParams, gpuMixParams);
    //main cycle
    test=gpuSingleBox[0].molNum;
    printf("test %f\n",gpuSingleBox[0].boxLen);
}

__device__ int single_calc_totenergy(int yDim, singleBox* gpuSingleBox, potentialParam* gpuParams, mixParam** gpuMixParams){
    float* en;
    //printf("my_test %d\n",blockIdx.y);
//    printf("my_test id %d blockIdx.x  %d blockIdx.y %d blockDim.x \n", threadIdx.x, threadIdx.y, blockDim.x);
//    en=(float*)malloc(yDim*blockDim.x*sizeof(float));
//    //hipMalloc(&en, yDim*blockDim.x*sizeof(float));
//    
//    for(int i=0;i<yDim;i++){    //for several molecules per thread
//        int curMol=threadIdx.x+i*blockDim.x;
//        //printf("id %d\n", curMol);
//        for(int j=curMol++;j<yDim*blockDim.x;j++){
//            //calculate energy curMol and j molecules
//            single_calc_one_potential(curMol, j, gpuSingleBox, gpuParams, gpuMixParams);
//        }
//    }
//    //summm potential
    return 0;
}

//calculate potential
__device__ void single_calc_one_potential(int a, int b, singleBox* gpuSingleBox, potentialParam* gpuParams, mixParam** gpuMixParams){
    intra_potential(a,b,gpuSingleBox,gpuParams, gpuMixParams);
    inter_potential();
}

__device__ void intra_potential(int a, int b, singleBox* gpuSingleBox, potentialParam* gpuParams, mixParam** gpuMixParams){
    //Lennard-Jones potential
    float sumE; //energy
    float sumV;    //virial 
    float ra;   //distance beetwin atoms 
    float xa;   //
    float ya;
    float za;
    float dx;   //
    float dy;
    float dz;
    float sig;
    float eps;
    
    sumE=0;
    sumV=0;
    //check length/2
    //X
    if(gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b]>0.5*gpuSingleBox[blockIdx.x].boxLen){
        dx=gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b]-gpuSingleBox[blockIdx.x].boxLen;
    }
    else if(gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b]<-0.5*gpuSingleBox[blockIdx.x].boxLen){
        dx=(gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b])+gpuSingleBox[blockIdx.x].boxLen;
    }
    else{
        dx=gpuSingleBox[blockIdx.x].xm[a]-gpuSingleBox[blockIdx.x].xm[b];
    }
    //Y
    if(gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]>0.5*gpuSingleBox[blockIdx.x].boxLen){
        dy=gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]-gpuSingleBox[blockIdx.x].boxLen;
    }
    else if(gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]<-0.5*gpuSingleBox[blockIdx.x].boxLen){
        dy=gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b]+gpuSingleBox[blockIdx.x].boxLen;
    }
    else{
        dy=gpuSingleBox[blockIdx.x].ym[a]-gpuSingleBox[blockIdx.x].ym[b];
    }
    //Z
    if(gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]>0.5*gpuSingleBox[blockIdx.x].boxLen){
        dz=gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]-gpuSingleBox[blockIdx.x].boxLen;
    }
    else if(gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]<-0.5*gpuSingleBox[blockIdx.x].boxLen){
        dz=gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b]+gpuSingleBox[blockIdx.x].boxLen;
    }
    else{
        dz=gpuSingleBox[blockIdx.x].zm[a]-gpuSingleBox[blockIdx.x].zm[b];
    }
    printf("check \n");
    for(int i=0;i<gpuSingleBox[blockIdx.x].aNum[a];i++){
        for(int j=0;j<gpuSingleBox[blockIdx.x].aNum[b];j++){
            //check LJ potentail
            //calculate r
            xa=gpuSingleBox[blockIdx.x].xa[i]-gpuSingleBox[blockIdx.x].xa[j]+dx;
            ya=gpuSingleBox[blockIdx.x].ya[i]-gpuSingleBox[blockIdx.x].ya[j]+dy;
            dz=gpuSingleBox[blockIdx.x].za[i]-gpuSingleBox[blockIdx.x].za[j]+dz;
            ra=xa*xa+ya*ya+za*za;
            
            eps=gpuMixParams[gpuSingleBox[blockIdx.x].aType[a][i]][gpuSingleBox[blockIdx.x].aType[b][j]].epsilon;
            sig=gpuMixParams[gpuSingleBox[blockIdx.x].aType[a][i]][gpuSingleBox[blockIdx.x].aType[b][j]].sigma;
            ra=sig*sig/ra;
            ra=ra*ra*ra;
            sumE+=4.0*eps*(ra*ra-ra);
            sumV+=4.0*eps*(6.0*ra-12*ra*ra);
        }
    }
    printf(" sumE  %f sum %f\n", sumE, sumV);
}

__device__ void inter_potential(){
    printf("ololo\n");
}



