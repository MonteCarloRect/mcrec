#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"

__device__ int single_conf_change(int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox, hiprandState &devStates);
__device__ int single_one_energy(int yDim, int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox);
__device__ int single_get_prop(int yDim, gOptions gConf, gMolecula gTop, gSingleBox &gBox, int curId);
__device__ int single_change_trans(gOptions gConf, gMolecula gTop, gSingleBox &gBox);

//
//#include <hiprand/hiprand_kernel.h>
//#include <hip/hip_runtime.h>

//

//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"

__global__ void single_calc(int yDim, gOptions gConf, gMolecula gTop, gSingleBox gBox){
    int test;
    __shared__ int randMol;
    hiprandState devStates;
    __shared__ float3 oldMol;
    __shared__ float3 oldState[MAXATOM];
    float deltaE;
    bool equlibrated;
    //calculate initial enegry, pressure and other
    //printf("ololo id %d blockIdx.x  %d blockIdx.y %d blockDim.x \n", threadIdx.x, threadIdx.y, blockDim.x);
    //test=single_calc_totenergy(yDim, gpuSingleBox, gpuParams, gpuMixParams);
    //main cycle
    //test=gpuSingleBox[0].molNum;
    test=1;
    //printf("deb 81 %d \n", gTop.aNum[0]);
    single_calc_totenergy(yDim, gConf, gTop, gBox);
    __syncthreads();
    hiprand_init(1234, threadIdx.x, 0, &devStates);
    //loop for change molecules position
    
    //EQULIBRATION
    equlibrated = false;
    while (equlibrated != true){
        for(int ncheck = 0; ncheck < EQBLOCKSIZE; ncheck++){
        for(int step= 0 ; step < EQBLOCKCHECK; step++){
            //get random numbers
            if(threadIdx.x==0){
                //gBox.curMol[blockIdx.x] = (hiprand_uniform(&devStates)*gBox.molNum[blockIdx.x]);
                randMol = hiprand_uniform(&devStates)*gBox.molNum[blockIdx.x] + gBox.fMol[blockIdx.x]; //number of molecule in array
                //printf("random %d %d  \n", i, randMol);
            }
            __syncthreads();
            //-------calculate old energy
            single_one_energy(yDim, randMol, gConf, gTop, gBox);
            if(threadIdx.x==0){ //get old energy
                gBox.oldEnergy[blockIdx.x] = gBox.mEnergy[gBox.fMol[blockIdx.x]]; //energy in zero molecule
                gBox.oldVirial[blockIdx.x] = gBox.mVirial[gBox.fMol[blockIdx.x]]; //virial i zero molecule
                //printf("old enegry %f virial %f \n", gBox.oldEnergy[blockIdx.x], gBox.oldVirial[blockIdx.x]);
                oldMol.x = gBox.xm[randMol];
                oldMol.y = gBox.ym[randMol];
                oldMol.z = gBox.zm[randMol];
                for(int i = 0; i < gTop.aNum[gBox.mType[randMol]]; i++){
                    oldState[i].x = gBox.xa[gBox.fAtom[randMol] + i];
                    oldState[i].y = gBox.ya[gBox.fAtom[randMol] + i];
                    oldState[i].z = gBox.za[gBox.fAtom[randMol] + i];
                }
            }
            __syncthreads();
            //------------change molecule configuration
            if(threadIdx.x==0){
                single_conf_change(randMol, gConf, gTop, gBox, devStates);
            }
            __syncthreads();
            //-------calculate new energy
            single_one_energy(yDim, randMol, gConf, gTop, gBox);
            if(threadIdx.x==0){ //get new energy
                gBox.newEnergy[blockIdx.x] = gBox.mEnergy[gBox.fMol[blockIdx.x]]; //energy in zero molecule
                gBox.newVirial[blockIdx.x] = gBox.mVirial[gBox.fMol[blockIdx.x]]; //virial i zero molecule
                //
            }
            //check aceptance
            if(threadIdx.x==0){
                deltaE=gBox.newEnergy[blockIdx.x] - gBox.oldEnergy[blockIdx.x];
                if(hiprand_uniform(&devStates) < exp(-gConf.Temp[blockIdx.x]*deltaE)){
//                printf("old enegry %f virial %f ", gBox.oldEnergy[blockIdx.x], gBox.oldVirial[blockIdx.x]);
//                printf("new enegry %f virial %f ", gBox.newEnergy[blockIdx.x], gBox.newVirial[blockIdx.x]);
//                printf(" dE %f w %f \n", deltaE, exp(-gConf.Temp[blockIdx.x]*deltaE));
                    //accept
                    //printf("accept\n");
                    gBox.accept[blockIdx.x]++;
                    //change energy
                    gBox.energy[blockIdx.x]+=gBox.newEnergy[blockIdx.x] - gBox.oldEnergy[blockIdx.x];
                    gBox.virial[blockIdx.x]+=gBox.newVirial[blockIdx.x] - gBox.oldVirial[blockIdx.x];
                }
                else{
                    //printf("reject\n");
                    gBox.reject[blockIdx.x]++;
                    //reject
                    //get old ccordinats
                    gBox.xm[randMol] = oldMol.x;
                    gBox.ym[randMol] = oldMol.y;
                    gBox.zm[randMol] = oldMol.z;
                    for(int i = 0; i < gTop.aNum[gBox.mType[randMol]]; i++){
                        gBox.xa[gBox.fAtom[randMol] + i] = oldState[i].x;
                        gBox.ya[gBox.fAtom[randMol] + i] = oldState[i].y;
                        gBox.za[gBox.fAtom[randMol] + i] = oldState[i].z;
                    }
                }
            }
            __syncthreads();
            
            
            //check maximumus
            if( (ncheck % 500 == 0) && (step==0) ){
                if(threadIdx.x==0){
                    printf("accept/reject %d / %d ", gBox.accept[blockIdx.x], gBox.reject[blockIdx.x]);
                }
                single_change_trans(gConf, gTop, gBox);
                if(threadIdx.x==0){
                    printf(" thr %d step %d energy %f virial %f max trans %f\n",threadIdx.x, ncheck, gBox.energy[blockIdx.x], gBox.virial[blockIdx.x], gBox.transMaxMove[blockIdx.x] );
                }
                single_calc_totenergy(yDim, gConf, gTop, gBox);
            }
            
        }
        //get properties
        single_get_prop(yDim, gConf, gTop, gBox, ncheck);
        
        }
        //check equlibration
        
        equlibrated = true;
    }
    if(threadIdx.x==0){
        printf("energy %f virial %f\n", gBox.energy[blockIdx.x], gBox.virial[blockIdx.x]);
        printf("accepted %d rejected %d\n", gBox.accept[blockIdx.x], gBox.reject[blockIdx.x]);
    }
    single_calc_totenergy(yDim, gConf, gTop, gBox);
}

__device__ int single_calc_totenergy(int yDim, gOptions gConf, gMolecula gTop, gSingleBox &gBox){
    int curMol;
    int curMol2;
    __shared__ int maxMol;
    float sumE;
    float sumV;
    __shared__ int reduce;
    maxMol = gBox.fMol[blockIdx.x]+blockDim.x*yDim;
    //printf("1 mol %d  lastmol %d\n", gBox.fMol[blockIdx.x], maxMol);
    for(int i = 0; i < yDim; i++){    //for several molecules per thread
        curMol=gBox.fMol[blockIdx.x]+i*blockDim.x+threadIdx.x;    //current number of molecule
//        gBox.mEnergyT[curMol]=0.0;
//        gBox.mVirialT[curMol]=0.0;
        gBox.mEnergy[curMol]=0.0;
        gBox.mVirial[curMol]=0.0;
//        sumE = 0.0;
//        sumV = 0.0;
        //intra_potential(curMol, gConf, gTop, gBox); //intramolecular energy
        for(int curMol2 = curMol; curMol2 < maxMol; curMol2++){  //get 
            //curMol2 = j; //gBox.fMol[blockIdx.x]+ i*blockDim.x + threadIdx.x;    //current molecule
            if(curMol != curMol2){
                inter_potential(curMol, curMol2, gConf, gTop, gBox, sumE, sumV);
            }
            else{
                intra_potential(curMol, gConf, gTop, gBox);
            }
        }
        gBox.mEnergyT[curMol] = gBox.mEnergy[curMol];
        gBox.mVirialT[curMol] = gBox.mVirial[curMol];
        //single_one_energy(yDim, curMol, gConf, gTop, gBox);
        //printf("nmol %d en %f vir %f \n", curMol, gBox.mEnergyT[curMol], gBox.mVirialT[curMol]);
    }
    __syncthreads();
  //sum potential   //change reduse
    if(threadIdx.x == 0){
        gBox.energy[blockIdx.x]=0.0;
        gBox.virial[blockIdx.x]=0.0;
        for(int i = gBox.fMol[blockIdx.x]; i < maxMol; i++){
            gBox.energy[blockIdx.x]+=gBox.mEnergyT[i];
            gBox.virial[blockIdx.x]+=gBox.mVirialT[i];
        }
        printf("0 mol %d last %d variant 1 total enegry %f virial %f \n", gBox.fMol[blockIdx.x], maxMol, gBox.energy[blockIdx.x], gBox.virial[blockIdx.x]);
    }
    __syncthreads();
    
// variant 2
    reduce = blockDim.x / 2;
//    gBox.energy[blockIdx.x]=0.0;
//    gBox.virial[blockIdx.x]=0.0;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            for(int i=0; i < yDim; i++){
                curMol2=gBox.fMol[blockIdx.x]+threadIdx.x+i*blockDim.x;
                curMol=gBox.fMol[blockIdx.x]+threadIdx.x+i*blockDim.x + reduce;
                gBox.mEnergyT[curMol2]+=gBox.mEnergyT[curMol];
                gBox.mVirialT[curMol2]+=gBox.mVirialT[curMol];
            }
        //printf(" reduce %d \n", reduce);
            reduce = reduce / 2;
        }
        __syncthreads();
        //
    }
    if(threadIdx.x == 0){
        gBox.energy[blockIdx.x] = 0.0;
        gBox.virial[blockIdx.x] = 0.0;
        for(int i=0; i<yDim; i++){
            gBox.energy[blockIdx.x] += gBox.mEnergyT[gBox.fMol[blockIdx.x]+blockDim.x*i];
            gBox.virial[blockIdx.x] += gBox.mVirialT[gBox.fMol[blockIdx.x]+blockDim.x*i];
        }
        //gBox.energy[blockIdx.x] = gBox.energy[blockIdx.x] / 2;
        //gBox.virial[blockIdx.x] = gBox.virial[blockIdx.x] / 2;
        //__syncthreads();
        printf("variant 2 total enegry %f virial %f \n", gBox.energy[blockIdx.x], gBox.virial[blockIdx.x]);
    }
    __syncthreads();
    return 0;
}

__device__ int inter_potential(int a, int b, gOptions gConf, gMolecula gTop, gSingleBox &gBox, float &En, float &Vir){
    //Lennard-Jones potential
    float sumE; //energy
    float sumV;    //virial 
    float ra;   //distance beetwin atoms 
    float xa;   //
    float ya;
    float za;
    
    float dx;   //distance between molecules 
    float dy;
    float dz;
    __shared__ float rcut;
    float rtest;
    
    //molecule number
    int curAtomNumA;
    int curAtomNumB;
    int id;
    
    //printf(" a %d type %d b %d typeb %d\n", a ,gBox.mType[a], b, gBox.mType[b]);
    
    curAtomNumA=gTop.aNum[gBox.mType[a]];   //number of atoms in molecule A
    curAtomNumB=gTop.aNum[gBox.mType[b]];   //numbers of atoms in molecule B
    sumE=0.0f;
    sumV=0.0f;
    rcut=0.5f*gBox.boxLen[blockIdx.x];
    //printf("rcut %f\n", rcut);
    
    //molecule periodic boundary condition
    dx=(gBox.xm[b] - gBox.xm[a]);
    dy=(gBox.ym[b] - gBox.ym[a]);
    dz=(gBox.zm[b] - gBox.zm[a]);
    if(dx > rcut){  //
        dx = -gBox.boxLen[blockIdx.x] + dx;
    }
    if(dy > rcut ){
        dy = -gBox.boxLen[blockIdx.x] + dy;
    }
    if(dz > rcut){
        dz = -gBox.boxLen[blockIdx.x] + dz;
    }
    if(dx < -rcut){ //
        dx = gBox.boxLen[blockIdx.x] + dx;
    }
    if(dy < -rcut){
        dy = gBox.boxLen[blockIdx.x] + dy;
    }
    if(dz < -rcut){
        dz = gBox.boxLen[blockIdx.x] + dz;
    }
    if(dx*dx+dy*dy+dz*dz < rcut*rcut){
        for(int i = 0; i < curAtomNumA; i++){
            for(int j = 0; j < curAtomNumB; j++){
                id = gBox.aType[gBox.fAtom[b] + j] * gConf.potNum[0] + gBox.aType[gBox.fAtom[a] + i];
                xa = gBox.xa[gBox.fAtom[b] + j] - gBox.xa[gBox.fAtom[a] + i] + dx;
                ya = gBox.ya[gBox.fAtom[b] + j] - gBox.ya[gBox.fAtom[a] + i] + dy;
                za = gBox.za[gBox.fAtom[b] + j] - gBox.za[gBox.fAtom[a] + i] + dz;
                ra = xa*xa + ya*ya + za*za;
                ra = gTop.sigma[id] * gTop.sigma[id] / ra;
                ra = ra * ra * ra;  //6 power
                //calculate potential
                sumE += gTop.epsi[id] * (ra* ra - ra);
                sumV += gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra);
            }
        }
    }
    //
    //printf("a %d b %d ra %f rm %f E %f V %f\n", a, b, rtest, sqrt(dx*dx+dy*dy+dz*dz), sumE, sumV);
    gBox.mEnergy[a]+=sumE;
    gBox.mVirial[a]+=sumV;
    En = 0.0;
    Vir = 0.0;
    return 0;
}

__device__ int intra_potential(int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox){
    gBox.mEnergy[a]+=0.0;
    gBox.mVirial[a]+=0.0;
    return 0;
}

__device__ int single_conf_change(int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox, hiprandState &devStates){
    float dx, dy, dz;
    //get type of change
    
    //molecule move
    dx = hiprand_uniform(&devStates)-0.5;
    dy = hiprand_uniform(&devStates)-0.5;
    dz = hiprand_uniform(&devStates)-0.5;
    gBox.xm[a]+=dx*gBox.transMaxMove[blockIdx.x];
    gBox.ym[a]+=dy*gBox.transMaxMove[blockIdx.x];
    gBox.zm[a]+=dz*gBox.transMaxMove[blockIdx.x];
    //
    if(gBox.xm[a]<0.0){
        gBox.xm[a]+=gBox.boxLen[blockIdx.x];
    }
    if(gBox.ym[a]<0.0){
        gBox.ym[a]+=gBox.boxLen[blockIdx.x];
    }
    if(gBox.zm[a]<0.0){
        gBox.zm[a]+=gBox.boxLen[blockIdx.x];
    }
    //
    if(gBox.xm[a]>gBox.boxLen[blockIdx.x]){
        gBox.xm[a]-=gBox.boxLen[blockIdx.x];
    }
    if(gBox.ym[a]>gBox.boxLen[blockIdx.x]){
        gBox.ym[a]-=gBox.boxLen[blockIdx.x];
    }
    if(gBox.zm[a]>gBox.boxLen[blockIdx.x]){
        gBox.zm[a]-=gBox.boxLen[blockIdx.x];
    }
    
    return 0;
}

__device__ int single_one_energy(int yDim, int a, gOptions gConf, gMolecula gTop, gSingleBox &gBox){
    __shared__ int maxMol;
    int curMol;
    int curMol2;
    float sumE;
    float sumV;
    __shared__ int reduce;

    maxMol = gBox.fMol[blockIdx.x] + blockDim.x*yDim;
    for(int i = 0; i < yDim; i++){    //for several molecules per thread
        curMol = gBox.fMol[blockIdx.x]+ i*blockDim.x + threadIdx.x;    //current molecule
        gBox.mEnergy[curMol]=0.0;
        gBox.mVirial[curMol]=0.0;
        if(a!=curMol){  //intermolecular energy
            inter_potential(curMol, a, gConf, gTop, gBox, sumE, sumV);
        }
        else{
            intra_potential(curMol, gConf, gTop, gBox);
        }
    }
    //reduse array
    __syncthreads();
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            for(int i = 0; i < yDim; i++){
                curMol2=gBox.fMol[blockIdx.x]+threadIdx.x+i*blockDim.x;
                curMol=gBox.fMol[blockIdx.x]+threadIdx.x+i*blockDim.x + reduce;
                gBox.mEnergy[curMol2]+=gBox.mEnergy[curMol];
                gBox.mVirial[curMol2]+=gBox.mVirial[curMol];
            }
        //printf(" reduce %d energy %f \n", reduce, gBox.mEnergy[reduce]);
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    if(threadIdx.x==0){
        for(int i=1; i < yDim; i++){
            gBox.mEnergy[gBox.fMol[blockIdx.x]]+=gBox.mEnergy[gBox.fMol[blockIdx.x]+i*blockDim.x];
            gBox.mVirial[gBox.fMol[blockIdx.x]]+=gBox.mVirial[gBox.fMol[blockIdx.x]+i*blockDim.x];
        }
        gBox.mEnergyT[a]=gBox.mEnergy[gBox.fMol[blockIdx.x]];
        gBox.mVirialT[a]=gBox.mVirial[gBox.fMol[blockIdx.x]];
    }
    //printf("nmol %d en %f vir %f\n", a, gBox.mEnergyT[a], gBox.mVirialT[a]);
    __syncthreads();
    return 0;
}

__device__ int single_get_prop(int yDim, gOptions gConf, gMolecula gTop, gSingleBox &gBox, int curId){
    //get energy
    //single_calc_totenergy(yDim, gConf, gTop, gBox);
    if(threadIdx.x==0){
        gBox.eqEnergy[blockIdx.x * EQBLOCKSIZE + curId] = gBox.energy[blockIdx.x] / gBox.molNum[blockIdx.x];
    
    // pressure p = nkT + 1/3 W Проверить коэффициенты
        gBox.eqPressure[blockIdx.x * EQBLOCKSIZE + curId] = gConf.Temp[blockIdx.x] * gBox.molNum[blockIdx.x] / gBox.boxVol[blockIdx.x] + gBox.virial[blockIdx.x] /3.0f / gBox.molNum[blockIdx.x];
//        printf("id %d energy %f pressure %f \n", curId, gBox.eqEnergy[blockIdx.x * EQBLOCKSIZE + curId], gBox.eqPressure[blockIdx.x * EQBLOCKSIZE + curId]);
    }
    //get virial
    
    //get density
    
    //get volume
    
    //get capacity
    
    
    
    return 0;
}

__device__ int single_change_trans(gOptions gConf, gMolecula gTop, gSingleBox &gBox){
    
    if(threadIdx.x==0){
        if((gBox.accept[blockIdx.x]+1)/(gBox.reject[blockIdx.x]+1)>0.6){
            gBox.transMaxMove[blockIdx.x]*=1.2;
        }
        if(gBox.transMaxMove[blockIdx.x]>gBox.boxLen[blockIdx.x]/2.0){
            gBox.transMaxMove[blockIdx.x]=gBox.boxLen[blockIdx.x]/2.0;
        }
    }
    if(threadIdx.x==1){
        if((gBox.accept[blockIdx.x]+1)/(gBox.reject[blockIdx.x]+1)<0.4){
            gBox.transMaxMove[blockIdx.x]*=0.8;
        }
        if(gBox.transMaxMove[blockIdx.x]<0.01){
            gBox.transMaxMove[blockIdx.x]=0.01;
        }
    }
    
    if(threadIdx.x==2){
        gBox.accept[blockIdx.x]=0;
        gBox.reject[blockIdx.x]=0;
    }
    __syncthreads;
    return 0;
}

