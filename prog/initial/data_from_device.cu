#include <stdio.h>
#include "../mcrec.h"
#include "../global.h"

int data_from_device(gSingleBox &gData, gSingleBox &hData, options config){
    //int sum;
    hipError_t cuErr;
    
    
    //copy molecules
    cuErr = hipMemcpy(hData.xm, gData.xm, hData.tMol*sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.xm file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(hData.ym, gData.ym, hData.tMol*sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.ym file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(hData.zm, gData.zm, hData.tMol*sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.zm file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    
    //copy atoms
    cuErr = hipMemcpy(hData.xa, gData.xa, hData.tAtom*sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.xa file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(hData.ya, gData.ya, hData.tAtom*sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.ya file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(hData.za, gData.za, hData.tAtom*sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.za file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    //copy averages energy
    cuErr = hipMemcpy(hData.avEnergy, gData.avEnergy, config.flowNum * sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.eqEnergy file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(hData.avPressure, gData.avPressure, config.flowNum * sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.eqPressure file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    //
    cuErr = hipMemcpy(hData.eqBlockEnergy, gData.eqBlockEnergy, config.flowNum * EQBLOCKS *sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.eqBlockEnergy file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(hData.eqBlockPressure, gData.eqBlockPressure, config.flowNum * EQBLOCKS * sizeof(float), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.eqBlockPressure file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    //copy total accept/rejected
    cuErr = hipMemcpy(hData.tAccept, gData.tAccept, config.flowNum * sizeof(int), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.eqPressure file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(hData.tReject, gData.tReject, config.flowNum * sizeof(int), hipMemcpyDeviceToHost);
    if(cuErr != hipSuccess){
        printf("Cannot copy from device box.eqPressure file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    return 0;
}
