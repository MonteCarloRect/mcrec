#include "hip/hip_runtime.h"
#include "../mcrec.h"

int data_to_device(gSingleBox &gBox, singleBox* &inputData, gOptions* &gConf, options &config, gMolecula* &gTop, potentialParam* Param, molecules* initMol, gSingleBox &hostData, gMolecula &hostTop, gOptions &hostConf, int deviceCount, gDoublebox* &gDBox){
    //allocate and copy data to GPU
    //gSingleBox hostData;
    int sum;
    int id;
    hipError_t cuErr;
    
    //===========================MOLECULES
    //numbers of molecules
    hostData.molNum=(int*)malloc(config.flowNum * sizeof(int));
    for(int i=0;i<config.flowNum;i++){
        hostData.molNum[i]=inputData[i].molNum;
    }
    cuErr = hipMalloc(&gBox.molNum, config.flowNum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.molNum memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.molNum, hostData.molNum, config.flowNum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    
    //coordinats of molecules
    sum=0;  //calculate total number of molecules
    for(int i=0; i<config.flowNum; i++){
        sum+=inputData[i].molNum;
    }
    hostData.tMol = sum;
    hostData.xm=(float*)malloc(sum * sizeof(float));
    hostData.ym=(float*)malloc(sum * sizeof(float));
    hostData.zm=(float*)malloc(sum * sizeof(float));
    hostData.mType=(int*)malloc(sum * sizeof(int));
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<inputData[i].molNum; j++){
            id=i*config.flowNum+j;
            hostData.xm[id]=inputData[i].xm[j];
            hostData.ym[id]=inputData[i].ym[j];
            hostData.zm[id]=inputData[i].zm[j];
            hostData.mType[id]=inputData[i].type[j];
            //printf("id %d xm %f ym %f zm %f type %d \n", id, hostData.xm[id], hostData.ym[id], hostData.zm[id], hostData.mType[id] );
        }
    }
    cuErr = hipMalloc(&gBox.xm,  sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.xm memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.ym,  sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.ym memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.zm,  sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.zm memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.mType,  sum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.mType memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    
    cuErr = hipMemcpy(gBox.xm, hostData.xm, sum*sizeof(float), hipMemcpyHostToDevice);
    cuErr = hipMemcpy(gBox.ym, hostData.ym, sum*sizeof(float), hipMemcpyHostToDevice);
    cuErr = hipMemcpy(gBox.zm, hostData.zm, sum*sizeof(float), hipMemcpyHostToDevice);
    cuErr = hipMemcpy(gBox.mType, hostData.mType, sum*sizeof(int), hipMemcpyHostToDevice);
    //numbers of molecules of each type
    sum=0;
    for(int i=0; i<config.flowNum; i++){
        sum+=config.subNum;
    }
    hostData.typeMolNum=(int*) malloc (sum * sizeof(int));
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<config.subNum; j++){
            id=config.flowNum*i+j;
            hostData.typeMolNum[id]=inputData[i].typeMolNum[j];
        }
    }
    hipMalloc(&gBox.typeMolNum,  sum*sizeof(int));
    hipMemcpy(gBox.typeMolNum, hostData.typeMolNum, sum*sizeof(int), hipMemcpyHostToDevice);
    
    //type of atoms
    
    
    
    //atoms
    sum=0;
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<config.subNum; j++){
            sum+=inputData[i].typeMolNum[j];
        }
    }
    hostData.tAtom = sum;
    printf(" 1 total numbers of atoms %d\n", sum);
    hostData.xa=(float*) malloc(sum*sizeof(float));
    hostData.ya=(float*) malloc(sum*sizeof(float));
    hostData.za=(float*) malloc(sum*sizeof(float));
    hostData.aType=(int*) malloc(sum*sizeof(int));
    id=0;
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<inputData[i].molNum; j++){
            for(int k=0; k<inputData[i].aNum[j]; k++){
                //id=i*(config.flowNum*config.subNum)+j*config.subNum+k;
                hostData.xa[id]=inputData[i].xa[j][k];
                hostData.ya[id]=inputData[i].ya[j][k];
                hostData.za[id]=inputData[i].za[j][k];
                hostData.aType[id]=inputData[i].aType[j][k];
                //printf("test 223 %d \n", hostData.aType[id]);
                id++;
            }
        }
    }
    cuErr = hipMalloc(&gBox.xa, sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.xa memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.ya, sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.ya memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.za, sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.za memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.aType, sum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.aType memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.xa, hostData.xa, sum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    cuErr = hipMemcpy(gBox.ya, hostData.ya, sum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    cuErr = hipMemcpy(gBox.za, hostData.za, sum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    cuErr = hipMemcpy(gBox.aType, hostData.aType, sum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    
    //get first molecules and atoms of flows
    hostData.fMol=(int*) malloc(config.flowNum*sizeof(int));
    hipMalloc(&gBox.fMol, config.flowNum*sizeof(int));
    hostData.fMol[0]=0; //first molecule index 0
    sum=inputData[0].molNum;
    for(int i=1; i<config.flowNum; i++){
        //printf("my test %d", i);
        hostData.fMol[i]=hostData.fMol[i-1]+inputData[i].molNum;
        sum+=inputData[i].molNum;
    }
    printf(" 2 total numbers of atoms %d\n", sum);
    hipMemcpy(gBox.fMol, hostData.fMol, config.flowNum*sizeof(int), hipMemcpyHostToDevice);
    hostData.fAtom=(int*) malloc(sum*sizeof(int));
    cuErr = hipMalloc(&gBox.fAtom, sum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.fAtom memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    hostData.fAtom[0]=0;    //first atom
    id=0;
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<inputData[i].molNum; j++){
            //printf("my test %d %d\n",i,j);
            //molecule type --- inputData[i].type[j]
            if(id!=0){
                hostData.fAtom[id]=hostData.fAtom[id-1]+initMol[inputData[i].type[j]].atomNum;
                //printf("test data %d\n", hostData.fAtom[id]);
            }
            id++;
        }
    }
    cuErr = hipMemcpy(gBox.fAtom, hostData.fAtom, sum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    //box length
    hostData.boxLen=(float*) malloc(config.flowNum*sizeof(float));
    hostData.boxVol=(float*) malloc(config.flowNum*sizeof(float));
    hipMalloc(&gBox.boxLen, config.flowNum * sizeof(float));
    hipMalloc(&gBox.boxVol, config.flowNum * sizeof(float));
    for(int i=0; i<config.flowNum; i++){
        hostData.boxLen[i]=inputData[i].boxLen;
        hostData.boxVol[i]=inputData[i].boxLen*inputData[i].boxLen*inputData[i].boxLen;
    }
    hipMemcpy(gBox.boxLen, hostData.boxLen, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gBox.boxVol, hostData.boxVol, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    //energy malloc
    cuErr = hipMalloc(&gBox.virial, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.virial memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.energy, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.energy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.pressure, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.pressure memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    
    cuErr =hipMalloc(&gBox.oldEnergy, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.oldEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.oldVirial, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.oldVirial memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.newEnergy, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.newEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.newVirial, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.newVirial memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    hostData.virial=(float*) malloc(config.flowNum*sizeof(float));
    hostData.energy=(float*) malloc(config.flowNum*sizeof(float));
    hostData.pressure=(float*) malloc(config.flowNum*sizeof(float));
    hostData.oldEnergy=(float*) malloc(config.flowNum*sizeof(float));
    hostData.oldVirial=(float*) malloc(config.flowNum*sizeof(float));
    hostData.newEnergy=(float*) malloc(config.flowNum*sizeof(float));
    hostData.newVirial=(float*) malloc(config.flowNum*sizeof(float));
    for(int i=0; i < config.flowNum; i++){
        hostData.virial[i]=0.0;
        hostData.energy[i]=0.0;
        hostData.pressure[i] = 0.0f;
        hostData.oldEnergy[i]=0.0;
        hostData.oldVirial[i]=0.0;
        hostData.newEnergy[i]=0.0;
        hostData.newVirial[i]=0.0;
    }
    hostData.mVirial=(float*) malloc(sum*sizeof(float));
    hostData.mEnergy=(float*) malloc(sum*sizeof(float));
    hostData.mVirialT=(float*) malloc(sum*sizeof(float));
    hostData.mEnergyT=(float*) malloc(sum*sizeof(float));
    for(int i=0; i < sum; i++){
        hostData.mVirial[i]=0.0;
        hostData.mEnergy[i]=0.0;
        hostData.mVirialT[i]=0.0;
        hostData.mEnergyT[i]=0.0;
    }
    cuErr = hipMemcpy(gBox.virial, hostData.virial, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.energy, hostData.energy, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.pressure, hostData.pressure, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.oldEnergy, hostData.oldEnergy, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    cuErr = hipMemcpy(gBox.oldVirial, hostData.oldVirial, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    cuErr = hipMemcpy(gBox.newEnergy, hostData.newEnergy, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    cuErr = hipMemcpy(gBox.newVirial, hostData.newVirial, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        
    cuErr = hipMalloc(&gBox.mEnergy, sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.mEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.mVirial, sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.mVirial memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.mEnergyT, sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.mEnergyT memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.mVirialT, sum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.mVirialT memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.mEnergy, hostData.mEnergy, sum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.mVirial, hostData.mVirial, sum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.mEnergyT, hostData.mEnergyT, sum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.mVirialT, hostData.mVirialT, sum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    //transformation
    cuErr =hipMalloc(&gBox.curMol, config.flowNum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.curMol memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.accept, config.flowNum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.accept memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.reject, config.flowNum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.reject memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.tAccept, config.flowNum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.tAccept memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.tReject, config.flowNum*sizeof(int));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.tReject memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    hostData.curMol=(int*) malloc(config.flowNum*sizeof(int));
    hostData.accept = (int*) malloc(config.flowNum*sizeof(int));
    hostData.reject = (int*) malloc(config.flowNum*sizeof(int));
    hostData.tAccept = (int*) malloc(config.flowNum * sizeof(int));
    hostData.tReject = (int*) malloc(config.flowNum * sizeof(int));
    for(int i=0; i<config.flowNum; i++){
        hostData.curMol[i]=0;
        hostData.accept[i]=0;
        hostData.reject[i]=0;
        hostData.tReject[i]=0;
        hostData.tAccept[i]=0;
    }
    cuErr = hipMemcpy(gBox.curMol, hostData.curMol, config.flowNum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.accept, hostData.accept, config.flowNum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.reject, hostData.reject, config.flowNum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.tAccept, hostData.tAccept, config.flowNum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.tReject, hostData.tReject, config.flowNum*sizeof(int), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    
    cuErr =hipMalloc(&gBox.transMaxMove, config.flowNum*sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.transMaxMove memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    hostData.transMaxMove=(float*) malloc(config.flowNum*sizeof(float));
    for(int i=0; i<config.flowNum; i++){
        hostData.transMaxMove[i]=0.05;
    }
    cuErr = hipMemcpy(gBox.transMaxMove, hostData.transMaxMove, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    //------------------block enegry
    cuErr =hipMalloc(&gBox.eqBlockEnergy, config.flowNum * EQBLOCKS * sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.eqBlockEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.eqBlockPressure, config.flowNum * EQBLOCKS * sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.eqBlockEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.avEnergy, config.flowNum * sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.avEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMalloc(&gBox.avPressure, config.flowNum * sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.avPressure memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    hostData.eqBlockEnergy = (float*) malloc(config.flowNum * EQBLOCKS * sizeof(float));
    hostData.eqBlockPressure = (float*) malloc(config.flowNum * EQBLOCKS * sizeof(float));
    hostData.avEnergy = (float*) malloc(config.flowNum * sizeof(float));
    hostData.avPressure = (float*) malloc(config.flowNum * sizeof(float));
    hostData.energyCorr = (float*) malloc(config.flowNum * EQBLOCKS * sizeof(float));
    hostData.pressureCorr = (float*) malloc(config.flowNum * EQBLOCKS * sizeof(float));
    for(int i = 0; i < config.flowNum * EQBLOCKS; i++){
        hostData.eqBlockEnergy[i]=0.0;
        hostData.eqBlockPressure[i]=0.0;
    }
    for(int i = 0; i < config.flowNum; i++){
        hostData.avEnergy[i] = 0.0f;
        hostData.avPressure[i] = 0.0f;
        //printf("testtststs  %f   %f", hostData.avEnergy[i], hostData.avPressure[i]);
    }
    cuErr = hipMemcpy(gBox.eqBlockEnergy, hostData.eqBlockEnergy, config.flowNum * EQBLOCKS * sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.eqBlockPressure, hostData.eqBlockPressure, config.flowNum * EQBLOCKS * sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.avEnergy, hostData.avEnergy, config.flowNum * sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        //return 2;
    }
    cuErr = hipMemcpy(gBox.avPressure, hostData.avPressure, config.flowNum * sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    //---------------------block size
    cuErr =hipMalloc(&gBox.eqEnergy, config.flowNum * EQBLOCKSIZE * sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.eqEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr =hipMalloc(&gBox.eqPressure, config.flowNum * EQBLOCKSIZE * sizeof(float));
    if(cuErr != hipSuccess){
        printf("Cannot allocate box.eqEnergy memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    hostData.eqEnergy = (float*)malloc(config.flowNum * EQBLOCKSIZE * sizeof(float));
    hostData.eqPressure = (float*)malloc(config.flowNum * EQBLOCKSIZE * sizeof(float));
    for(int i=0; i < config.flowNum * EQBLOCKSIZE; i++){
        hostData.eqEnergy[i] = 0.0;
        hostData.eqPressure[i] = 0.0;
    }
    cuErr = hipMemcpy(gBox.eqEnergy, hostData.eqEnergy, config.flowNum * EQBLOCKSIZE * sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    cuErr = hipMemcpy(gBox.eqPressure, hostData.eqPressure, config.flowNum * EQBLOCKSIZE * sizeof(float), hipMemcpyHostToDevice);
    if(cuErr != hipSuccess){
        printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    
    //============================TOPOLOGY
    //matrix
    hostTop.sigma=(float*) malloc (config.potNum * config.potNum * sizeof(float));
    hostTop.epsi=(float*) malloc (config.potNum * config.potNum * sizeof(float));
    hostTop.charge=(float*) malloc (config.potNum * config.potNum * sizeof(float));
    //single atom
    hostTop.aNum=(int*) malloc (config.subNum * sizeof(int));
    for(int i=0; i<config.subNum; i++){
        
        hostTop.aNum[i]=initMol[i].atomNum;
        printf("test topology %d %d \n", i, hostTop.aNum[i]);
    }
    
//    printf("potnum %d\n", config.potNum);
    for(int i = 0; i < config.potNum; i++){
        for(int j = 0; j < config.potNum; j++){
            id = i * config.potNum + j;
//            printf("id %d i %d j %f\n", id, i, Param[i].sigma);
            //add mixture rule
            hostTop.sigma[id]=0.5*(Param[i].sigma+Param[j].sigma);
//            printf("sigma %f\n", hostTop.sigma[id]);
            hostTop.epsi[id]=4.0f*sqrt(Param[i].epsilon*Param[j].epsilon)*1000.0/R; //from kJ/mol to [K] kB
//            printf("epsi %f\n", hostTop.epsi[id]);
            hostTop.charge[id]=Param[i].charge*Param[j].charge;
//            printf("charge %f\n", hostTop.charge[id]);
            //add mixture rule
            printf("sigma %f  %f   %f\n", hostTop.sigma[id] ,Param[i].sigma, Param[j].sigma);
            printf("epsilon %f  %f   %f\n", hostTop.epsi[id] ,Param[i].epsilon, Param[j].epsilon);
            printf("charge %f  %f    %f\n", hostTop.charge[id] ,Param[i].charge, Param[j].charge);
        }
    }
//    printf("OLOLO5\n");
    //gpu data
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipStreamCreate(&gDBox[curDev].stream);  //create stream
        if(cuErr != hipSuccess){
            printf("Cannot create stream on device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        cuErr = hipMalloc(&gTop[curDev].sigma, config.potNum * config.potNum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate top[curDev].sigma memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gTop[curDev].epsi, config.potNum * config.potNum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate top[curDev].epsilon memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gTop[curDev].charge, config.potNum * config.potNum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate top[curDev].charge memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gTop[curDev].aNum, config.subNum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate top[curDev].aNum memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gTop[curDev].sigma, hostTop.sigma, config.potNum * config.potNum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gTop[curDev].epsi, hostTop.epsi, config.potNum * config.potNum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gTop[curDev].charge, hostTop.charge, config.potNum * config.potNum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gTop[curDev].aNum, hostTop.aNum, config.subNum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //set atom types
        hostTop.aType = (int*) malloc(config.subNum * MAXATOM * sizeof(int));
        cuErr = hipMalloc(&gTop[curDev].aType, config.subNum * MAXATOM * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate top[curDev].aType memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        for(int i = 0; i < config.subNum; i++){
            for(int j = 0; j < MAXATOM; j++){
                if(j < initMol[i].atomNum){
                    hostTop.aType[i * MAXATOM + j] = initMol[i].aType[j];
                }
                else{
                    hostTop.aType[i * MAXATOM + j] = 0.0;
                }
                
            }
        }
        cuErr = hipMemcpy(gTop[curDev].aType, hostTop.aType, config.subNum  * MAXATOM * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    }
    //============================CONFIG
    hostConf.Temp=(float*) malloc(config.flowNum*sizeof(float));
    hostConf.potNum=(int*) malloc(sizeof(int));
    hostConf.subNum=(int*) malloc(sizeof(int));
    for(int i=0; i<config.flowNum; i++){
        hostConf.Temp[i]=config.flowT[i];
    }
    hostConf.potNum[0]=config.potNum;   //to arrays
    hostConf.subNum[0] = config.subNum; //to array
    printf("hostconf %d \n", hostConf.potNum[0]);
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gConf[curDev].subNum, sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gConf[curDev].subNum memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gConf[curDev].Temp, config.flowNum*sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gConf[curDev].Temp memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gConf[curDev].subNum, hostConf.subNum, sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gConf[curDev].Temp, hostConf.Temp, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gConf[curDev].potNum, sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gConf[curDev].potNum memory file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gConf[curDev].potNum, hostConf.potNum, sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    }
    
    return 0;
}



