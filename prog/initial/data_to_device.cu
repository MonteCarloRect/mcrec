#include "hip/hip_runtime.h"
#include "../mcrec.h"

int data_to_device(gSingleBox &gBox, singleBox* &inputData, gOptions gConf, options &config, gMolecula gTop, potentialParam* Param, molecules* initMol){
    //allocate and copy data to GPU
    gSingleBox hostData;
    int sum;
    int id;
    //===========================MOLECULES
    //numbers of molecules
    hostData.molNum=(unsigned int*)malloc(config.flowNum * sizeof(int));
    for(int i=0;i<config.flowNum;i++){
        hostData.molNum[i]=inputData[i].molNum;
    }
    hipMalloc(&gBox.molNum, config.flowNum*sizeof(int));
    hipMemcpy(gBox.molNum, hostData.molNum, config.flowNum*sizeof(singleBox), hipMemcpyHostToDevice);
    
    //coordinats of molecules
    sum=0;  //calculate total number oof molecules
    for(int i=0;i<config.flowNum;i++){
        sum+=inputData[i].molNum;
    }
    hostData.xm=(float*)malloc(sum * sizeof(float));
    hostData.ym=(float*)malloc(sum * sizeof(float));
    hostData.zm=(float*)malloc(sum * sizeof(float));
    hostData.mType=(unsigned int*)malloc(sum * sizeof(int));
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<inputData[i].molNum; j++){
            id=i*config.flowNum+j;
            hostData.xm[id]=inputData[i].xm[j];
            hostData.ym[id]=inputData[i].ym[j];
            hostData.zm[id]=inputData[i].zm[j];
            hostData.mType[id]=inputData[i].type[j];
        }
    }
    hipMalloc(&gBox.xm,  sum*sizeof(float));
    hipMalloc(&gBox.ym,  sum*sizeof(float));
    hipMalloc(&gBox.zm,  sum*sizeof(float));
    hipMalloc(&gBox.mType,  sum*sizeof(int));
    
    hipMemcpy(gBox.xm, hostData.xm, sum*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gBox.ym, hostData.ym, sum*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gBox.zm, hostData.zm, sum*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gBox.mType, hostData.mType, sum*sizeof(int), hipMemcpyHostToDevice);
    //numbers of molecules of each type
    sum=0;
    for(int i=0; i<config.flowNum; i++){
        sum+=config.subNum;
    }
    hostData.typeMolNum=(unsigned int*) malloc (sum * sizeof(int));
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<config.subNum; j++){
            id=config.flowNum*i+j;
            hostData.typeMolNum[id]=inputData[i].typeMolNum[j];
        }
    }
    hipMalloc(&gBox.typeMolNum,  sum*sizeof(int));
    hipMemcpy(gBox.typeMolNum, hostData.typeMolNum, sum*sizeof(int), hipMemcpyHostToDevice);
    
    //type of atoms
    
    
    
    //atoms
    sum=0;
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<config.subNum; j++){
            sum+=inputData[i].typeMolNum[j];
        }
    }
    hostData.xa=(float*) malloc(sum*sizeof(float));
    hostData.ya=(float*) malloc(sum*sizeof(float));
    hostData.za=(float*) malloc(sum*sizeof(float));
    hostData.aType=(unsigned int*) malloc(sum*sizeof(int));
    for(int i=0; i<config.flowNum; i++){
        for(int j=0; j<config.subNum; j++){
            for(int k=0; k<inputData[i].typeMolNum[j]; k++){
                id=i*(config.flowNum*config.subNum)+j*config.subNum+k;
                hostData.xa[id]=inputData[i].xa[j][k];
                hostData.ya[id]=inputData[i].ya[j][k];
                hostData.za[id]=inputData[i].za[j][k];
                hostData.aType[id]=inputData[i].aType[j][k];
            }
        }
    }
    hipMalloc(&gBox.xa, sum*sizeof(float));
    hipMalloc(&gBox.ya, sum*sizeof(float));
    hipMalloc(&gBox.za, sum*sizeof(float));
    hipMalloc(&gBox.aType, sum*sizeof(int));
    hipMemcpy(gBox.xa, hostData.xa, sum*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gBox.ya, hostData.ya, sum*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gBox.za, hostData.za, sum*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gBox.aType, hostData.aType, sum*sizeof(int), hipMemcpyHostToDevice);
    
    //============================TOPOLOGY
    gMolecula hostTop;
    //matrix
    hostTop.sigma=(float*) malloc (config.potNum * config.potNum * sizeof(float));
    hostTop.epsi=(float*) malloc (config.potNum * config.potNum * sizeof(float));
    hostTop.charge=(float*) malloc (config.potNum * config.potNum * sizeof(float));
    //single atom
    hostTop.aNum=(unsigned int*) malloc (config.subNum * sizeof(int));
    for(int i=0; i<config.subNum; i++){
        hostTop.aNum[i]=initMol[i].atomNum;
    }
    
//    printf("potnum %d\n", config.potNum);
    for(int i=0; i<config.potNum; i++){
        for(int j=0; j<config.potNum; j++){
            id=i*config.potNum+j;
//            printf("id %d i %d j %f\n", id, i, Param[i].sigma);
            //add mixture rule
            hostTop.sigma[id]=0.5*(Param[i].sigma+Param[j].sigma);
//            printf("sigma %f\n", hostTop.sigma[id]);
            hostTop.epsi[id]=sqrt(Param[i].epsilon*Param[j].epsilon);
//            printf("epsi %f\n", hostTop.epsi[id]);
            hostTop.charge[id]=Param[i].charge*Param[i].charge;
//            printf("charge %f\n", hostTop.charge[id]);
            //add mixture rule
        }
    }
//    printf("OLOLO5\n");
    //gpu data
    hipMalloc(&gTop.sigma, config.potNum * config.potNum * sizeof(float));
    hipMalloc(&gTop.epsi, config.potNum * config.potNum * sizeof(float));
    hipMalloc(&gTop.charge, config.potNum * config.potNum * sizeof(float));
    hipMalloc(&gTop.aNum, config.subNum * sizeof(int));
    
    hipMemcpy(gTop.sigma, hostTop.sigma, config.potNum * config.potNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gTop.epsi, hostTop.epsi, config.potNum * config.potNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gTop.charge, hostTop.charge, config.potNum * config.potNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gTop.aNum, hostTop.aNum, config.subNum * sizeof(int), hipMemcpyHostToDevice);
    
    //============================CONFIG
    gOptions hostConf;
    hostConf.Temp=(float*) malloc(config.flowNum*sizeof(float));
    for(int i=0; i<config.flowNum; i++){
        hostConf.Temp[i]=config.flowT[i];
    }
    hipMalloc(&gConf.Temp, config.flowNum*sizeof(float));
    hipMemcpy(gConf.Temp, hostConf.Temp, config.flowNum*sizeof(float), hipMemcpyHostToDevice);
    
    return 0;
}



