//
#include <stdio.h>
#include "../mcrec.h"

int freeAll(singleBox* &gpuSingleBox,singleBox* &initFlows,options config){

hipFree(gpuSingleBox);

for(int i=0; i<config.flowNum;i++){
    for(int j=0;j<initFlows[i].molNum;j++){
        free(initFlows[i].xa[j]);
        free(initFlows[i].ya[j]);
        free(initFlows[i].za[j]);
    }
    free(initFlows[i].xm);
    free(initFlows[i].ym);
    free(initFlows[i].zm);
    free(initFlows[i].typeMolNum);
    free(initFlows[i].type);
}
free(initFlows);
return 1;
}
