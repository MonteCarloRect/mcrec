#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"
//#include "double_trans.cu"

//__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int curDev);
//__device__  int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim);
//__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int fmol, int smol, float rcut);
//__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int mol);

//double_step 
__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int curDev);
__device__  int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim);
__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int fmol, int smol, float rcut);
__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int mol);

//double_trans
__device__ int double_transition(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, int curMol, hiprandState devStates);

//double volume change
__device__ int double_vol_change(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState devStates);

///////

int double_equilibration(gDoublebox* gDBox, hDoubleBox doubleBox, gOptions* gConf, gMolecula* gTop){
    hipError_t cuErr;
    int* xDim;
    int temp;
    
    xDim = (int*) malloc(deviceCount * sizeof(int));
    for(int curDev = 0; curDev < deviceCount; curDev++){
        
        //set numbers of block equal to numbers of plate per device
        //numbers of thread equal to maximum numbers of molecules 
        temp = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //get maximum number of molecules in liquib phase
//            printf("liq %d plate %d molecules %d\n", curDev, doubleBox.platesPerDevice[curDev][i], doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]]);
            if(doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]] > temp){
                temp = doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]];
            }
        }
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //get maximum numbers of molecules in vapor phase
//            printf("vap %d plate %d molecules %d\n", curDev, doubleBox.platesPerDevice[curDev][i], doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]]);
            if(doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]] > temp){
                temp = doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]];
            }
        }
//        printf("device max %d\n",doubleBox.devicePlates[curDev]);
        if(temp == 0){
            xDim[curDev] = 1;
        }
        else{
            if(log2(temp) > 7){
                xDim[curDev] = MAXDIM; //set blocksize as maximum dimension
            }
            else{
                xDim[curDev] = pow(2,ceil(log2(temp))); //set block size 
            }
        }
    }
    hipDeviceSynchronize();    //sync
    printf("device count %d\n", deviceCount);
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //calculate equlibration
        printf("dev %d block %d thread %d\n", curDev, doubleBox.devicePlates[curDev], xDim[curDev]);
        double_equilib_cycle<<<doubleBox.devicePlates[curDev], xDim[curDev] >>>(gDBox[curDev], gConf[curDev], gTop[curDev], curDev);
        cuErr = hipGetLastError();
        if(cuErr != hipSuccess){
            printf("Error on device %d %s line %d, err: %s\n",curDev, __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //0, gDBox[curDev].stream
    }
    hipDeviceSynchronize();    //sync after complit all equlibrations
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        hipStreamSynchronize(gDBox[curDev].stream);
        
    }
    return 0;
}

__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int curDev){
    int temp;
    int yDim;   //y Dimension of array
    float randTrans;    //random transition
    int randMol;    //random molecule
    hiprandState devStates;
    
    
    hiprand_init(1234, threadIdx.x, 0, &devStates);
    
    //check
    
    yDim = (gDBox.molNum[blockIdx.x] / blockDim.x) + 1; //set y dimension
    if(gDBox.molNum[blockIdx.x] > 0){
        if(threadIdx.x == 0){
            
            printf("molecules on device %d mol %d bdim %d ydim %d \n", curDev, gDBox.molNum[blockIdx.x], blockDim.x , yDim);
        }
        __syncthreads;  //sync yDim
        if(threadIdx.x == 0){   //shared for whjole block
    //        printf("device %d thread %d\n", curDev, threadIdx.x);
    //        printf("sigma %f %f %f %f, cuurent device %d\n", gTop.sigma[0], gTop.sigma[1], gTop.sigma[2], gTop.sigma[3], curDev );
    //        printf("ref en %f device %d nmol %d\n", gDBox.refEn[blockIdx.x], curDev, gDBox.molNum[blockIdx.x]);
    //        printf("mol id %d x %f y %f z %f \n", gDBox.fMolOnPlate[blockIdx.x], gDBox.xm[4], gDBox.ym[4], gDBox.zm[4]);
        }
    //    __syncthreads();
        double_totalen(gDBox, gConf, gTop, yDim);
    //    printf("block %d molecules %d\n", blockIdx.x, gDBox.molNum[blockIdx.x]);
        if(gDBox.molNum[blockIdx.x] > 0){ //cycle only if plate not empty
            
        
            for(int i = 0; i < 1000; i++){  //loop for some (set as a parameter to option file)
                randTrans = hiprand_uniform(&devStates); //get random transition
                if(randTrans < 0.5){    //move molecule
                    randMol = hiprand_uniform(&devStates) * gDBox.molNum[blockIdx.x] + gDBox.fMolOnPlate[blockIdx.x];
                    double_transition(gDBox, gConf, gTop, yDim, randMol, devStates);
                }
                else if(randTrans < 0.01){  //change volume
                    double_vol_change(gDBox, gConf, gTop, yDim, devStates);
                }
                else if(randTrans < 0.001){ //molecule transition
                    
                }
                //print properties
                if(threadIdx.x ==0){
                    printf("step %d liquid energy %f accept %d\n", i, gDBox.liqEn[blockIdx.x], gDBox.accLiqTrans[blockIdx.x]);
                }
            }
            //check equlibration status
        }
        else{
            //mark block as a compleet
        }
    }
    else{
        if(threadIdx.x == 0){
            printf("no molecules on block %d device %d\n", blockIdx.x, curDev);
        }
    }
}

__device__ int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim){
    int curMol; //current molecule
    int curMol2;    //second molecule
    int curId;  //current index
    int reduce;
    //nuber of plate equal block number 
//        if(threadIdx.x == 0){   //shared for whjole block
//         printf("test 13.02-2\n");
//        }
    //printf("start totalen %d\n",threadIdx.x);
    
    //======calculate liquid phase energy/virial
    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;  //set energy to zero
    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0; //set virial to zero
    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;  //set energy to zero
    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0; //set virial to zero
    //printf("done %d yDim %d\n", threadIdx.x, yDim);

    for(int i = 0; i < yDim; i++){
        curId = threadIdx.x * yDim + i; //current id of molecule in liqud list
        curMol = gDBox.fMolOnPlate[blockIdx.x] + gDBox.liqList[curId];  //get curent molecule GPU index
        if(gDBox.phaseType[curMol] == LIQ){  //calculate liquid energy
            //printf("cur mol %d mol liq %d \n", curId, gDBox.nLiq[blockIdx.x]);
            if(curId < gDBox.nLiq[blockIdx.x]){
                
                //set here intermolecular potential
                //printf("mol1 %d mol2 %d\n", curMol, curId);
                //calculate inermolecullar interaction
                for(int j = curId + 1; j < gDBox.nLiq[blockIdx.x]; j++){
                    curMol2 = gDBox.fMolOnPlate[blockIdx.x] + gDBox.liqList[j];
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
                }
            }
            else{
                //set energy to zero 
                gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += 0.0;
                gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += 0.0;
            }
        }
        else{   //calculate vapor energy
            if(curId < gDBox.nVap[blockIdx.x]){
                
                //set here intermolecular potential
                //printf("mol1 %d mol2 %d\n", curMol, curId);
                //calculate inermolecullar interaction
                for(int j = curId + 1; j < gDBox.nLiq[blockIdx.x]; j++){
                    curMol2 = gDBox.fMolOnPlate[blockIdx.x] + gDBox.liqList[j];
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
                }
            }
        }
    }
    //printf("%f ", gDBox.tempEn[blockIdx.x * MAXDIM + threadIdx.x]);
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    __syncthreads();
    if(threadIdx.x == 0){
        printf("device block %d liq molecules %d\n", blockIdx.x, gDBox.nLiq[blockIdx.x]);
        printf("plate %d total liquid energy %f vapor energy %f\n", blockIdx.x, gDBox.tempLiqEn[blockIdx.x * MAXDIM], gDBox.tempVapEn[blockIdx.x * MAXDIM]);
        gDBox.liqEn[blockIdx.x] = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
        gDBox.vapEn[blockIdx.x] = gDBox.tempVapEn[blockIdx.x * MAXDIM];
        gDBox.liqVir[blockIdx.x] = gDBox.tempLiqVir[blockIdx.x * MAXDIM];
        gDBox.vapVir[blockIdx.x] = gDBox.tempVapVir[blockIdx.x * MAXDIM];
    }
    __syncthreads();
    return 0;
}

__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int fmol, int smol, float rcut){
    //fmol smol - indexes of firs and second molecules
    //get atom numbers
    float dmx;  //diffs in molecule coordinats
    float dmy;
    float dmz;
    float dax;  //diff in atoms coordinate
    float day;
    float daz;
    float r;    //distance ^2 of molecules
    float ra;   //distance ^2 of atoms
    int id;     //array indexes for potential parameters
    
    dmx = gDBox.xm[fmol] - gDBox.xm[smol];
    dmy = gDBox.ym[fmol] - gDBox.ym[smol];
    dmz = gDBox.zm[fmol] - gDBox.zm[smol];
    
    if(dmx > rcut){  //
        dmx = -rcut * 2.0 + dmx;
    }
    if(dmy > rcut ){
        dmy = -rcut * 2.0 + dmy;
    }
    if(dmz > rcut){
        dmz = -rcut * 2.0 + dmz;
    }
    if(dmx < -rcut){ //
        dmx = rcut * 2.0 + dmx;
    }
    if(dmy < -rcut){
        dmy = rcut * 2.0 + dmy;
    }
    if(dmz < -rcut){
        dmz = rcut * 2.0 + dmz;
    }
    r = dmx * dmx + dmy * dmy + dmz * dmz;
    //printf("i %d  j %d r %f rcut %f\n", fmol, smol, sqrt(r), rcut);
    if(r < rcut*rcut){ //if distance < rcut
        // for all atoms
        for(int i = 0; i < gTop.aNum[gDBox.mType[fmol]] ; i++){
            for(int j = 0; j < gTop.aNum[gDBox.mType[smol]]; j++){
                //id = 1; // gBox.aType[gBox.fAtom[b] + j] * gConf.potNum[0] + gBox.aType[gBox.fAtom[a] + i];
                id = gTop.aType[i] * gConf.potNum[0] + gTop.aType[j];
                dax = gDBox.xa[gDBox.fAtomOfMol[fmol]+i] - gDBox.xa[gDBox.fAtomOfMol[smol] + j] + dmx;
                day = gDBox.ya[gDBox.fAtomOfMol[fmol]+i] - gDBox.ya[gDBox.fAtomOfMol[smol] + j] + dmy;
                daz = gDBox.za[gDBox.fAtomOfMol[fmol]+i] - gDBox.za[gDBox.fAtomOfMol[smol] + j] + dmz;
                
                ra = dax * dax + day * day + daz * daz;
                //printf("x %f %f y %f %f z %f %f r %f ra %f \n", dmx, dax, dmy, day, dmz, daz, r, ra);
                //printf("mola %f mola %f\n", gDBox.za[gDBox.fAtomOfMol[fmol]+i], gDBox.za[gDBox.fAtomOfMol[smol] + j]);
                //printf("%d %d rm %f ra %f en %f vir %f\n",fmol, smol, sqrt(r), sqrt(ra), gTop.epsi[id] * (ra* ra - ra), gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra));
                ra = gTop.sigma[id] * gTop.sigma[id] / ra;
                ra = ra * ra * ra;  //6 power
                //calculate potential
                if(gDBox.phaseType[fmol] == LIQ){
                    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (ra* ra - ra);
                    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra);
                }
                else{
                    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (ra* ra - ra);
                    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra);
                }
                
            }
        }
        //printf("block %d thread %d en %f vir %f\n", blockIdx.x, threadIdx.x, gDBox.tempEn[blockIdx.x * MAXDIM + threadIdx.x], gDBox.tempEn[blockIdx.x * MAXDIM + threadIdx.x]);
    }
    
    return 0;
}

__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int mol){
    
    return 1;
}

__device__ int double_transition(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, int curMol, hiprandState devStates){
    //curMol -- gpu indexed molecule
    float oldEn;
    float newEn;
    float oldVir;
    float newVir;
    float oldX, oldY, oldZ; //old coordinates
    int curMol2;    //second molecule
    int reduce;
    
    //save old state------------------------------------------------------------
    if(threadIdx.x == 0){
        oldX = gDBox.xm[curMol];
        oldY = gDBox.ym[curMol];
        oldZ = gDBox.zm[curMol];
    }
    //zeros energy
    for(int i = 0; i < yDim; i++){
        gDBox.tempLiqEn[threadIdx.x * yDim + i] = 0.0;
        gDBox.tempVapEn[threadIdx.x * yDim + i] = 0.0;
        gDBox.tempLiqVir[threadIdx.x * yDim + i] = 0.0;
        gDBox.tempVapVir[threadIdx.x * yDim + i] = 0.0;
    }
    
    //calculate old energy------------------------------------------------------
    for(int i = 0; i < yDim; i++){
        curMol2 = threadIdx.x * yDim + i + gDBox.fMolOnPlate[blockIdx.x];
        if(gDBox.phaseType[curMol2] == gDBox.phaseType[curMol]){
            if(gDBox.phaseType[curMol2] == LIQ){
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
            }
            else{
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.vapRcut[blockIdx.x]);
            }
        }
    }
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            oldEn = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
        }
        else{
            oldEn = gDBox.tempVapEn[blockIdx.x * MAXDIM];
        }
    }
    //generate new state--------------------------------------------------------
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            gDBox.xm[curMol] += (1.0 + hiprand_uniform(&devStates)) * gDBox.maxLiqTrans[blockIdx.x];
            gDBox.ym[curMol] += (1.0 + hiprand_uniform(&devStates)) * gDBox.maxLiqTrans[blockIdx.x];
            gDBox.zm[curMol] += (1.0 + hiprand_uniform(&devStates)) * gDBox.maxLiqTrans[blockIdx.x];
        }
        else{
            gDBox.xm[curMol] += (1.0 + hiprand_uniform(&devStates)) * gDBox.maxVapTrans[blockIdx.x];
            gDBox.ym[curMol] += (1.0 + hiprand_uniform(&devStates)) * gDBox.maxVapTrans[blockIdx.x];
            gDBox.zm[curMol] += (1.0 + hiprand_uniform(&devStates)) * gDBox.maxVapTrans[blockIdx.x];
        }
    }
    __syncthreads();
    //calculate new energy------------------------------------------------------
    //zeros energy
    for(int i = 0; i < yDim; i++){
        gDBox.tempLiqEn[threadIdx.x * yDim + i] = 0.0;
        gDBox.tempVapEn[threadIdx.x * yDim + i] = 0.0;
        gDBox.tempLiqVir[threadIdx.x * yDim + i] = 0.0;
        gDBox.tempVapVir[threadIdx.x * yDim + i] = 0.0;
    }
    for(int i = 0; i < yDim; i++){
        curMol2 = threadIdx.x * yDim + i + gDBox.fMolOnPlate[blockIdx.x];
        if(gDBox.phaseType[curMol2] == gDBox.phaseType[curMol]){
            if(gDBox.phaseType[curMol2] == LIQ){
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
            }
            else{
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.vapRcut[blockIdx.x]);
            }
        }
    }
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            newEn = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
        }
        else{
            newEn = gDBox.tempVapEn[blockIdx.x * MAXDIM];
        }
    }
    __syncthreads;
    
    //check aceptance
    if(threadIdx.x == 0){
        if(hiprand_uniform(&devStates) < exp(-(newEn - oldEn)/gDBox.temp[blockIdx.x])){  //accept
            if(gDBox.phaseType[curMol] == LIQ){
                gDBox.accLiqTrans[blockIdx.x]++;
                gDBox.liqEn[blockIdx.x] += newEn - oldEn;
            }
            else{
                gDBox.accVapTrans[blockIdx.x]++;
                gDBox.vapEn[blockIdx.x] += newEn -oldEn;
            }
        }
        else{    //reject
            if(gDBox.phaseType[curMol] == LIQ){
                gDBox.rejLiqTrans[blockIdx.x]++;
            }
            else{
                gDBox.rejVapTrans[blockIdx.x]++;
            }
            //coordinates back
            gDBox.xm[curMol] = oldX;
            gDBox.ym[curMol] = oldY;
            gDBox.zm[curMol] = oldZ;
        }
    }
    __syncthreads;
    //calculate properties
    return 0;
}

__device__ int double_vol_change(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState devStates){
    //save state
    
    //calculate old enegry
    
    //change volumes
    
    //calculate new energy
    
    //check aceptance
    
    
    return 0;
}
