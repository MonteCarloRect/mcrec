#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"

__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop);
__device__ int double_totalen(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir);
__device__ int double_mol_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir);

int double_equilibration(gDoublebox gDBox, hDoubleBox doubleBox, gOptions gConf, gMolecula gTop){
    hipError_t cuErr;
    int* xDim;
    int temp;
    
    xDim = (int*) malloc(deviceCount * sizeof(int));
    for(int curDev = 0; curDev < deviceCount; curDev++){
        //set numbers of block equal to numbers of plate per device
        //numbers of thread equal to maximum numbers of molecules 
        temp = 0;
        for(int i = 0; i < doubleBox.devicePlates[i]; i++){    //get maximum number of molecules
            if(doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]] > temp){
                temp = doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];
            }
        }
        if(temp == 0){
            xDim[curDev] = 1;
        }
        else{
            if(log2(temp) > 7){
                xDim[curDev] = 512;
            }
            else{
                xDim[curDev] = pow(2,ceil(log2(temp)));
            }
        }
    }
    hipDeviceSynchronize();    //sync
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //calculate equlibration
        double_equilib_cycle<<<doubleBox.devicePlates[curDev], xDim[curDev]>>>(gDBox, gConf, gTop);
    }
    hipDeviceSynchronize();    //sync after complit all equlibrations
    return 0;
}

__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop){
    int temp;
    __shared__ int yDim;   //y Dimension of array
    float* tempTotEn;   //temp array for calculate total enegry
    float* tempMolEn;   //temp array for calculate molcule energy
    float* tempTotVir;  //temp array for calculate total virial
    float* tempMolVir;  //temp array for calculate molecule virial
    if(threadIdx.x == 0){
        tempTotEn = (float*) malloc(MAXDIM * sizeof(float)); //enegry of yDim molecules
        tempMolEn = (float*) malloc(MAXDIM * sizeof(float));
    }
    __syncthreads();
    
    //calculate total energy of phases
    if(threadIdx.x == 0){   //shared for whjole block
        yDim = ceilf(gDBox.molNum[blockIdx.x] / blockDim.x);
    }
    
    
    double_totalen(gDBox, gConf, gTop, yDim, tempTotEn, tempMolEn, tempTotVir, tempMolVir);
    if(gDBox.molNum[blockIdx.x] > 0){ //cycle only if plate not empty
        for(int i = 0; i < 1000; i++){  //loop for some (set as a parameter to option file)
            //get random number of molecule for block (plate)
            
            //get random transition
                //move molecule
                
                //change volume
                
                //molecule transition
                
        }
        //check equlibration status
    }
    else{
        //mark block as a compleet
    }
    //free arrays
    free(tempTotEn);
    free(tempMolEn);
}

__device__ int double_totalen(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir){
    int curMol; //current molecule
    //nuber of plate equal block number 
    
    for(int i = 0; i < yDim; i++){  //go 
        //get current molecule number
        curMol = gDBox.fMolOnPlate[blockIdx.x] + threadIdx.x * yDim + i;    //current molecule
        //calculate energy for current molecule
        if(curMol < gDBox.molNum[blockIdx.x]){
            double_mol_energy(gDBox, gConf, gTop, yDim, tempTotEn, tempMolEn, tempTotVir, tempMolVir);
        }
        else{   //epty slot - set energy to zero
            tempTotEn[curMol] = 0.0;
            tempTotVir[curMol] = 0.0;
        }
    }
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    
    
    return 0;
}

__device__ int double_mol_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir){
    
    return 0;
}




