#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"
//#include "double_trans.cu"

//__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int curDev);
//__device__  int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim);
//__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int fmol, int smol, float rcut);
//__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int mol);

//double_step 
__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int curDev);
__device__  int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim);
__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int fmol, int smol, float rcut);
__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int mol);

//double_trans
__device__ int double_transition(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState &devStates);

//double volume change
__device__ int double_vol_change(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState &devStates);

//double volume change
__device__ int double_liq_2_vap(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState &devStates);

//
__device__ int double_prop_calc(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim);

__device__ int double_prop_block_average(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, int curId);

__device__ int double_max_tran_change(gDoublebox &gDBox, gOptions gConf, gMolecula gTop);

__device__ int double_check_equilibration(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int curId);

///////

int double_equilibration(gDoublebox* gDBox, hDoubleBox doubleBox, gOptions* gConf, gMolecula* gTop){
    hipError_t cuErr;
    int* xDim;
    int temp;
    
    xDim = (int*) malloc(deviceCount * sizeof(int));
    for(int curDev = 0; curDev < deviceCount; curDev++){
        
        //set numbers of block equal to numbers of plate per device
        //numbers of thread equal to maximum numbers of molecules 
        temp = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //get maximum number of molecules in liquib phase
//            printf("liq %d plate %d molecules %d\n", curDev, doubleBox.platesPerDevice[curDev][i], doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]]);
            if(doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]] > temp){
                temp = doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]];
            }
        }
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //get maximum numbers of molecules in vapor phase
//            printf("vap %d plate %d molecules %d\n", curDev, doubleBox.platesPerDevice[curDev][i], doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]]);
            if(doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]] > temp){
                temp = doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]];
            }
        }
//        printf("device max %d\n",doubleBox.devicePlates[curDev]);
        if(temp == 0){
            xDim[curDev] = 1;
        }
        else{
            if(log2(temp) > 5){
                xDim[curDev] = MAXDIM; //set blocksize as maximum dimension
            }
            else{
                xDim[curDev] = pow(2,ceil(log2(temp))); //set block size 
            }
        }
    }
    hipDeviceSynchronize();    //sync
    printf("device count %d\n", deviceCount);
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //calculate equlibration
        printf("dev %d block %d thread %d\n", curDev, doubleBox.devicePlates[curDev], xDim[curDev]);
        double_equilib_cycle<<<doubleBox.devicePlates[curDev], xDim[curDev] >>>(gDBox[curDev], gConf[curDev], gTop[curDev], curDev);
        cuErr = hipGetLastError();
        if(cuErr != hipSuccess){
            printf("Error on device %d %s line %d, err: %s\n",curDev, __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
    }
    
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        hipStreamSynchronize(gDBox[curDev].stream);
        
    }
    hipDeviceSynchronize();    //sync after complit all equlibrations
    return 0;
}

__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int curDev){
    int temp;
    int yDim;   //y Dimension of array
    //float randTrans;    //random transition
    int randMol;    //random molecule
    hiprandState devStates;
    int curBlock;
    
    
    
    hiprand_init(1234, threadIdx.x+blockDim.x*blockIdx.x, 0, &devStates);
    //zeros all accept-reject
    gDBox.accLiqTrans[blockIdx.x] = 0;    //accept move in liquid phase
    gDBox.rejLiqTrans[blockIdx.x] = 0;
    gDBox.accVapTrans[blockIdx.x] = 0;   //accept move in vapor phase
    gDBox.rejVapTrans[blockIdx.x] = 0;
    gDBox.accVolChange[blockIdx.x] = 0;   //accept volume change
    gDBox.rejVolChange[blockIdx.x] = 0;
    gDBox.accLiq2Vap[blockIdx.x] = 0;    //accept liquid to vapor phase
    gDBox.rejLiq2Vap[blockIdx.x] = 0;
    gDBox.accVap2Liq[blockIdx.x] = 0;
    gDBox.rejVap2Liq[blockIdx.x] = 0;
    
    //check
    yDim = (gDBox.molNum[blockIdx.x] / blockDim.x) + 1; //set y dimension
    if(gDBox.molNum[blockIdx.x] > 0){
        if(threadIdx.x == 0){
            printf("molecules on device %d mol %d bdim %d ydim %d \n", curDev, gDBox.molNum[blockIdx.x], blockDim.x , yDim);
            curBlock = 0;
        }
        __syncthreads();  //sync yDim
//        if(threadIdx.x == 0){   //shared for whjole block
////            printf("device %d thread %d\n", curDev, threadIdx.x);
////            printf("sigma %f %f %f %f, cuurent device %d\n", gTop.sigma[0], gTop.sigma[1], gTop.sigma[2], gTop.sigma[3], curDev );
////            printf("ref en %f device %d nmol %d\n", gDBox.refEn[blockIdx.x], curDev, gDBox.molNum[blockIdx.x]);
////            printf("mol id %d x %f y %f z %f \n", gDBox.fMolOnPlate[blockIdx.x], gDBox.xm[4], gDBox.ym[4], gDBox.zm[4]);
//        }
    //    __syncthreads();
        double_totalen(gDBox, gConf, gTop, yDim);
        //printf("block %d molecules %d\n", blockIdx.x, gDBox.molNum[blockIdx.x]);
//        if(threadIdx.x == 0){
//            printf("energy 1 %f vir1 %f \n", gDBox.liqEn[blockIdx.x], gDBox.liqVir[blockIdx.x]);
//        }
//        double_totalen(gDBox, gConf, gTop, yDim);
        //printf("block %d molecules %d\n", blockIdx.x, gDBox.molNum[blockIdx.x]);
//        if(threadIdx.x == 0){
//            printf("energy 2 %f vir2 %f \n", gDBox.liqEn[blockIdx.x], gDBox.liqVir[blockIdx.x]);
//        }
        if(gDBox.molNum[blockIdx.x] > 0){ //cycle only if plate not empty
//            for(int i = 0; i < yDim; i++){
//                randMol = threadIdx.x + i + gDBox.fMolOnPlate[blockIdx.x];
//                if(randMol < gDBox.molNum[blockIdx.x]){
//                    printf("thread %d x %f y %f z %f\n", threadIdx.x, gDBox.xm[randMol], gDBox.ym[randMol], gDBox.zm[randMol]);
//                }
//                
//            }
            while(gDBox.eqStep[blockIdx.x] == 0){

            for(int nblk =0; nblk < EQBLOCKSIZE; nblk++){
            for(int step = 0; step < EQBLOCKCHECK; step++){  //loop for some (set as a parameter to option file)
                //randMol = hiprand_uniform(&devStates) * gDBox.molNum[blockIdx.x] + gDBox.fMolOnPlate[blockIdx.x];
                double_transition(gDBox, gConf, gTop, yDim, devStates);
                //double_totalen(gDBox, gConf, gTop, yDim);
                //if(threadIdx.x == 0){
                //    printf("energy 3 %f vir3 %f \n", gDBox.liqEn[blockIdx.x], gDBox.liqVir[blockIdx.x]);
                //}
                //print properties
//                if(step % 10000 == 0){
//                    if(threadIdx.x == 0){
//                        printf("step %d liquid energy %f vapor %f accept %d virial %f\n", step, gDBox.liqEn[blockIdx.x], gDBox.vapVir[blockIdx.x], gDBox.accLiqTrans[blockIdx.x], gDBox.liqVir[blockIdx.x]);
//                    }
//                    //check total energy
//                    double_totalen(gDBox, gConf, gTop, yDim);
//                    if(threadIdx.x == 0){
//                        printf("checked liquid en %f liqud vir %f\n", gDBox.liqEn[blockIdx.x], gDBox.liqVir[blockIdx.x]);
//                    }
//                }
                
                
            }   //end step loop
            if(nblk % 10 == 0){
                //printf("i %d second %d thread %d\n", nblk, blockIdx.x, threadIdx.x);
                double_vol_change(gDBox, gConf, gTop, yDim, devStates);
                double_liq_2_vap(gDBox, gConf, gTop, yDim, devStates);
            }
//            if(threadIdx.x == 0){
//                        printf("checked liquid en %f liqud vir %f\n", gDBox.liqEn[blockIdx.x], gDBox.liqVir[blockIdx.x]);
//                    }
            double_prop_calc(gDBox, gConf, gTop, yDim); //add to properties summ
            
            }   //end nblk loop
            double_prop_block_average(gDBox, gConf, gTop, yDim, curBlock);
            double_max_tran_change(gDBox, gConf, gTop);
            
            if(threadIdx.x == 0){
                curBlock++;
//                if(curBlock > 10){
//                    gDBox.eqStep[blockIdx.x] = curBlock;
//                }
            }
            __syncthreads();
            //check equlibration status
            double_check_equilibration(gDBox, gConf, gTop, curBlock);
            }   //end while 
        }
        else{
            //mark block as a compleet
        }
    }
    else{
        if(threadIdx.x == 0){
            printf("no molecules on block %d device %d\n", blockIdx.x, curDev);
        }
    }
}

__device__ int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim){
    int curMol; //current molecule
    int curMol2;    //second molecule
    int curId;  //current index
    __shared__ int reduce;
    //nuber of plate equal block number 
//        if(threadIdx.x == 0){   //shared for whjole block
//         printf("test 13.02-2\n");
//        }
    //printf("start totalen %d\n",threadIdx.x);
    
    //======calculate liquid phase energy/virial
    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;  //set energy to zero
    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0; //set virial to zero
    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;  //set energy to zero
    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0; //set virial to zero
    //printf("done %d yDim %d\n", threadIdx.x, yDim);
    __syncthreads;

    for(int i = 0; i < yDim; i++){
        curId = threadIdx.x * yDim + i; //current id of molecule in liqud list
        curMol = gDBox.fMolOnPlate[blockIdx.x] + gDBox.liqList[curId];  //get curent molecule GPU index
        if(gDBox.phaseType[curMol] == LIQ){  //calculate liquid energy
            //printf("cur mol %d mol liq %d \n", curId, gDBox.nLiq[blockIdx.x]);
            if(curId < gDBox.nLiq[blockIdx.x]){
                
                //set here intermolecular potential
                //printf("mol1 %d mol2 %d\n", curMol, curId);
                //calculate inermolecullar interaction
                for(int j = curId + 1; j < gDBox.nLiq[blockIdx.x]; j++){
                    curMol2 = gDBox.fMolOnPlate[blockIdx.x] + gDBox.liqList[j];
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
                }
            }
            else{
                //set energy to zero 
                gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += 0.0;
                gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += 0.0;
            }
        }
        else{   //calculate vapor energy
            if(curId < gDBox.nVap[blockIdx.x]){
                
                //set here intermolecular potential
                //printf("mol1 %d mol2 %d\n", curMol, curId);
                //calculate inermolecullar interaction
                for(int j = curId + 1; j < gDBox.nVap[blockIdx.x]; j++){
                    curMol2 = gDBox.fMolOnPlate[blockIdx.x] + gDBox.vapList[j];
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.vapRcut[blockIdx.x]);
                }
            }
        }
    }
    //printf("%f ", gDBox.tempEn[blockIdx.x * MAXDIM + threadIdx.x]);
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    __syncthreads();
    if(threadIdx.x == 0){
//        printf("device block %d liq molecules %d\n", blockIdx.x, gDBox.nLiq[blockIdx.x]);
//        printf("plate %d total liquid energy %f vapor energy %f\n", blockIdx.x, gDBox.tempLiqEn[blockIdx.x * MAXDIM], gDBox.tempVapEn[blockIdx.x * MAXDIM]);
        gDBox.liqEn[blockIdx.x] = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
        gDBox.vapEn[blockIdx.x] = gDBox.tempVapEn[blockIdx.x * MAXDIM];
        gDBox.liqVir[blockIdx.x] = gDBox.tempLiqVir[blockIdx.x * MAXDIM];
        gDBox.vapVir[blockIdx.x] = gDBox.tempVapVir[blockIdx.x * MAXDIM];
    }
    __syncthreads();
    return 0;
}

__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int fmol, int smol, float rcut){
    //fmol smol - indexes of firs and second molecules
    //get atom numbers
    float dmx;  //diffs in molecule coordinats
    float dmy;
    float dmz;
    float dax;  //diff in atoms coordinate
    float day;
    float daz;
    float r;    //distance ^2 of molecules
    float ra;   //distance ^2 of atoms
    int id;     //array indexes for potential parameters
    
    dmx = gDBox.xm[fmol] - gDBox.xm[smol];
    dmy = gDBox.ym[fmol] - gDBox.ym[smol];
    dmz = gDBox.zm[fmol] - gDBox.zm[smol];
    
    if(dmx > rcut){  //
        dmx = -rcut * 2.0 + dmx;
    }
    if(dmy > rcut ){
        dmy = -rcut * 2.0 + dmy;
    }
    if(dmz > rcut){
        dmz = -rcut * 2.0 + dmz;
    }
    if(dmx < -rcut){ //
        dmx = rcut * 2.0 + dmx;
    }
    if(dmy < -rcut){
        dmy = rcut * 2.0 + dmy;
    }
    if(dmz < -rcut){
        dmz = rcut * 2.0 + dmz;
    }
    r = dmx * dmx + dmy * dmy + dmz * dmz;
    //printf("i %d  j %d r %f rcut %f\n", fmol, smol, sqrt(r), rcut);
    if(r < rcut*rcut){ //if distance < rcut
        // for all atoms
        for(int i = 0; i < gTop.aNum[gDBox.mType[fmol]] ; i++){
            for(int j = 0; j < gTop.aNum[gDBox.mType[smol]]; j++){
                //id = 1; // gBox.aType[gBox.fAtom[b] + j] * gConf.potNum[0] + gBox.aType[gBox.fAtom[a] + i];
                id = gTop.aType[i] * gConf.potNum[0] + gTop.aType[j];
                dax = gDBox.xa[gDBox.fAtomOfMol[fmol] + i] - gDBox.xa[gDBox.fAtomOfMol[smol] + j] + dmx;
                day = gDBox.ya[gDBox.fAtomOfMol[fmol] + i] - gDBox.ya[gDBox.fAtomOfMol[smol] + j] + dmy;
                daz = gDBox.za[gDBox.fAtomOfMol[fmol] + i] - gDBox.za[gDBox.fAtomOfMol[smol] + j] + dmz;
                
                ra = dax * dax + day * day + daz * daz;
                //printf("x %f %f y %f %f z %f %f r %f ra %f \n", dmx, dax, dmy, day, dmz, daz, r, ra);
                //printf("mola %f mola %f\n", gDBox.za[gDBox.fAtomOfMol[fmol]+i], gDBox.za[gDBox.fAtomOfMol[smol] + j]);
                //printf("%d %d rm %f ra %f en %f vir %f\n",fmol, smol, sqrt(r), sqrt(ra), gTop.epsi[id] * (ra* ra - ra), gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra));
                ra = gTop.sigma[id] * gTop.sigma[id] / ra;
                ra = ra * ra * ra;  //6 power
                //calculate potential
                if(gDBox.phaseType[fmol] == LIQ){
                    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (ra* ra - ra);
                    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra);
                }
                else{
                    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (ra* ra - ra);
                    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra);
                }
                
            }
        }
        //printf("block %d thread %d en %f vir %f\n", blockIdx.x, threadIdx.x, gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x], gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x]);
    }
    
    return 0;
}

__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, int mol){
    
    return 1;
}

__device__ int double_transition(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState &devStates){
    //curMol -- gpu indexed molecule
    __shared__ float oldEn;
    __shared__ float newEn;
    __shared__ float oldVir;
    __shared__ float newVir;
    __shared__ float3 sav; //old coordinates
    __shared__ int curMol;
    int curMol2;    //second molecule
    __shared__ int reduce;
    int id1;
    int id2;
    

    //save old state------------------------------------------------------------
    if(threadIdx.x == 0){
        curMol = hiprand_uniform(&devStates) * gDBox.molNum[blockIdx.x] + gDBox.fMolOnPlate[blockIdx.x];
        sav.x = gDBox.xm[curMol];
        sav.y = gDBox.ym[curMol];
        sav.z = gDBox.zm[curMol];
        oldEn = 0.0;
        newEn = 0.0;
        oldVir = 0.0;
        newVir = 0.0;
    }
//    //zeros energy
//    if(threadIdx.x == 0){
//        printf("part1\n");
//    }
    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    __syncthreads;
    //printf("thread %d curmol %d\n", threadIdx.x, curMol);
    //calculate old energy------------------------------------------------------
    for(int i = 0; i < yDim; i++){
        curMol2 = threadIdx.x * yDim + i + gDBox.fMolOnPlate[blockIdx.x];
        if((gDBox.phaseType[curMol2] == gDBox.phaseType[curMol]) && (curMol2 != curMol)){
            if(gDBox.phaseType[curMol2] == LIQ){
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
            }
            else{
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.vapRcut[blockIdx.x]);
            }
        }
    }
    //printf("thread %d curmol %d fmop %d en %f\n", threadIdx.x, curMol,gDBox.fMolOnPlate[blockIdx.x],gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x]);
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    //printf("thread %d energy %f\n", threadIdx.x, gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x]);
    reduce = blockDim.x / 2;
    __syncthreads;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            oldEn = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
            oldVir = gDBox.tempLiqVir[blockIdx.x * MAXDIM];
        }
        else{
            oldEn = gDBox.tempVapEn[blockIdx.x * MAXDIM];
            oldVir = gDBox.tempVapVir[blockIdx.x * MAXDIM];
        }
    }
    //generate new state--------------------------------------------------------
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            gDBox.xm[curMol] += (1.0 - hiprand_uniform(&devStates)) * gDBox.maxLiqTrans[blockIdx.x];
            gDBox.ym[curMol] += (1.0 - hiprand_uniform(&devStates)) * gDBox.maxLiqTrans[blockIdx.x];
            gDBox.zm[curMol] += (1.0 - hiprand_uniform(&devStates)) * gDBox.maxLiqTrans[blockIdx.x];
            //check out of boxes
            if(gDBox.xm[curMol] > 2.0 * gDBox.liqRcut[blockIdx.x]){
                gDBox.xm[curMol] -= 2.0 * gDBox.liqRcut[blockIdx.x];
            }
            if(gDBox.ym[curMol] > 2.0 * gDBox.liqRcut[blockIdx.x]){
                gDBox.ym[curMol] -= 2.0 * gDBox.liqRcut[blockIdx.x];
            }
            if(gDBox.zm[curMol] > 2.0 * gDBox.liqRcut[blockIdx.x]){
                gDBox.zm[curMol] -= 2.0 * gDBox.liqRcut[blockIdx.x];
            }
            if(gDBox.xm[curMol] < 0.0){
                gDBox.xm[curMol] += 2.0 * gDBox.liqRcut[blockIdx.x];
            }
            if(gDBox.ym[curMol] < 0.0){
                gDBox.ym[curMol] += 2.0 * gDBox.liqRcut[blockIdx.x];
            }
            if(gDBox.zm[curMol] < 0.0){
                gDBox.zm[curMol] += 2.0 * gDBox.liqRcut[blockIdx.x];
            }
        }
        else{
            gDBox.xm[curMol] += (1.0 - hiprand_uniform(&devStates)) * gDBox.maxVapTrans[blockIdx.x];
            gDBox.ym[curMol] += (1.0 - hiprand_uniform(&devStates)) * gDBox.maxVapTrans[blockIdx.x];
            gDBox.zm[curMol] += (1.0 - hiprand_uniform(&devStates)) * gDBox.maxVapTrans[blockIdx.x];
            //check out of boxes
            if(gDBox.xm[curMol] > 2.0 * gDBox.vapRcut[blockIdx.x]){
                gDBox.xm[curMol] -= 2.0 * gDBox.vapRcut[blockIdx.x];
            }
            if(gDBox.ym[curMol] > 2.0 * gDBox.vapRcut[blockIdx.x]){
                gDBox.ym[curMol] -= 2.0 * gDBox.vapRcut[blockIdx.x];
            }
            if(gDBox.zm[curMol] > 2.0 * gDBox.vapRcut[blockIdx.x]){
                gDBox.zm[curMol] -= 2.0 * gDBox.vapRcut[blockIdx.x];
            }
            if(gDBox.xm[curMol] < 0.0){
                gDBox.xm[curMol] += 2.0 * gDBox.vapRcut[blockIdx.x];
            }
            if(gDBox.ym[curMol] < 0.0){
                gDBox.ym[curMol] += 2.0 * gDBox.vapRcut[blockIdx.x];
            }
            if(gDBox.zm[curMol] < 0.0){
                gDBox.zm[curMol] += 2.0 * gDBox.vapRcut[blockIdx.x];
            }
        }
        
    }
    __syncthreads();
    //calculate new energy------------------------------------------------------
    //zeros energy
//    if(threadIdx.x == 0){
//        printf("part2\n");
//    }
    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    __syncthreads;
    
    //calculate new energy------------------------------------------------------
    for(int i = 0; i < yDim; i++){
        curMol2 = threadIdx.x * yDim + i + gDBox.fMolOnPlate[blockIdx.x];
        if((gDBox.phaseType[curMol2] == gDBox.phaseType[curMol]) && (curMol2 != curMol)){
            if(gDBox.phaseType[curMol2] == LIQ){
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
            }
            else{
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.vapRcut[blockIdx.x]);
            }
        }
    }
    //printf("thread %d curmol %d fmop %d en %f\n", threadIdx.x, curMol,gDBox.fMolOnPlate[blockIdx.x],gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x]);
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    //printf("thread %d energy %f\n", threadIdx.x, gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x]);
    reduce = blockDim.x / 2;
    __syncthreads;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            newEn = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
            newVir = gDBox.tempLiqVir[blockIdx.x * MAXDIM];
        }
        else{
            newEn = gDBox.tempVapEn[blockIdx.x * MAXDIM];
            newVir = gDBox.tempVapVir[blockIdx.x * MAXDIM];
        }
    }
    __syncthreads;
    
    //check aceptance
    if(threadIdx.x == 0){
        
        if(hiprand_uniform(&devStates) < exp(-(newEn - oldEn)/gDBox.temp[blockIdx.x])){  //accept
            //printf("delta en %f %f curMol %d exp %f del en %f\n", oldEn, newEn, curMol, exp(-(newEn - oldEn)/gDBox.temp[blockIdx.x]), newEn - oldEn);
            if(gDBox.phaseType[curMol] == LIQ){
                gDBox.accLiqTrans[blockIdx.x]++;
                gDBox.liqEn[blockIdx.x] += newEn - oldEn;
                gDBox.liqVir[blockIdx.x] += newVir - oldVir;
            }
            else{
                gDBox.accVapTrans[blockIdx.x]++;
                gDBox.vapEn[blockIdx.x] += newEn - oldEn;
                gDBox.vapVir[blockIdx.x] += newVir - oldVir;
            }
            //rebuild lists

        }
        else{    //reject
            if(gDBox.phaseType[curMol] == LIQ){
                gDBox.rejLiqTrans[blockIdx.x]++;
                gDBox.phaseType[curMol] == VAP;
            }
            else{
                gDBox.rejVapTrans[blockIdx.x]++;
                gDBox.phaseType[blockIdx.x] = LIQ;
                
            }
            //coordinates back
            gDBox.xm[curMol] = sav.x;
            gDBox.ym[curMol] = sav.y;
            gDBox.zm[curMol] = sav.z;
            
        }
    }
    __syncthreads;
    //
    return 0;
}

__device__ int double_vol_change(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState &devStates){
    float oldLiqEn;
    float oldVapEn;
    float newLiqEn;
    float newVapEn;
    float oldLiqVir;
    float oldVapVir;
    int curMol;
    int curId;
    
    float savLiqRcut;
    float savVapRcut;
    float savLiqVol;
    float savVapVol;
    
    float dv;
    __shared__ float coef1;
    __shared__ float coef2;
    
    
    //save state
    for(int i = 0; i < yDim; i++){
        curId = threadIdx.x * yDim + i;
        curMol = threadIdx.x * yDim + i + gDBox.fMolOnPlate[blockIdx.x];
        if(curMol < gDBox.molNum[blockIdx.x]){
            gDBox.tempXm[curId] = gDBox.xm[curMol];
            gDBox.tempYm[curId] = gDBox.ym[curMol];
            gDBox.tempZm[curId] = gDBox.zm[curMol];
        }
    }
    if(threadIdx.x == 0){
        savLiqRcut = gDBox.liqRcut[blockIdx.x];
        savVapRcut = gDBox.vapRcut[blockIdx.x];
        savLiqVol = gDBox.liqVol[blockIdx.x];
        savVapVol = gDBox.vapVol[blockIdx.x];
    }
    //printf("thread %d rcut %f %f vol %f %f\n", threadIdx.x, savLiqRcut, savVapRcut, savLiqVol, savVapVol);
    __syncthreads;
    //calculate old enegry  //current energy is old energy
    //double_totalen(gDBox, gConf, gTop, yDim);
    //__syncthreads;
    if(threadIdx.x == 0){
        oldLiqEn = gDBox.liqEn[blockIdx.x];
        oldVapEn = gDBox.vapEn[blockIdx.x];
        oldLiqVir = gDBox.liqVir[blockIdx.x];
        oldVapVir = gDBox.vapVir[blockIdx.x];
    }
    //printf("thread %d old energy %f\n",threadIdx.x, oldLiqEn);
    //change volumes
    if(threadIdx.x == 0){
        if(gDBox.liqVol[blockIdx.x] > gDBox.vapVol[blockIdx.x]){
            dv = gDBox.vapVol[blockIdx.x] * gDBox.maxVolChange[blockIdx.x] * (hiprand_uniform(&devStates) - 0.5);
        }
        else{
            dv = gDBox.vapVol[blockIdx.x] * gDBox.maxVolChange[blockIdx.x] * (hiprand_uniform(&devStates) - 0.5);
        }
        //printf("delta vol %f\n", dv);
        coef1 = pow((gDBox.liqVol[blockIdx.x] + dv) / gDBox.liqVol[blockIdx.x], 1.0/3.0);
        coef2 = pow((gDBox.vapVol[blockIdx.x] - dv) / gDBox.vapVol[blockIdx.x], 1.0/3.0);
    }
    __syncthreads;
//    //printf("thread %d coef1 %f, coef2 %f\n", threadIdx.x, coef1, coef2);
//    if(threadIdx.x == 0){   //VARIANT 1=========================================
//        for(int i = 0; i < gDBox.molNum[blockIdx.x]; i++){
//            curMol = i + gDBox.fMolOnPlate[blockIdx.x];
//            if(gDBox.phaseType[curMol] == LIQ){
//                gDBox.xm[curMol] *= coef1;
//                gDBox.ym[curMol] *= coef1;
//                gDBox.zm[curMol] *= coef1;
//            }
//            else{
//                gDBox.xm[curMol] *= coef2;
//                gDBox.ym[curMol] *= coef2;
//                gDBox.zm[curMol] *= coef2;
//            }
//            //printf("thread %d x %f y %f z %f\n", threadIdx.x, gDBox.xm[curMol], gDBox.ym[curMol], gDBox.zm[curMol]);
//        }
//    }
//    __syncthreads;
//    //VARIANT 2 ===============================================================
    for(int i = 0; i < yDim; i++){
        curMol = threadIdx.x + i + gDBox.fMolOnPlate[blockIdx.x];
        if(gDBox.phaseType[curMol] == LIQ){
            gDBox.xm[curMol] *= coef1;
            gDBox.ym[curMol] *= coef1;
            gDBox.zm[curMol] *= coef1;
        }
        else{
            gDBox.xm[curMol] *= coef2;
            gDBox.ym[curMol] *= coef2;
            gDBox.zm[curMol] *= coef2;
        }
//        printf("thread vol %d curMol %d x %f y %f z %f c1 %f c2 %f\n", threadIdx.x, curMol, gDBox.xm[curMol], gDBox.ym[curMol], gDBox.zm[curMol], coef1, coef2);
    }
    __syncthreads;
    //change volume and rcut
    if(threadIdx.x == 0){
        //printf("thread %d x %f y %f z %f\n", threadIdx.x, gDBox.xm[1], gDBox.ym[1], gDBox.zm[1]);
        gDBox.liqVol[blockIdx.x] += dv;
        gDBox.vapVol[blockIdx.x] -= dv;
        gDBox.liqRcut[blockIdx.x] = pow(gDBox.liqVol[blockIdx.x],1.0/3.0);
        gDBox.vapRcut[blockIdx.x] = pow(gDBox.vapVol[blockIdx.x],1.0/3.0);
        //printf("lv %f vv %f lr %f vr %f\n", gDBox.liqVol[blockIdx.x], gDBox.vapVol[blockIdx.x], gDBox.liqRcut[blockIdx.x], gDBox.vapRcut[blockIdx.x] );
    }
    __syncthreads;
    //calculate new energy
    double_totalen(gDBox, gConf, gTop, yDim);
    __syncthreads;
    if(threadIdx.x == 0){
        newLiqEn = gDBox.liqEn[blockIdx.x];
        newVapEn = gDBox.vapEn[blockIdx.x];
    }
    __syncthreads;
//    if(threadIdx.x == 100){
//        printf("volchange ver %f %f %f exp %f \n", dv, newLiqEn, oldLiqEn, exp(-(newLiqEn + newVapEn - oldLiqEn - oldVapEn)/gDBox.temp[blockIdx.x]) );
//    }
//    if(threadIdx.x == 0){
//        printf("Vol change dV %f nE %f oE %f exp %f \n", dv, newLiqEn, oldLiqEn, exp(-(newLiqEn + newVapEn - oldLiqEn - oldVapEn)/gDBox.temp[blockIdx.x] - gDBox.nLiq[blockIdx.x] * log(coef1 * coef1 * coef1) - gDBox.nVap[blockIdx.x] * log(coef2 * coef2 * coef2) ) );
//    }
    //check aceptance
    if(threadIdx.x == 0){
        if(hiprand_uniform(&devStates) < exp(-(newLiqEn + newVapEn - oldLiqEn - oldVapEn)/gDBox.temp[blockIdx.x] - gDBox.nLiq[blockIdx.x] * log(coef1 * coef1 * coef1) - gDBox.nVap[blockIdx.x] * log(coef2 * coef2 * coef2) ) ){  //accept
            //add to accept
            //printf("volume change accept\n");
            gDBox.accVolChange[blockIdx.x]++;
        }
        else{    //reject
            //coordinates back
            //printf("volume change reject s1 %f s2 %f\n", newLiqEn, oldVapEn);
            for(int i = 0; i < gDBox.molNum[blockIdx.x]; i++){
                //printf("i %d txm %f ym %f zm %f\n",i, gDBox.tempXm[i], gDBox.tempYm[i], gDBox.tempZm[i]);
                curMol = i + gDBox.fMolOnPlate[blockIdx.x];
                gDBox.xm[curMol] = gDBox.tempXm[i];
                gDBox.ym[curMol] = gDBox.tempYm[i];
                gDBox.zm[curMol] = gDBox.tempZm[i];
                //printf("i %d txm %f ym %f zm %f\n",i, gDBox.xm[curMol], gDBox.ym[curMol], gDBox.zm[curMol]);
            }
            gDBox.liqRcut[blockIdx.x] = savLiqRcut;
            gDBox.vapRcut[blockIdx.x] = savVapRcut;
            gDBox.liqVol[blockIdx.x] = savLiqVol;
            gDBox.vapVol[blockIdx.x] = savVapVol;
            gDBox.liqEn[blockIdx.x] = oldLiqEn;
            gDBox.vapEn[blockIdx.x] = oldVapEn;
            gDBox.liqVir[blockIdx.x] = oldLiqVir;
            gDBox.vapVir[blockIdx.x] = oldVapVir;
            gDBox.rejVolChange[blockIdx.x]++;
        }
    }
    __syncthreads;
    return 0;
}

__device__ int double_liq_2_vap(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, hiprandState &devStates){
    float newEn;
    float oldEn;
    float oldVir;
    float newVir;
    int curMol2;
    
    float oldx;
    float oldy;
    float oldz;
    __shared__ int curId;
    __shared__ int curMol;
    int reduce;
    int id1;
    int id2;
    
//    if(threadIdx.x < 50){
//        printf("test === thread %d\n", threadIdx.x);
//    }
    if(threadIdx.x == 0){
    //random molecule
        curId = hiprand_uniform(&devStates) * gDBox.molNum[blockIdx.x];
        curMol = curId + gDBox.fMolOnPlate[blockIdx.x];
        //save old state
        oldx = gDBox.xm[curMol];
        oldy = gDBox.ym[curMol];
        oldz = gDBox.zm[curMol];
        //printf("1 - old %f %f %f new %f %f %f\n", oldx, oldy, oldz, gDBox.xm[curMol], gDBox.ym[curMol], gDBox.zm[curMol]);
        //printf("==============thread %d curId %d curMol %d\n", threadIdx.x, curId, curMol);
    }
    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    __syncthreads;
//    
    //calculate old energy------------------------------------------------------
    for(int i = 0; i < yDim; i++){
        curMol2 = threadIdx.x * yDim + i + gDBox.fMolOnPlate[blockIdx.x];
        if(curMol2 < gDBox.molNum[blockIdx.x]){
            if((gDBox.phaseType[curMol2] == gDBox.phaseType[curMol]) && (curMol != curMol2)){
                if(gDBox.phaseType[curMol2] == LIQ){
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
                }
                else{
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.vapRcut[blockIdx.x]);
                }
            }
        }
    }
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            oldEn = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
            oldVir = gDBox.tempLiqVir[blockIdx.x * MAXDIM];
        }
        else{
            oldEn = gDBox.tempVapEn[blockIdx.x * MAXDIM];
            oldVir = gDBox.tempVapVir[blockIdx.x * MAXDIM];
        }
    }
    //random new position 
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            gDBox.xm[curMol] = hiprand_uniform(&devStates) * gDBox.vapRcut[blockIdx.x] * 2.0;
            gDBox.ym[curMol] = hiprand_uniform(&devStates) * gDBox.vapRcut[blockIdx.x] * 2.0;
            gDBox.zm[curMol] = hiprand_uniform(&devStates) * gDBox.vapRcut[blockIdx.x] * 2.0;
            gDBox.phaseType[curMol] = VAP;
            gDBox.nLiq[blockIdx.x]--;
            gDBox.nVap[blockIdx.x]++;
        }
        else{
            gDBox.xm[curMol] = hiprand_uniform(&devStates) * gDBox.liqRcut[blockIdx.x] * 2.0;
            gDBox.ym[curMol] = hiprand_uniform(&devStates) * gDBox.liqRcut[blockIdx.x] * 2.0;
            gDBox.zm[curMol] = hiprand_uniform(&devStates) * gDBox.liqRcut[blockIdx.x] * 2.0;
            gDBox.phaseType[curMol] = LIQ;
            gDBox.nLiq[blockIdx.x]++;
            gDBox.nVap[blockIdx.x]--;
        }
    }
    __syncthreads;
    //calculate new energy
    gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;
    //calculate new energy------------------------------------------------------
    for(int i = 0; i < yDim; i++){
        curMol2 = threadIdx.x * yDim + i + gDBox.fMolOnPlate[blockIdx.x];
        if(curMol2 < gDBox.molNum[blockIdx.x]){
            if((gDBox.phaseType[curMol2] == gDBox.phaseType[curMol]) && (curMol != curMol2)){
                if(gDBox.phaseType[curMol2] == LIQ){
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
                }
                else{
                    double_mol_pair_energy(gDBox, gConf, gTop, yDim, curMol, curMol2, gDBox.vapRcut[blockIdx.x]);
                }
            }
        }
    }
    
    
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempLiqVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x] += gDBox.tempVapVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    if(threadIdx.x == 0){
        if(gDBox.phaseType[curMol] == LIQ){
            newEn = gDBox.tempLiqEn[blockIdx.x * MAXDIM];
        }
        else{
            newEn = gDBox.tempVapEn[blockIdx.x * MAXDIM];
        }
    }
    //check acceptance
    if(threadIdx.x == 0){
        //printf("2 - old %f %f %f new %f %f %f\n", oldx, oldy, oldz, gDBox.xm[curMol], gDBox.ym[curMol], gDBox.zm[curMol]);
        //printf("trans curmol %d oldEn %f newEn %f oldVir %f newVir %f \n", curMol, oldEn, newEn, oldVir, newVir);
        if(gDBox.phaseType[curMol] == LIQ){ //swiched to liquid
            if(hiprand_uniform(&devStates) < exp(-(newEn - oldEn)/gDBox.temp[blockIdx.x] + log(gDBox.vapVol[blockIdx.x] * (gDBox.nLiq[blockIdx.x]+1) / gDBox.liqVol[blockIdx.x] / gDBox.nVap[blockIdx.x]) ) ){   //accept
                //printf("vap2liq molecule transition accept\n");
                gDBox.vapEn[blockIdx.x] -= oldEn;
                gDBox.vapVir[blockIdx.x] -= oldVir;
                gDBox.liqEn[blockIdx.x] += newEn;
                gDBox.liqVir[blockIdx.x] += newVir;
                gDBox.accVap2Liq[blockIdx.x]++;
                id1 = 0;
                id2 = 0;
                for(int i = 0; i < gDBox.molNum[blockIdx.x]; i++){
                    if(gDBox.phaseType[gDBox.fMolOnPlate[blockIdx.x] + i] == LIQ){
                        gDBox.liqList[gDBox.fMolOnPlate[blockIdx.x] + id1] = i;
                        id1++;
                    }
                    else{
                        gDBox.vapList[gDBox.fMolOnPlate[blockIdx.x] + id2] = i;
                        id2++;
                    }
                }
            }
            else{   //reject
                //printf("vap2liq molecule transition reject\n");
                gDBox.xm[curMol] = oldx;
                gDBox.ym[curMol] = oldy;
                gDBox.zm[curMol] = oldz;
                gDBox.phaseType[curMol] = VAP;
                gDBox.nLiq[blockIdx.x]--;
                gDBox.nVap[blockIdx.x]++;
                gDBox.rejVap2Liq[blockIdx.x]++;
            }
        }
        else{   //switched to vapor
            if(hiprand_uniform(&devStates) < exp(-(newEn - oldEn)/gDBox.temp[blockIdx.x] + log(gDBox.liqVol[blockIdx.x] * (gDBox.nVap[blockIdx.x]+1) / gDBox.vapVol[blockIdx.x] / gDBox.nLiq[blockIdx.x]) ) ){   //accept
                //printf("liq2vap molecule transition accept\n");
                gDBox.liqEn[blockIdx.x] += newEn;
                gDBox.liqVir[blockIdx.x] += newVir;
                gDBox.vapEn[blockIdx.x] -= oldEn;
                gDBox.vapVir[blockIdx.x] -= oldVir;
                gDBox.accLiq2Vap[blockIdx.x]++;
                id1 = 0;
                id2 = 0;
                for(int i = 0; i < gDBox.molNum[blockIdx.x]; i++){
                    if(gDBox.phaseType[gDBox.fMolOnPlate[blockIdx.x] + i] == LIQ){
                        gDBox.liqList[gDBox.fMolOnPlate[blockIdx.x] + id1] = i;
                        id1++;
                    }
                    else{
                        gDBox.vapList[gDBox.fMolOnPlate[blockIdx.x] + id2] = i;
                        id2++;
                    }
                }
            }
            else{   //reject
                //printf("liq2vap molecule transition reject\n");
                gDBox.xm[curMol] = oldx;
                gDBox.ym[curMol] = oldy;
                gDBox.zm[curMol] = oldz;
                gDBox.phaseType[curMol] = LIQ;
                gDBox.nLiq[blockIdx.x]++;
                gDBox.nVap[blockIdx.x]--;
                gDBox.rejLiq2Vap[blockIdx.x]++;
            }
        }
    }
    __syncthreads;
    //printf("thread %d curMol %d\n", threadIdx.x, reduce);
    return 0;
}

__device__ int double_prop_calc(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim){

    if(threadIdx.x == 0){
        //energy
        gDBox.sumLiqEn[blockIdx.x] += gDBox.liqEn[blockIdx.x];
        gDBox.sumVapEn[blockIdx.x] += gDBox.vapEn[blockIdx.x];
        //molecule energy
        gDBox.sumLiqMolEn[blockIdx.x] += gDBox.nLiq[blockIdx.x];
        gDBox.sumVapMolEn[blockIdx.x] += gDBox.nVap[blockIdx.x];
        //molecule numbers by type
        for(int i = 0; i < gDBox.nLiq[blockIdx.x]; i++){
            gDBox.sumLiqMol[blockIdx.x * gConf.subNum[0] + gDBox.mType[gDBox.liqList[gDBox.fMolOnPlate[blockIdx.x] + i]]] += 1.0;
            gDBox.sumLiqMassDens[blockIdx.x] += 1.0 / gDBox.liqVol[blockIdx.x]; //now number density swich to mass
        }
        for(int i = 0; i < gDBox.nVap[blockIdx.x]; i++){
            gDBox.sumVapMol[blockIdx.x * gConf.subNum[0] + gDBox.mType[gDBox.vapList[gDBox.fMolOnPlate[blockIdx.x] + i]]] += 1.0;
            gDBox.sumVapMassDens[blockIdx.x] += 1.0 / gDBox.vapVol[blockIdx.x];
        } 
        for(int i = 0; i < gConf.subNum[0]; i++){
            if(gDBox.nLiq[blockIdx.x] > 0){
                gDBox.sumLiqConc[blockIdx.x * gConf.subNum[0] + i] = gDBox.sumLiqMol[blockIdx.x + i] / gDBox.nLiq[blockIdx.x];
            }
            if(gDBox.nVap[blockIdx.x] > 0){
                gDBox.sumVapConc[blockIdx.x * gConf.subNum[0] + i] = gDBox.sumVapMol[blockIdx.x + i] / gDBox.nVap[blockIdx.x];
            }
            
        }
        //pressure p = nkT + 1/3 W 
        gDBox.sumLiqPress[blockIdx.x] += gDBox.temp[blockIdx.x] * gDBox.nLiq[blockIdx.x] / gDBox.liqVol[blockIdx.x] * 1.38064852*10.0 - gDBox.liqVir[blockIdx.x] / gDBox.liqVol[blockIdx.x] / 3.0f * 1.38064852*10.0;
        gDBox.sumVapPress[blockIdx.x] += gDBox.temp[blockIdx.x] * gDBox.nVap[blockIdx.x] / gDBox.vapVol[blockIdx.x] * 1.38064852*10.0 - gDBox.vapVir[blockIdx.x] / gDBox.vapVol[blockIdx.x] / 3.0f * 1.38064852*10.0;
        //
        
    }
    
    __syncthreads();
    return 0;
}

__device__ int double_prop_block_average(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, int curId){
    
    if(threadIdx.x == 0){
        if(curId > EQBLOCKS-1){
            //swith up
            for(int i = 0; i < EQBLOCKS-1; i++){
                gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + i] = gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + i + 1];
                gDBox.blockVapEn[blockIdx.x * EQBLOCKS + i] = gDBox.blockVapEn[blockIdx.x * EQBLOCKS + i + 1];
                gDBox.blockLiqMolEn[blockIdx.x * EQBLOCKS + i] = gDBox.blockLiqMolEn[blockIdx.x * EQBLOCKS + i + 1];
                gDBox.blockVapMolEn[blockIdx.x * EQBLOCKS + i] = gDBox.blockVapMolEn[blockIdx.x * EQBLOCKS + i + 1];
                gDBox.blockLiqPress[blockIdx.x * EQBLOCKS + i] = gDBox.blockLiqPress[blockIdx.x * EQBLOCKS + i + 1];
                gDBox.blockVapPress[blockIdx.x * EQBLOCKS + i] = gDBox.blockVapPress[blockIdx.x * EQBLOCKS + i + 1];
                gDBox.blockLiqMassDens[blockIdx.x * EQBLOCKS + i] = gDBox.blockLiqMassDens[blockIdx.x * EQBLOCKS + i + 1];
                gDBox.blockVapMassDens[blockIdx.x * EQBLOCKS + i] = gDBox.blockVapMassDens[blockIdx.x * EQBLOCKS + i + 1];
                for(int j = 0; j < gConf.subNum[0]; j++){
                    gDBox.blockLiqConc[blockIdx.x * EQBLOCKS + i * gConf.subNum[0] + j] = gDBox.blockLiqConc[blockIdx.x * EQBLOCKS + (i + 1) * gConf.subNum[0] + j];
                    gDBox.blockVapConc[blockIdx.x * EQBLOCKS + i * gConf.subNum[0] + j] = gDBox.blockVapConc[blockIdx.x * EQBLOCKS + (i + 1) * gConf.subNum[0] + j];
                }
                curId = EQBLOCKS - 1;
            }
        }
        //average
        gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + curId] = gDBox.sumLiqEn[blockIdx.x] / (float) EQBLOCKSIZE;
        gDBox.blockVapEn[blockIdx.x * EQBLOCKS + curId] = gDBox.sumVapEn[blockIdx.x] / (float) EQBLOCKSIZE;
        gDBox.blockLiqMolEn[blockIdx.x * EQBLOCKS + curId] = gDBox.sumLiqMolEn[blockIdx.x] / (float) EQBLOCKSIZE;
        gDBox.blockVapMolEn[blockIdx.x * EQBLOCKS + curId] = gDBox.sumVapMolEn[blockIdx.x] / (float) EQBLOCKSIZE;
        gDBox.blockLiqPress[blockIdx.x * EQBLOCKS + curId] = gDBox.sumLiqPress[blockIdx.x] / (float) EQBLOCKSIZE;
        gDBox.blockVapPress[blockIdx.x * EQBLOCKS + curId] = gDBox.sumVapPress[blockIdx.x] / (float) EQBLOCKSIZE;
        gDBox.blockLiqMassDens[blockIdx.x * EQBLOCKS + curId] = gDBox.sumLiqMassDens[blockIdx.x] / (float) EQBLOCKSIZE;
        gDBox.blockVapMassDens[blockIdx.x * EQBLOCKS + curId] = gDBox.sumVapMassDens[blockIdx.x] / (float) EQBLOCKSIZE;
        for(int i = 0; i < gConf.subNum[0]; i++){
            gDBox.blockLiqConc[blockIdx.x * EQBLOCKS + curId * gConf.subNum[0] + i] = gDBox.sumLiqConc[blockIdx.x + i] / (float) EQBLOCKSIZE;
            gDBox.blockVapConc[blockIdx.x * EQBLOCKS + curId * gConf.subNum[0] + i] = gDBox.sumVapConc[blockIdx.x + i] / (float) EQBLOCKSIZE;
        }
        
        
        //zeros block
        gDBox.sumLiqEn[blockIdx.x] = 0.0;
        gDBox.sumVapEn[blockIdx.x] = 0.0;
        gDBox.sumLiqMolEn[blockIdx.x] = 0.0;
        gDBox.sumVapMolEn[blockIdx.x] = 0.0;
        gDBox.sumLiqPress[blockIdx.x] = 0.0;
        gDBox.sumVapPress[blockIdx.x] = 0.0;
        gDBox.sumLiqMassDens[blockIdx.x] = 0.0;
        gDBox.sumVapMassDens[blockIdx.x] = 0.0;
        for(int i = 0; i < gConf.subNum[0]; i++){
            gDBox.sumLiqMol[blockIdx.x * gConf.subNum[0] + i] = 0.0;
            gDBox.sumVapMol[blockIdx.x * gConf.subNum[0] + i] = 0.0;
            gDBox.sumLiqConc[blockIdx.x * gConf.subNum[0] + i] = 0.0;
            gDBox.sumVapConc[blockIdx.x * gConf.subNum[0] + i] = 0.0;
        }
        
         //PRINT propertyes
        printf("---------------------------------------------------------------\n");
        for(int curBlock = 0; curBlock < EQBLOCKS; curBlock++){
            printf("dev block %d %d liq en %f vap en\n", blockIdx.x, curBlock, gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + curBlock], curBlock, gDBox.blockVapEn[blockIdx.x * EQBLOCKS + curBlock]);
        }
        
        printf("liq en %f liq press %f vap en %f vap press %f\n", gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + curId], gDBox.blockLiqPress[blockIdx.x * EQBLOCKS + curId], gDBox.blockVapEn[blockIdx.x * EQBLOCKS + curId], gDBox.blockVapPress[blockIdx.x * EQBLOCKS + curId] );
        for(int i = 0; i < gConf.subNum[0]; i++){
            printf("sub %d of subnum %d liq conc %d %f vap conc %d %f\n",i, gConf.subNum[0], gDBox.blockLiqConc[blockIdx.x * gConf.subNum[0] + i], gDBox.blockLiqMol[blockIdx.x * gConf.subNum[0] + i], gDBox.blockVapConc[blockIdx.x * gConf.subNum[0] + i], gDBox.blockVapMol[blockIdx.x * gConf.subNum[0]]);
        }
        printf("dens liq %f vap %f\n", gDBox.blockLiqMassDens[blockIdx.x * EQBLOCKS + curId], gDBox.blockVapMassDens[blockIdx.x * EQBLOCKS + curId]);
        printf("liq trans accept %d reject %d vap trans accept %d reject %d  \n", gDBox.accLiqTrans[blockIdx.x], gDBox.rejLiqTrans[blockIdx.x], gDBox.accVapTrans[blockIdx.x], gDBox.rejVapTrans[blockIdx.x]);
        printf("nliq %d nvap %d\n", gDBox.nLiq[blockIdx.x], gDBox.nVap[blockIdx.x]);
        printf("vol change accept %d reject %d\n", gDBox.accVolChange[blockIdx.x], gDBox.rejVolChange[blockIdx.x]);
        printf("trans move accept %d %d reject %d %d\n", gDBox.accLiq2Vap[blockIdx.x], gDBox.accVap2Liq[blockIdx.x], gDBox.rejLiq2Vap[blockIdx.x], gDBox.rejVap2Liq[blockIdx.x]);
    }
   
    
    __syncthreads();
    return 0;
}

__device__ int double_max_tran_change(gDoublebox &gDBox, gOptions gConf, gMolecula gTop){
    
    if(threadIdx.x == 0){
    //liqud
        if((gDBox.accLiqTrans[blockIdx.x]+1)/(gDBox.rejLiqTrans[blockIdx.x]+1) > 0.6){
            gDBox.maxLiqTrans[blockIdx.x]*=1.2;
        }
        if(gDBox.maxLiqTrans[blockIdx.x] > gDBox.liqRcut[blockIdx.x]){
            gDBox.maxLiqTrans[blockIdx.x] = gDBox.liqRcut[blockIdx.x];
        }
        if((gDBox.accLiqTrans[blockIdx.x]+1)/(gDBox.rejLiqTrans[blockIdx.x]+1) < 0.4){
            gDBox.maxLiqTrans[blockIdx.x]*=0.8;
        }
        if(gDBox.maxLiqTrans[blockIdx.x] < 0.01){
            gDBox.maxLiqTrans[blockIdx.x] < 0.01;
        }
    //vapor
        if((gDBox.accVapTrans[blockIdx.x]+1)/(gDBox.rejVapTrans[blockIdx.x]+1) > 0.6){
            gDBox.maxVapTrans[blockIdx.x]*=1.2;
        }
        if(gDBox.maxVapTrans[blockIdx.x] > gDBox.vapRcut[blockIdx.x]){
            gDBox.maxVapTrans[blockIdx.x] = gDBox.vapRcut[blockIdx.x];
        }
        if((gDBox.accVapTrans[blockIdx.x]+1)/(gDBox.rejVapTrans[blockIdx.x]+1) < 0.4){
            gDBox.maxLiqTrans[blockIdx.x]*=0.8;
        }
        if(gDBox.maxLiqTrans[blockIdx.x] < 0.01){
            gDBox.maxLiqTrans[blockIdx.x] < 0.01;
        }
    //vol change
        if((gDBox.accVolChange[blockIdx.x]+1)/(gDBox.rejVolChange[blockIdx.x]+1) > 0.5){
            gDBox.maxVolChange[blockIdx.x] *= 0.8;
        }
        if(gDBox.accVolChange[blockIdx.x] < 1){
            gDBox.maxVolChange[blockIdx.x] *= 1.1;
        }
    //zeroes accept/reject
        gDBox.accLiqTrans[blockIdx.x] = 0;
        gDBox.rejLiqTrans[blockIdx.x] = 0;
        gDBox.accVapTrans[blockIdx.x] = 0;
        gDBox.rejVapTrans[blockIdx.x] = 0;
        gDBox.accVolChange[blockIdx.x] = 0;
        gDBox.rejVolChange[blockIdx.x] = 0;
        gDBox.accLiq2Vap[blockIdx.x] = 0;
        gDBox.rejLiq2Vap[blockIdx.x] = 0;
        gDBox.accVap2Liq[blockIdx.x] = 0;
        gDBox.rejVap2Liq[blockIdx.x] = 0;
        
    }
    __syncthreads;
    return 0;
}

__device__ int double_check_equilibration(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int curId){
    float maxEn;
    float maxDens;
    float maxPres;
    
    if((threadIdx.x == 0) && (curId > EQBLOCKS)){
        //get average value
        gDBox.avLiqEn[blockIdx.x] = 0.0;
        gDBox.avVapEn[blockIdx.x] = 0.0;
        gDBox.avLiqPress[blockIdx.x] = 0.0;
        gDBox.avVapPress[blockIdx.x] = 0.0;
        gDBox.avLiqMol[blockIdx.x] = 0.0;
        gDBox.avVapMol[blockIdx.x] = 0.0;
        gDBox.avLiqMassDens[blockIdx.x] = 0.0;
        gDBox.avVapMassDens[blockIdx.x] = 0.0;
        for(int i = 0; i < gConf.subNum[0]; i++){
            gDBox.avLiqConc[blockIdx.x * gConf.subNum[0] + i] = 0.0;
            gDBox.avVapConc[blockIdx.x * gConf.subNum[0] + i] = 0.0;
        }
        for(int i = 0; i < EQBLOCKS; i++){
            gDBox.avLiqEn[blockIdx.x] += gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + i];
            gDBox.avVapEn[blockIdx.x] += gDBox.blockVapEn[blockIdx.x * EQBLOCKS + i];
            gDBox.avLiqPress[blockIdx.x] += gDBox.blockLiqPress[blockIdx.x * EQBLOCKS + i];
            gDBox.avVapPress[blockIdx.x] += gDBox.blockVapPress[blockIdx.x * EQBLOCKS + i];
            gDBox.avLiqMassDens[blockIdx.x] += gDBox.blockLiqMassDens[blockIdx.x * EQBLOCKS + i];
            gDBox.avVapMassDens[blockIdx.x] += gDBox.blockVapMassDens[blockIdx.x * EQBLOCKS + i];
        }
        gDBox.avLiqEn[blockIdx.x] /= EQBLOCKS;
        gDBox.avVapEn[blockIdx.x] /= EQBLOCKS;
        gDBox.avLiqPress[blockIdx.x] /= EQBLOCKS;
        gDBox.avVapPress[blockIdx.x] /= EQBLOCKS;
        gDBox.avLiqMassDens[blockIdx.x] /= EQBLOCKS;
        gDBox.avVapMassDens[blockIdx.x] /= EQBLOCKS;
        
        maxEn = 0.0;
        maxPres = 0.0;
        maxPres = 0.0;
        //check energy
        for(int i = 0; i < EQBLOCKS; i++){
            if(maxEn < abs(gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + i] - gDBox.avLiqEn[blockIdx.x]) / abs(gDBox.avLiqEn[blockIdx.x] )){
                maxEn = abs(gDBox.blockLiqEn[blockIdx.x * EQBLOCKS + i] - gDBox.avLiqEn[blockIdx.x]) / abs(gDBox.avLiqEn[blockIdx.x]);
            }
            if(maxEn < abs(gDBox.blockVapEn[blockIdx.x * EQBLOCKS + i] - gDBox.avVapEn[blockIdx.x]) / abs(gDBox.avVapEn[blockIdx.x] )){
                maxEn = abs(gDBox.blockVapEn[blockIdx.x * EQBLOCKS + i] - gDBox.avVapEn[blockIdx.x]) / abs(gDBox.avVapEn[blockIdx.x]);
            }
            if(maxDens < abs(gDBox.blockLiqMassDens[blockIdx.x * EQBLOCKS + i] - gDBox.avLiqMassDens[blockIdx.x]) / abs(gDBox.avLiqMassDens[blockIdx.x] )){
                maxDens = abs(gDBox.blockLiqMassDens[blockIdx.x * EQBLOCKS + i] - gDBox.avLiqMassDens[blockIdx.x]) / abs(gDBox.avLiqMassDens[blockIdx.x]);
            }
            if(maxDens < abs(gDBox.blockVapMassDens[blockIdx.x * EQBLOCKS + i] - gDBox.avVapMassDens[blockIdx.x]) / abs(gDBox.avVapMassDens[blockIdx.x] )){
                maxDens = abs(gDBox.blockVapMassDens[blockIdx.x * EQBLOCKS + i] - gDBox.avVapMassDens[blockIdx.x]) / abs(gDBox.avVapMassDens[blockIdx.x]);
            }
            if(maxPres < abs(gDBox.blockVapPress[blockIdx.x * EQBLOCKS + i] - gDBox.avVapPress[blockIdx.x]) / abs(gDBox.avVapPress[blockIdx.x] )){
                maxPres = abs(gDBox.blockVapPress[blockIdx.x * EQBLOCKS + i] - gDBox.avVapPress[blockIdx.x]) / abs(gDBox.avVapPress[blockIdx.x]);
            }
        }
        if(maxEn > 0.05){
            printf("====energy not equlibrated %f\n", maxEn);
        }
        if(maxDens > 0.05){
            printf("====density not equlibrated %f\n", maxDens);
        }
        if(maxPres > 0.05){
            printf("====pressure not equlibrated %f\n", maxPres);
        }
        if((maxEn < 0.05) && (maxDens < 0.05) && (maxPres < 0.05)){
            gDBox.eqStep[blockIdx.x] = curId;
        }
    }
    else{
        if(threadIdx.x == 0){
            printf("current block %d less %d\n", curId, EQBLOCKS);
        }
    }
    
    return 0;
}

