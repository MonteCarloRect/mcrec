#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"

__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop);
__device__  int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir);
__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempTotVir, int fmol, int smol, float rcut);
__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir, int mol);

///////

int double_equilibration(gDoublebox* &gDBox, hDoubleBox doubleBox, gOptions* gConf, gMolecula* gTop){
    hipError_t cuErr;
    int* xDim;
    int temp;
    
    xDim = (int*) malloc(deviceCount * sizeof(int));
    for(int curDev = 0; curDev < deviceCount; curDev++){
        //set numbers of block equal to numbers of plate per device
        //numbers of thread equal to maximum numbers of molecules 
        temp = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //get maximum number of molecules in liquib phase
            if(doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]] > temp){
                temp = doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]];
            }
        }
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //get maximum numbers of molecules in vapor phase
            if(doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]] > temp){
                temp = doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]];
            }
        }
        if(temp == 0){
            xDim[curDev] = 1;
        }
        else{
            if(log2(temp) > 7){
                xDim[curDev] = MAXDIM; //set blocksize as maximum dimension
            }
            else{
                xDim[curDev] = pow(2,ceil(log2(temp))); //set block size 
            }
        }
    }
    hipDeviceSynchronize();    //sync
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //calculate equlibration
        double_equilib_cycle<<<doubleBox.devicePlates[curDev], xDim[curDev]>>>(gDBox[curDev], gConf[curDev], gTop[curDev]);
    }
    hipDeviceSynchronize();    //sync after complit all equlibrations
    return 0;
}

__global__ void double_equilib_cycle(gDoublebox gDBox, gOptions gConf, gMolecula gTop){
    int temp;
    __shared__ int yDim;   //y Dimension of array
    float* tempTotEn;   //temp array for calculate total enegry
    float* tempMolEn;   //temp array for calculate molcule energy
    float* tempTotVir;  //temp array for calculate total virial
    float* tempMolVir;  //temp array for calculate molecule virial
    if(threadIdx.x == 0){
        tempTotEn = (float*) malloc(MAXDIM * gridDim.x * sizeof(float)); //enegry of yDim molecules
        tempMolEn = (float*) malloc(MAXDIM * gridDim.x * sizeof(float));
    }
    
    //calculate total energy of phases
    if(threadIdx.x == 0){   //shared for whjole block
        yDim = ceilf(gDBox.molNum[blockIdx.x] / blockDim.x);
    }
    __syncthreads();
    
    double_totalen(gDBox, gConf, gTop, yDim, tempTotEn, tempMolEn, tempTotVir, tempMolVir);
    if(gDBox.molNum[blockIdx.x] > 0){ //cycle only if plate not empty
        for(int i = 0; i < 1000; i++){  //loop for some (set as a parameter to option file)
            //get random number of molecule for block (plate)
            
            //get random transition
                //move molecule
                
                //change volume
                
                //molecule transition
                
        }
        //check equlibration status
    }
    else{
        //mark block as a compleet
    }
    //free arrays
    free(tempTotEn);
    free(tempMolEn);
    free(tempTotVir);
    free(tempMolVir);
}

__device__ int double_totalen(gDoublebox &gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir){
    int curMol; //current molecule
    int curMol2;    //second molecule
    int curId;  //current index
    int reduce;
    //nuber of plate equal block number 
    
    //======calculate liquid phase energy/virial
    tempTotEn[blockIdx.x * MAXDIM + threadIdx.x] = 0.0;  //set energy to zero
    tempTotVir[blockIdx.x * MAXDIM + threadIdx.x] = 0.0; //set virial to zero
    for(int i = 0; i < yDim; i++){
        curId = threadIdx.x * yDim + i; //current id of molecule in liqud list
        if(curId < gDBox.nLiq[blockIdx.x]){
            curMol = gDBox.fMolOnPlate[blockIdx.x] + gDBox.liqList[curId];  //get curent molecule GPU index
            //set here intermolecular potential
            
            //calculate inermolecullar interaction
            for(int j = i+1; j < gDBox.nLiq[blockIdx.x]; j++){
                curMol2 = gDBox.fMolOnPlate[blockIdx.x] + gDBox.liqList[j];
                double_mol_pair_energy(gDBox, gConf, gTop, yDim, tempTotEn, tempTotVir, curMol, curMol2, gDBox.liqRcut[blockIdx.x]);
            }
        }
        else{
            //set energy to zero (plase not used)
            tempTotEn[blockIdx.x * MAXDIM + threadIdx.x] += 0.0;
            tempTotVir[blockIdx.x * MAXDIM + threadIdx.x] += 0.0;
        }
    }
    __syncthreads();    //chech all slots are calculated
    //summ all energyes
    reduce = blockDim.x / 2;
    while(reduce > 0){
        if(threadIdx.x < reduce){
            tempTotEn[blockIdx.x * MAXDIM + threadIdx.x] += tempTotEn[blockIdx.x * MAXDIM + threadIdx.x + reduce];
            tempTotVir[blockIdx.x * MAXDIM + threadIdx.x] += tempTotVir[blockIdx.x * MAXDIM + threadIdx.x + reduce];
        }
        reduce = reduce / 2;
        __syncthreads();
        //
    }
    __syncthreads();
    printf("total energy %f ", tempTotEn[blockIdx.x * MAXDIM]);
    
    return 0;
}

__device__ int double_mol_pair_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempTotVir, int fmol, int smol, float rcut){
    //fmol smol - indexes of firs and second molecules
    //get atom numbers
    float dmx;  //diffs in molecule coordinats
    float dmy;
    float dmz;
    float dax;  //diff in atoms coordinate
    float day;
    float daz;
    float r;    //distance ^2 of molecules
    float ra;   //distance ^2 of atoms
    int id;     //array indexes for potential parameters
    
    dmx = gDBox.xm[fmol] - gDBox.xm[smol];
    dmy = gDBox.ym[fmol] - gDBox.ym[smol];
    dmz = gDBox.zm[fmol] - gDBox.zm[smol];
    
    if(dmx > rcut){  //
        dmx = -rcut * 2.0 + dmx;
    }
    if(dmy > rcut ){
        dmy = -rcut * 2.0 + dmy;
    }
    if(dmz > rcut){
        dmz = -rcut * 2.0 + dmz;
    }
    if(dmx < -rcut){ //
        dmx = rcut * 2.0 + dmx;
    }
    if(dmy < -rcut){
        dmy = rcut*2.0 + dmy;
    }
    if(dmz < -rcut){
        dmz = rcut*2.0 + dmz;
    }
    r = dmx * dmx + dmy * dmy + dmz * dmz;
    
    if(r < rcut*rcut){ //if distance < rcut
        // for all atoms
        for(int i = 0; i < gTop.aNum[gDBox.mType[fmol]] ; i++){
            for(int j = 0; j < gTop.aNum[gDBox.mType[smol]]; j++){
                id = 1; // gBox.aType[gBox.fAtom[b] + j] * gConf.potNum[0] + gBox.aType[gBox.fAtom[a] + i];
                id = gTop.aType[i] * gConf.potNum[0] + gTop.aType[j];
                dax = gDBox.xa[gDBox.fAtomOfMol[fmol]+i] - gDBox.xa[gDBox.fAtomOfMol[smol] + j] + dmx;
                day = gDBox.ya[gDBox.fAtomOfMol[fmol]+i] - gDBox.ya[gDBox.fAtomOfMol[smol] + j] + dmy;
                daz = gDBox.za[gDBox.fAtomOfMol[fmol]+i] - gDBox.za[gDBox.fAtomOfMol[smol] + j] + dmy;
                
                ra = dax * dax + day * day + daz * daz;
                ra = gTop.sigma[id] * gTop.sigma[id] / ra;
                ra = ra * ra * ra;  //6 power
                //calculate potential
                tempTotEn[threadIdx.x] += gTop.epsi[id] * (ra* ra - ra);
                tempTotVir[threadIdx.x] += gTop.epsi[id] * (6.0f*ra - 12.0f*ra*ra);
            }
        }
    }
    
    return 0;
}

__device__ int double_mol_single_energy(gDoublebox gDBox, gOptions gConf, gMolecula gTop, int yDim, float* &tempTotEn, float* &tempMolEn, float* &tempTotVir, float* &tempMolVir, int mol){
    
    return 1;
}


