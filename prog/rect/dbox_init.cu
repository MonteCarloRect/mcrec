#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "../mcrec.h"
#include "../global.h"
//#include "../initial.h"

int plates_initial_state(options &config, hDoubleBox &doubleBox, gSingleBox &hostData, molecules* initMol, int deviceCount){
int insert; //number of inserted molecules
int latticeRow; //rows on cubic lattice

//temporally lattice for molecules insert
float* tempXm;
float* tempYm;
float* tempZm;
int* tempFree;  //1 if place is free
float latticeDelta; //lattice delta
float volFrac;  //fraction of liqud volume

int id; //tmporally id
int maxArrSize; //maximum size of temp array
int randMol;    //random molecules


if(config.plateInit == VAK){    //vakuum at all plates
    printf("%s \n Initial plate state: vakuum %s\n",ANSI_COLOR_GREEN, ANSI_COLOR_RESET);
    //allocate arrays
    //.flowEns = (int *) malloc(config.flowNum * sizeof(int));
    
    doubleBox.molNum = (int*) malloc(config.plateNum * sizeof(int));    //numbers of molecules per plate
    //set initial coordinates of molecules and atoms
    doubleBox.xm = (float**) malloc(config.plateNum * sizeof(float*));
    doubleBox.ym = (float**) malloc(config.plateNum * sizeof(float*));
    doubleBox.zm = (float**) malloc(config.plateNum * sizeof(float*));
    doubleBox.mType = (int**) malloc(config.plateNum * sizeof(int*));
    doubleBox.gpuIndex = (int**) malloc(config.plateNum * sizeof(int*));
    
    doubleBox.nVap = (int*) malloc(config.plateNum * sizeof(int));
    doubleBox.nLiq = (int*) malloc(config.plateNum * sizeof(int));
    doubleBox.liqList = (int**) malloc(config.plateNum * sizeof(int*));
    doubleBox.vapList = (int**) malloc(config.plateNum * sizeof(int*));
    
    doubleBox.xa = (float***) malloc(config.plateNum * sizeof(float**));
    doubleBox.ya = (float***) malloc(config.plateNum * sizeof(float**));
    doubleBox.za = (float***) malloc(config.plateNum * sizeof(float**));
    
    doubleBox.refEnergy = (float*) malloc(config.plateNum * sizeof(float)); //referance energy of plate
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.nVap[i] = 0; //initial molecules in vapor
        doubleBox.nLiq[i] = 0;  //initial molecules in liquid 
        doubleBox.molNum[i] = 0;    //if vak initial number is 0
        doubleBox.refEnergy[i] = 0.0;   //set energy of phases to zero
        //doubleBox.liqEnergy[i] = 0.0;
    }
    printf("\n test 0 \n");
    for(int i = 0; i < config.flowNum; i++){
        for(int j = 0; j < config.subNum; j++){
            //set molecules in inputs of molecules
            doubleBox.molNum[config.plateIn[i]] = doubleBox.molNum[config.plateIn[i]] + config.flowIns[i][j];
            //set reference energy of plate
            doubleBox.refEnergy[config.plateIn[i]] = doubleBox.refEnergy[config.plateIn[i]] + config.flowIns[i][j] * hostData.avEnergy[i];
        }
        doubleBox.nLiq[i] += doubleBox.molNum[i];   //all molecules goes to liquid phase
    }
    
    //set numbers of molecules
    doubleBox.molNumType = (int**) malloc(config.plateNum * sizeof(int*));
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.molNumType[i] = (int*) malloc(config.subNum * sizeof(int));
        for(int j = 0; j < config.subNum; j++){
            doubleBox.molNumType[i][j] = 0;
        }
    }
    
    //set atoms
    for(int i = 0; i < config.flowNum; i++){
        for(int j = 0; j < config.subNum; j++){
            printf(" i %d j %d inplate %d ", i, j, config.plateIn[i]);
            doubleBox.molNumType[config.plateIn[i]][j] = config.flowIns[i][j];
            printf(" molecules %d \n", doubleBox.molNumType[config.plateIn[i]][j]);
        }
    }
    
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.xm[i] = (float*) malloc(doubleBox.molNum[i] * sizeof(float) + 1);
        doubleBox.ym[i] = (float*) malloc(doubleBox.molNum[i] * sizeof(float) + 1);
        doubleBox.zm[i] = (float*) malloc(doubleBox.molNum[i] * sizeof(float) + 1);
        doubleBox.mType[i] = (int*) malloc(doubleBox.molNum[i] * sizeof(int) +1);
        doubleBox.gpuIndex[i] = (int*) malloc(doubleBox.molNum[i] * sizeof(int) +1);
        doubleBox.vapList[i] = (int*) malloc(doubleBox.molNum[i] * sizeof(int) +1);
        doubleBox.liqList[i] = (int*) malloc(doubleBox.molNum[i] * sizeof(int) +1);
        
        doubleBox.xa[i] = (float**) malloc(doubleBox.molNum[i] * sizeof(float*) + 1);
        doubleBox.ya[i] = (float**) malloc(doubleBox.molNum[i] * sizeof(float*) + 1);
        doubleBox.za[i] = (float**) malloc(doubleBox.molNum[i] * sizeof(float*) + 1);
    }
    //allocate volumes of boxes
    doubleBox.liqVol = (float*) malloc(config.plateNum * sizeof(float));
    doubleBox.vapVol = (float*) malloc(config.plateNum * sizeof(float));
    //allocate type of atoms
    printf("\n test 2 \n");
    doubleBox.type = (int**) malloc(config.plateNum * sizeof(int*));
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.type[i] = (int*) malloc(doubleBox.molNum[i] * sizeof(int));
//        id = 0;
//        for(int j = 0; j < config.subNum; j++){
//            for(int k = 0; k < doubleBox.molNumType[i][j]; k++){
//                doubleBox.type[i][id] = j; //initMol[j].atomNum
//                id++;
//            }
//        }
    }
    
    volFrac = 0.1;  //fraction of liquid phase
    //molecules inserted
    for(int i = 0; i < config.plateNum; i++){   //for each plate
        //set initial lattice for molecules insert
        printf(" insert molecules to plate %d \n", i);
        latticeRow = pow(doubleBox.molNum[i], 1.0/3.0) + 2;
        tempXm = (float*) malloc(latticeRow * latticeRow * latticeRow * sizeof(float));
        tempYm = (float*) malloc(latticeRow * latticeRow * latticeRow * sizeof(float));
        tempZm = (float*) malloc(latticeRow * latticeRow * latticeRow * sizeof(float));
        tempFree = (int*) malloc(latticeRow * latticeRow * latticeRow * sizeof(int));
        //initial volume of gas and liqud phase
        doubleBox.liqVol[i] = volFrac * config.plateVol;
        doubleBox.vapVol[i] = config.plateVol - doubleBox.liqVol[i];
        
        id = 0;
        for(int ii = 0; ii < latticeRow; ii++){
            for(int jj = 0; jj < latticeRow; jj++){
                for(int kk = 0; kk < latticeRow; kk++){
                    tempFree[id] = 1;
                    tempXm[id] = (ii + 0.5) * latticeDelta;
                    tempYm[id] = (jj + 0.5) * latticeDelta;
                    tempZm[id] = (kk + 0.5) * latticeDelta;
                    id++;
                }
            }
        }
        srand(time(0));
        maxArrSize = latticeRow * latticeRow * latticeRow;
        id = 0;
        for(int j = 0; j < config.subNum; j++){ //for each substance type
            printf("plate %d substance %d molecules to insert %d \n", i, j, doubleBox.molNumType[i][j]);
            insert = 0;
            while(insert < doubleBox.molNumType[i][j]){   //insert molecule at random plase
                randMol = rand() % maxArrSize;
                if(tempFree[randMol] == 1){
                    //set coordinates of molecules
                    doubleBox.xm[i][id] = tempXm[randMol];  
                    doubleBox.ym[i][id] = tempYm[randMol];
                    doubleBox.zm[i][id] = tempZm[randMol];
                    doubleBox.mType[i][id] = j; //set type of molecules
                    doubleBox.liqList[i][id] = id;  //set current molecule to liquid phase
                    //doubleBox.nLiq[i]++;    //add one more molecule to liquid phase
                    //doubleBox.nVap[i]++;    //summ molecules DO THIS UPPER
                    
                    doubleBox.xa[i][id] = (float*) malloc(initMol[j].atomNum * sizeof(float));
                    doubleBox.ya[i][id] = (float*) malloc(initMol[j].atomNum * sizeof(float));
                    doubleBox.za[i][id] = (float*) malloc(initMol[j].atomNum * sizeof(float));
                    //set coordinates of atoms
                    for(int k = 0; k < initMol[j].atomNum; k++){
                        doubleBox.xa[i][id][k] = initMol[j].x[k];
                        doubleBox.ya[i][id][k] = initMol[j].y[k];
                        doubleBox.za[i][id][k] = initMol[j].z[k];
                    }
                    id++;
                    insert++;
                    tempFree[randMol] = 0;  //plase not free
                }
                
            }
        }
        free(tempXm);
        free(tempYm);
        free(tempZm);
        free(tempFree);
    }
    //allocate devices plate
    doubleBox.plateDevice = (int*) malloc(config.plateNum * sizeof(int));
    doubleBox.devicePlates = (int*) malloc(deviceCount * sizeof(int));
    doubleBox.platesPerDevice = (int **) malloc(deviceCount * sizeof(int*));
    for(int i = 0; i < deviceCount; i++){
        doubleBox.platesPerDevice[i] = (int*) malloc(config.plateNum * sizeof(int));
    }
    
}
printf("%s initial state of plate done %s\n",ANSI_COLOR_GREEN, ANSI_COLOR_RESET);
return 0;
}
