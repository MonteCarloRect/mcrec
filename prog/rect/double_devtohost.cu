#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"


//transfer data from host to devices
int double_box_host_to_device(options &config, hDoubleBox &doubleBox, gDoublebox* &gDBox, gDoublebox &hDBox, gSingleBox &hostData, molecules* initMol, int deviceCount){
    hipError_t cuErr;
    int sum;
    int sum2;
    int id;
    int id2;
    int idL;
    int idV;
    
    
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //printf("cuda set device \n");
        //allocate total number of plates
        hDBox.pltNum = (int*) malloc(deviceCount * sizeof(int));
        for(int i = 0; i < deviceCount; i++){
            hDBox.pltNum[i] = doubleBox.devicePlates[i];
        }
        cuErr = hipMalloc(&gDBox[curDev].pltNum, deviceCount*sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].molNum %s line %d, err: %s, device %d\n", __FILE__, __LINE__, hipGetErrorString(cuErr), curDev);
        }
        cuErr = hipMemcpy(gDBox[curDev].pltNum, hDBox.pltNum, deviceCount * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate global number of plates for device
        
        
        //allocate total molecules per plate
        hDBox.molNum = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.molNum[i] = doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].molNum, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].molNum %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].molNum, hDBox.molNum, doubleBox.devicePlates[curDev] * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate molecules coordinates
        sum = 0;
        sum2 = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            sum += doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];  //add molecules in current plate
            //get number of atoms in each molecules
            for(int j = 0; j < doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]]; j++){
                sum2 += initMol[doubleBox.mType[doubleBox.platesPerDevice[curDev][i]][j]].atomNum;
            }
        }
        hDBox.xm = (float*) malloc(sum * sizeof(float));
        hDBox.ym = (float*) malloc(sum * sizeof(float));
        hDBox.zm = (float*) malloc(sum * sizeof(float));
        hDBox.mType = (int*) malloc(sum * sizeof(int));
        
        //allocate numbers of liquid/vapor molecules
        hDBox.nVap = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        hDBox.nLiq = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.nVap[i] = doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]];
            hDBox.nLiq[i] = doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].nVap, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].nVap %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].nLiq, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].nLiq %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].nVap, hDBox.nVap, doubleBox.devicePlates[curDev] * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].nLiq, hDBox.nLiq, doubleBox.devicePlates[curDev] * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate energyes and virial
        hDBox.liqEn = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        hDBox.vapEn = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        hDBox.refEn = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        hDBox.liqVir = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        hDBox.vapVir = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        hDBox.temp = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.liqEn[i] = 0.0;
            hDBox.vapEn[i] = 0.0;
            hDBox.refEn[i] = doubleBox.refEnergy[doubleBox.platesPerDevice[curDev][i]];
            hDBox.temp[i] = doubleBox.temp[doubleBox.platesPerDevice[curDev][i]];
            hDBox.liqVir[i] = 0.0;
            hDBox.vapVir[i] = 0.0;
        }
        cuErr = hipMalloc(&gDBox[curDev].liqEn, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].liqEn %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].vapEn, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapEn %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].refEn, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].refEn %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].liqVir, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].liqVir %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].vapVir, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapVir %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].temp, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].temp %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        cuErr = hipMemcpy(gDBox[curDev].liqEn, hDBox.liqEn, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].vapEn, hDBox.vapEn, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].refEn, hDBox.refEn, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].liqVir, hDBox.liqVir, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].vapVir, hDBox.vapVir, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].temp, hDBox.temp, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        cuErr = hipMalloc(&gDBox[curDev].tempLiqEn, doubleBox.devicePlates[curDev] * MAXDIM * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapVir %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].tempLiqVir, doubleBox.devicePlates[curDev] * MAXDIM * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapVir %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].tempVapEn, doubleBox.devicePlates[curDev] * MAXDIM * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapVir %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].tempVapVir, doubleBox.devicePlates[curDev] * MAXDIM * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapVir %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        
        
        //allocate atoms
        hDBox.xa = (float*) malloc(sum2 * sizeof(float));
        hDBox.ya = (float*) malloc(sum2 * sizeof(float));
        hDBox.za = (float*) malloc(sum2 * sizeof(float));
        
        //allocate GPU molecules data
        cuErr = hipMalloc(&gDBox[curDev].xm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].xm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].ym, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].ym %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].zm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].zm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].mType, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].mType %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate GPU atoms data
        cuErr = hipMalloc(&gDBox[curDev].xa, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].xa %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].ya, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].ya %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].za, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].za %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate first molecules and first atoms
        hDBox.fMolOnPlate = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        hDBox.fAtomOfMol = (int*) malloc(sum * sizeof(int));
        cuErr = hipMalloc(&gDBox[curDev].fMolOnPlate, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].fMolOnPlate %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].fAtomOfMol, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].fAtomOfMol %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //printf("test21-1 \n");
        //calculate first molecules index in plate
        id = 0; //set to zero index for each device for molecule
        id2 = 0;    //set to zero index for each first atopm in molecule
        idL = 0;
        idV = 0;
        hDBox.liqList = (int*) malloc(sum * sizeof(int));
        hDBox.vapList = (int*) malloc(sum * sizeof(int));
        hDBox.phaseType = (int*) malloc(sum * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.fMolOnPlate[i] = id;
            int curPlate = doubleBox.platesPerDevice[curDev][i];    //current plate
            for(int j = 0; j < doubleBox.molNum[curPlate]; j++){   //over all molecules on plate
                hDBox.xm[id] = doubleBox.xm[curPlate][j];   //[number of plate] [number of molecule]
                hDBox.ym[id] = doubleBox.ym[curPlate][j];
                hDBox.zm[id] = doubleBox.zm[curPlate][j];
                hDBox.mType[id] = doubleBox.mType[curPlate][j];
                if(doubleBox.phaseType[curPlate][j] == LIQ){
                    hDBox.liqList[hDBox.fMolOnPlate[i] + idL] = j;
                    idL++;
                }
                else{
                    hDBox.vapList[hDBox.fMolOnPlate[i] + idV] = j;
                    idV++;
                }
                hDBox.phaseType[id] = doubleBox.phaseType[curPlate][j];
                hDBox.fAtomOfMol[id] = id2;
                id++;
                //printf("device %d plate %d liq %d vap %d\n", curDev, curPlate, hDBox.liqList[hDBox.fMolOnPlate[i] + j], hDBox.vapList[hDBox.fMolOnPlate[i] + j] );
                for(int k = 0; k < initMol[doubleBox.mType[curPlate][j]].atomNum; k++){
                    hDBox.xa[id2] = doubleBox.xa[curPlate][j][k];
                    hDBox.ya[id2] = doubleBox.ya[curPlate][j][k];
                    hDBox.za[id2] = doubleBox.za[curPlate][j][k];
                    id2++;
                }
            }
        }        
        cuErr = hipMalloc(&gDBox[curDev].liqList, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].liqList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].vapList, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].phaseType, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].phaseType %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].liqList, hDBox.liqList, sum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].vapList, hDBox.vapList, sum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].phaseType, hDBox.phaseType, sum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //copy molecules data to GPU
        cuErr = hipMemcpy(gDBox[curDev].xm, hDBox.xm, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].ym, hDBox.ym, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].zm, hDBox.zm, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //copy first atoms
        cuErr = hipMemcpy(gDBox[curDev].fMolOnPlate, hDBox.fMolOnPlate, doubleBox.devicePlates[curDev]*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].fAtomOfMol, hDBox.fMolOnPlate, sum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //copy atoms to GPU
        cuErr = hipMemcpy(gDBox[curDev].xa, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].ya, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].za, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //equilibrated cycle
        hDBox.eqStep = (int*) malloc(config.plateNum * sizeof(int));
        for(int i = 0; i < config.plateNum; i++){
            hDBox.eqStep[i] = 0;
        }        
        cuErr = hipMalloc(&gDBox[curDev].eqStep, config.plateNum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].eqStep %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].eqStep, hDBox.eqStep, config.plateNum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //phases volumes
        hDBox.liqVol = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.liqVol[i] = doubleBox.liqVol[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].liqVol, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].nVap %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].liqVol, hDBox.liqVol, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        hDBox.vapVol = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.vapVol[i] = doubleBox.vapVol[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].vapVol, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapVol %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].vapVol, hDBox.vapVol, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //phases cut radius
        hDBox.liqRcut = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.liqRcut[i] = doubleBox.liqRcut[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].liqRcut, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].liqRcut %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].liqRcut, hDBox.liqRcut, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        hDBox.vapRcut = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.vapRcut[i] = doubleBox.vapRcut[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].vapRcut, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapRcut %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].vapRcut, hDBox.vapRcut, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate maximum transition
        hDBox.maxLiqTrans = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        hDBox.maxVapTrans = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.maxLiqTrans[i] = 0.3;
            hDBox.maxVapTrans[i] = 0.3;
        }
        cuErr = hipMalloc(&gDBox[curDev].maxLiqTrans, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].maxLiqTrans %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].maxVapTrans, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].maxLiqTrans %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].maxLiqTrans, hDBox.maxLiqTrans, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].maxVapTrans, hDBox.maxVapTrans, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate accept move counter
        cuErr = hipMalloc(&gDBox[curDev].accLiqTrans, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].accLiqTrans %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].rejLiqTrans, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].rejLiqTrans %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].accVapTrans, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].accVapTrans %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].rejVapTrans, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].rejVapTrans %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        
        
        //free host arrays
        free(hDBox.pltNum);
        free(hDBox.molNum);
        free(hDBox.xm);
        free(hDBox.ym);
        free(hDBox.zm);
        free(hDBox.mType);
        free(hDBox.nVap);
        free(hDBox.nLiq);
        free(hDBox.xa);
        free(hDBox.ya);
        free(hDBox.za);
        free(hDBox.fMolOnPlate);
        free(hDBox.fAtomOfMol);
        free(hDBox.liqList);
        free(hDBox.vapList);
        free(hDBox.eqStep);
        free(hDBox.vapVol);
        free(hDBox.vapRcut);
        free(hDBox.liqVol);
        free(hDBox.liqRcut);
        free(hDBox.liqEn);
        free(hDBox.vapEn);
        free(hDBox.refEn);
        free(hDBox.liqVir);
        free(hDBox.vapVir);
        free(hDBox.maxLiqTrans);
        free(hDBox.maxVapTrans);
        printf("\n free %d done\n", curDev);
    }
    
    return 1;
}


//first time allocation plates to devices
int double_box_init_allocate(options &config, hDoubleBox &doubleBox, int deviceCount){
    int id;
    //allocate plates evenly
    printf("distribution of plates\n");
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.plateDevice[i] = (int) i * deviceCount / config.plateNum;
        printf(" plate %d device %d\n", i, doubleBox.plateDevice[i]);
    }
    for(int i = 0; i < deviceCount; i++){
        doubleBox.devicePlates[i] = 0;
        for(int j = 0; j < config.plateNum; j++){
            doubleBox.platesPerDevice[i][j] = 0;
        }
    }
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.devicePlates[doubleBox.plateDevice[i]]++;
    }
    for(int i = 0; i < deviceCount; i++){
        id = 0;
        for(int j = 0; j < config.plateNum; j++){
            if(doubleBox.plateDevice[j] == i){
                doubleBox.platesPerDevice[i][id] = j;
                id++;
            }
        }
    }
    
    for(int i = 0; i < deviceCount; i++){
        printf(" device %d calculate %d plates \n ", i, doubleBox.devicePlates[i]);
        for(int j = 0; j < doubleBox.devicePlates[i]; j++){
            printf(" pl  %d   \n", doubleBox.platesPerDevice[i][j]);
        }
    }
    
    return 1;
}


