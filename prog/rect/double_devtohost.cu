#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"


//transfer data from host to devices
int double_box_host_to_device(options &config, hDoubleBox &doubleBox, gDoublebox gDBox, gDoublebox hDBox, gSingleBox &hostData, molecules* initMol, int deviceCount){
    hipError_t cuErr;
    int sum;
    
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate total number of plates
        hDBox.pltNum = (int*) malloc(deviceCount * sizeof(int));
        for(int i = 0; i < deviceCount; i++){
            hDBox.pltNum[i] = doubleBox.devicePlates[i];
        }
        cuErr = hipMalloc(&gDBox.pltNum, deviceCount*sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.molNum %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.pltNum, hDBox.pltNum, deviceCount*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate global number of plates for device
        
        
        //allocate total molecules per plate
        hDBox.molNum = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.molNum[i] = doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox.molNum, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.molNum %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.molNum, hDBox.molNum, deviceCount*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate molecules coordinates
        sum = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            sum += doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];
        }
        hDBox.xm = (float*) malloc(sum * sizeof(float));
        hDBox.ym = (float*) malloc(sum * sizeof(float));
        hDBox.zm = (float*) malloc(sum * sizeof(float));
        hDBox.mType = (int*) malloc(sum * sizeof(int));
        hDBox.nVap = (int*) malloc(sum * sizeof(int));
        hDBox.nLiq = (int*) malloc(sum * sizeof(int));
        
        
        cuErr = hipMalloc(&gDBox.xm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.xm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.ym, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.ym %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.zm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.zm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        
        
        
        //free host arrays
        free(hDBox.pltNum);
        free(hDBox.molNum);
        free(hDBox.xm);
        free(hDBox.ym);
        free(hDBox.zm);
        printf("\n free %d done\n", curDev);
    }
    
    return 1;
}


//first time allocation plates to devices
int double_box_init_allocate(options &config, hDoubleBox &doubleBox, int deviceCount){
    int id;
    //allocate plates evenly
    printf("distribution of plates\n");
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.plateDevice[i] = (int) i * deviceCount / config.plateNum;
        printf(" plate %d device %d\n", i, doubleBox.plateDevice[i]);
    }
    for(int i = 0; i < deviceCount; i++){
        doubleBox.devicePlates[i] = 0;
        for(int j = 0; j < config.plateNum; j++){
            doubleBox.platesPerDevice[i][j] = 0;
        }
    }
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.devicePlates[doubleBox.plateDevice[i]]++;
    }
    for(int i = 0; i < deviceCount; i++){
        id = 0;
        for(int j = 0; j < config.plateNum; j++){
            if(doubleBox.plateDevice[j] == i){
                doubleBox.platesPerDevice[i][id] = j;
                id++;
            }
        }
    }
    
    for(int i = 0; i < deviceCount; i++){
        printf(" device %d calculate %d plates \n ", i, doubleBox.devicePlates[i]);
        for(int j = 0; j < doubleBox.devicePlates[i]; j++){
            printf(" pl  %d   \n", doubleBox.platesPerDevice[i][j]);
        }
    }
    
    return 1;
}


