#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"


//transfer data from host to devices
int double_box_host_to_device(options &config, hDoubleBox &doubleBox, gDoublebox &gDBox, gDoublebox &hDBox, gSingleBox &hostData, molecules* initMol, int deviceCount){
    hipError_t cuErr;
    int sum;
    int sum2;
    int id;
    int id2;
    
    for(int curDev = 0; curDev < deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate total number of plates
        hDBox.pltNum = (int*) malloc(deviceCount * sizeof(int));
        for(int i = 0; i < deviceCount; i++){
            hDBox.pltNum[i] = doubleBox.devicePlates[i];
        }
        cuErr = hipMalloc(&gDBox.pltNum, deviceCount*sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.molNum %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.pltNum, hDBox.pltNum, deviceCount*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate global number of plates for device
        
        
        //allocate total molecules per plate
        hDBox.molNum = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.molNum[i] = doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox.molNum, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.molNum %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.molNum, hDBox.molNum, deviceCount*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate molecules coordinates
        sum = 0;
        sum2 = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            sum += doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];  //add molecules in current plate
            //get number of atoms in each molecules
            for(int j = 0; j < doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]]; j++){
                sum2 += initMol[doubleBox.mType[doubleBox.platesPerDevice[curDev][i]][j]].atomNum;
            }
        }
        hDBox.xm = (float*) malloc(sum * sizeof(float));
        hDBox.ym = (float*) malloc(sum * sizeof(float));
        hDBox.zm = (float*) malloc(sum * sizeof(float));
        hDBox.mType = (int*) malloc(sum * sizeof(int));
        
        //allocate numbers of liquid/vapor molecules
        hDBox.nVap = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        hDBox.nLiq = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.nVap[i] = doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]];
            hDBox.nLiq[i] = doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox.nVap, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.nVap %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.nLiq, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.nLiq %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.nVap, hDBox.nVap, doubleBox.devicePlates[curDev] * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.nLiq, hDBox.nLiq, doubleBox.devicePlates[curDev] * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate atoms
        hDBox.xa = (float*) malloc(sum2 * sizeof(float));
        hDBox.ya = (float*) malloc(sum2 * sizeof(float));
        hDBox.za = (float*) malloc(sum2 * sizeof(float));
        
        //allocate GPU molecules data
        cuErr = hipMalloc(&gDBox.xm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.xm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.ym, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.ym %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.zm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.zm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.mType, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.mType %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate GPU atoms data
        cuErr = hipMalloc(&gDBox.xa, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.xa %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.ya, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.ya %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.za, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.za %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate first molecules and first atoms
        hDBox.fMolOnPlate = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        hDBox.fAtomOfMol = (int*) malloc(sum * sizeof(int));
        cuErr = hipMalloc(&gDBox.fMolOnPlate, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.fMolOnPlate %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.fAtomOfMol, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.fAtomOfMol %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //printf("test21-1 \n");
        //calculate first molecules index in plate\u00d1
        id = 0; //set to zero index for each device for molecule
        id2 = 0;    //set to zero index for each first atopm in molecule
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.fMolOnPlate[i] = id;
            int curPlate = doubleBox.platesPerDevice[curDev][i];    //current plate
            for(int j = 0; j < doubleBox.molNum[curPlate]; j++){   //over all molecules on plate
                hDBox.xm[id] = doubleBox.xm[curPlate][j];   //[number of plate] [number of molecule]
                hDBox.ym[id] = doubleBox.ym[curPlate][j];
                hDBox.zm[id] = doubleBox.zm[curPlate][j];
                hDBox.mType[id] = doubleBox.mType[curPlate][j];
                doubleBox.gpuIndex[curPlate][j] = id;   //set internal GPU index
                id++;
                hDBox.fAtomOfMol[id] = id2;
                for(int k = 0; k < initMol[doubleBox.mType[curPlate][j]].atomNum; k++){
                    hDBox.xa[id2] = doubleBox.xa[curPlate][j][k];
                    hDBox.ya[id2] = doubleBox.ya[curPlate][j][k];
                    hDBox.za[id2] = doubleBox.za[curPlate][j][k];
                    id2++;
                }
            }
        }
        //printf("test21-2 \n");
        //set liquid/vapor lists
        hDBox.liqList = (int*) malloc(sum * sizeof(int));
        hDBox.vapList = (int*) malloc(sum * sizeof(int));
        cuErr = hipMalloc(&gDBox.liqList, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.liqList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox.vapList, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.vapList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        id = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //over plates
            int curPlate = doubleBox.platesPerDevice[curDev][i];    //current plate
            for(int j = 0; j < hDBox.nLiq[i]; j++){            //all molecules in liquid
                hDBox.liqList[hDBox.fMolOnPlate[i] + j] = doubleBox.gpuIndex[curPlate][j];
            }
            for(int j = 0; j < hDBox.nVap[i]; j++){            //all molecules in vapor
                hDBox.vapList[hDBox.fMolOnPlate[i] + j] = doubleBox.gpuIndex[curPlate][j];
            }
        }
        
        
        //printf("test21-3 \n");
        //copy molecules data to GPU
        cuErr = hipMemcpy(gDBox.xm, hDBox.xm, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.ym, hDBox.ym, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.zm, hDBox.zm, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //copy first atoms
        cuErr = hipMemcpy(gDBox.fMolOnPlate, hDBox.fMolOnPlate, doubleBox.devicePlates[curDev]*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.fAtomOfMol, hDBox.fMolOnPlate, sum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //copy atoms to GPU
        cuErr = hipMemcpy(gDBox.xa, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.ya, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.za, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //equilibrated cycle
        hDBox.eqStep = (int*) malloc(config.plateNum * sizeof(int));
        for(int i = 0; i < config.plateNum; i++){
            hDBox.eqStep[i] = 0;
        }        
        cuErr = hipMalloc(&gDBox.eqStep, config.plateNum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox.liqList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox.eqStep, hDBox.eqStep, config.plateNum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //free host arrays
        free(hDBox.pltNum);
        free(hDBox.molNum);
        free(hDBox.xm);
        free(hDBox.ym);
        free(hDBox.zm);
        free(hDBox.mType);
        free(hDBox.nVap);
        free(hDBox.nLiq);
        free(hDBox.xa);
        free(hDBox.ya);
        free(hDBox.za);
        free(hDBox.fMolOnPlate);
        free(hDBox.fAtomOfMol);
        free(hDBox.liqList);
        free(hDBox.vapList);
        printf("\n free %d done\n", curDev);
    }
    
    return 1;
}


//first time allocation plates to devices
int double_box_init_allocate(options &config, hDoubleBox &doubleBox, int deviceCount){
    int id;
    //allocate plates evenly
    printf("distribution of plates\n");
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.plateDevice[i] = (int) i * deviceCount / config.plateNum;
        printf(" plate %d device %d\n", i, doubleBox.plateDevice[i]);
    }
    for(int i = 0; i < deviceCount; i++){
        doubleBox.devicePlates[i] = 0;
        for(int j = 0; j < config.plateNum; j++){
            doubleBox.platesPerDevice[i][j] = 0;
        }
    }
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.devicePlates[doubleBox.plateDevice[i]]++;
    }
    for(int i = 0; i < deviceCount; i++){
        id = 0;
        for(int j = 0; j < config.plateNum; j++){
            if(doubleBox.plateDevice[j] == i){
                doubleBox.platesPerDevice[i][id] = j;
                id++;
            }
        }
    }
    
    for(int i = 0; i < deviceCount; i++){
        printf(" device %d calculate %d plates \n ", i, doubleBox.devicePlates[i]);
        for(int j = 0; j < doubleBox.devicePlates[i]; j++){
            printf(" pl  %d   \n", doubleBox.platesPerDevice[i][j]);
        }
    }
    
    return 1;
}


