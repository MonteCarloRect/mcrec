#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../mcrec.h"


//transfer data from host to devices
int double_box_host_to_device(options &config, hDoubleBox &doubleBox, gDoublebox* &gDBox, gDoublebox &hDBox, gSingleBox &hostData, molecules* initMol, int deviceCount){
    hipError_t cuErr;
    int sum;
    int sum2;
    int id;
    int id2;
    
    
    
    for(int curDev = 0; curDev > deviceCount; curDev++){
        cuErr = hipSetDevice(curDev);  //set to current device
        if(cuErr != hipSuccess){
            printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //printf("cuda set device \n");
        //allocate total number of plates
        hDBox.pltNum = (int*) malloc(deviceCount * sizeof(int));
        for(int i = 0; i < deviceCount; i++){
            hDBox.pltNum[i] = doubleBox.devicePlates[i];
        }
        cuErr = hipMalloc(&gDBox[curDev].pltNum, deviceCount*sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].molNum %s line %d, err: %s, device %d\n", __FILE__, __LINE__, hipGetErrorString(cuErr), curDev);
        }
        cuErr = hipMemcpy(gDBox[curDev].pltNum, hDBox.pltNum, deviceCount * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate global number of plates for device
        
        
        //allocate total molecules per plate
        hDBox.molNum = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.molNum[i] = doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].molNum, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].molNum %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].molNum, hDBox.molNum, deviceCount*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //allocate molecules coordinates
        sum = 0;
        sum2 = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            sum += doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]];  //add molecules in current plate
            //get number of atoms in each molecules
            for(int j = 0; j < doubleBox.molNum[doubleBox.platesPerDevice[curDev][i]]; j++){
                sum2 += initMol[doubleBox.mType[doubleBox.platesPerDevice[curDev][i]][j]].atomNum;
            }
        }
        hDBox.xm = (float*) malloc(sum * sizeof(float));
        hDBox.ym = (float*) malloc(sum * sizeof(float));
        hDBox.zm = (float*) malloc(sum * sizeof(float));
        hDBox.mType = (int*) malloc(sum * sizeof(int));
        
        //allocate numbers of liquid/vapor molecules
        hDBox.nVap = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        hDBox.nLiq = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.nVap[i] = doubleBox.nVap[doubleBox.platesPerDevice[curDev][i]];
            hDBox.nLiq[i] = doubleBox.nLiq[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].nVap, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].nVap %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].nLiq, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].nLiq %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].nVap, hDBox.nVap, doubleBox.devicePlates[curDev] * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].nLiq, hDBox.nLiq, doubleBox.devicePlates[curDev] * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate atoms
        hDBox.xa = (float*) malloc(sum2 * sizeof(float));
        hDBox.ya = (float*) malloc(sum2 * sizeof(float));
        hDBox.za = (float*) malloc(sum2 * sizeof(float));
        
        //allocate GPU molecules data
        cuErr = hipMalloc(&gDBox[curDev].xm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].xm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].ym, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].ym %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].zm, sum * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].zm %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].mType, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].mType %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate GPU atoms data
        cuErr = hipMalloc(&gDBox[curDev].xa, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].xa %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].ya, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].ya %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].za, sum2 * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].za %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //allocate first molecules and first atoms
        hDBox.fMolOnPlate = (int*) malloc(doubleBox.devicePlates[curDev] * sizeof(int));
        hDBox.fAtomOfMol = (int*) malloc(sum * sizeof(int));
        cuErr = hipMalloc(&gDBox[curDev].fMolOnPlate, doubleBox.devicePlates[curDev] * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].fMolOnPlate %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].fAtomOfMol, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].fAtomOfMol %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //printf("test21-1 \n");
        //calculate first molecules index in plate
        id = 0; //set to zero index for each device for molecule
        id2 = 0;    //set to zero index for each first atopm in molecule
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.fMolOnPlate[i] = id;
            int curPlate = doubleBox.platesPerDevice[curDev][i];    //current plate
            for(int j = 0; j < doubleBox.molNum[curPlate]; j++){   //over all molecules on plate
                hDBox.xm[id] = doubleBox.xm[curPlate][j];   //[number of plate] [number of molecule]
                hDBox.ym[id] = doubleBox.ym[curPlate][j];
                hDBox.zm[id] = doubleBox.zm[curPlate][j];
                hDBox.mType[id] = doubleBox.mType[curPlate][j];
                doubleBox.gpuIndex[curPlate][j] = id;   //set internal GPU index
                id++;
                hDBox.fAtomOfMol[id] = id2;
                for(int k = 0; k < initMol[doubleBox.mType[curPlate][j]].atomNum; k++){
                    hDBox.xa[id2] = doubleBox.xa[curPlate][j][k];
                    hDBox.ya[id2] = doubleBox.ya[curPlate][j][k];
                    hDBox.za[id2] = doubleBox.za[curPlate][j][k];
                    id2++;
                }
            }
        }
        //printf("test21-2 \n");
        //set liquid/vapor lists
        hDBox.liqList = (int*) malloc(sum * sizeof(int));
        hDBox.vapList = (int*) malloc(sum * sizeof(int));
        cuErr = hipMalloc(&gDBox[curDev].liqList, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].liqList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMalloc(&gDBox[curDev].vapList, sum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        id = 0;
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){    //over plates
            int curPlate = doubleBox.platesPerDevice[curDev][i];    //current plate
            for(int j = 0; j < hDBox.nLiq[i]; j++){            //all molecules in liquid
                hDBox.liqList[hDBox.fMolOnPlate[i] + j] = doubleBox.gpuIndex[curPlate][j];
            }
            for(int j = 0; j < hDBox.nVap[i]; j++){            //all molecules in vapor
                hDBox.vapList[hDBox.fMolOnPlate[i] + j] = doubleBox.gpuIndex[curPlate][j];
            }
        }
        
        
        //printf("test21-3 \n");
        //copy molecules data to GPU
        cuErr = hipMemcpy(gDBox[curDev].xm, hDBox.xm, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].ym, hDBox.ym, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].zm, hDBox.zm, sum * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //copy first atoms
        cuErr = hipMemcpy(gDBox[curDev].fMolOnPlate, hDBox.fMolOnPlate, doubleBox.devicePlates[curDev]*sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].fAtomOfMol, hDBox.fMolOnPlate, sum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //copy atoms to GPU
        cuErr = hipMemcpy(gDBox[curDev].xa, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].ya, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].za, hDBox.xa, sum2 * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //equilibrated cycle
        hDBox.eqStep = (int*) malloc(config.plateNum * sizeof(int));
        for(int i = 0; i < config.plateNum; i++){
            hDBox.eqStep[i] = 0;
        }        
        cuErr = hipMalloc(&gDBox[curDev].eqStep, config.plateNum * sizeof(int));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].liqList %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].eqStep, hDBox.eqStep, config.plateNum * sizeof(int), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        //phases volumes
        hDBox.liqVol = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.liqVol[i] = doubleBox.liqVol[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].liqVol, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].nVap %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].liqVol, hDBox.liqVol, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        hDBox.vapVol = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.vapVol[i] = doubleBox.vapVol[doubleBox.platesPerDevice[curDev][i]];
        }
        cuErr = hipMalloc(&gDBox[curDev].vapVol, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapVol %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].vapVol, hDBox.vapVol, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        //phases cut radius
        hDBox.liqRcut = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.liqRcut[i] = pow(doubleBox.liqVol[doubleBox.platesPerDevice[curDev][i]],1.0/3.0);
        }
        cuErr = hipMalloc(&gDBox[curDev].liqRcut, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].liqRcut %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].liqRcut, hDBox.liqRcut, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        hDBox.vapRcut = (float*) malloc(doubleBox.devicePlates[curDev] * sizeof(float));
        for(int i = 0; i < doubleBox.devicePlates[curDev]; i++){
            hDBox.vapRcut[i] = pow(doubleBox.vapVol[doubleBox.platesPerDevice[curDev][i]], 1.0/3.0);
        }
        cuErr = hipMalloc(&gDBox[curDev].vapRcut, doubleBox.devicePlates[curDev] * sizeof(float));
        if(cuErr != hipSuccess){
            printf("Cannot allocate gDBox[curDev].vapRcut %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        cuErr = hipMemcpy(gDBox[curDev].vapRcut, hDBox.vapRcut, doubleBox.devicePlates[curDev] * sizeof(float), hipMemcpyHostToDevice);
        if(cuErr != hipSuccess){
            printf("Cannot copy memory to device file %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
        }
        
        
        //free host arrays
        free(hDBox.pltNum);
        free(hDBox.molNum);
        free(hDBox.xm);
        free(hDBox.ym);
        free(hDBox.zm);
        free(hDBox.mType);
        free(hDBox.nVap);
        free(hDBox.nLiq);
        free(hDBox.xa);
        free(hDBox.ya);
        free(hDBox.za);
        free(hDBox.fMolOnPlate);
        free(hDBox.fAtomOfMol);
        free(hDBox.liqList);
        free(hDBox.vapList);
        free(hDBox.eqStep);
        free(hDBox.vapVol);
        free(hDBox.vapRcut);
        free(hDBox.liqVol);
        free(hDBox.liqRcut);
        printf("\n free %d done\n", curDev);
    }
    
    return 1;
}


//first time allocation plates to devices
int double_box_init_allocate(options &config, hDoubleBox &doubleBox, int deviceCount){
    int id;
    //allocate plates evenly
    printf("distribution of plates\n");
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.plateDevice[i] = (int) i * deviceCount / config.plateNum;
        printf(" plate %d device %d\n", i, doubleBox.plateDevice[i]);
    }
    for(int i = 0; i < deviceCount; i++){
        doubleBox.devicePlates[i] = 0;
        for(int j = 0; j < config.plateNum; j++){
            doubleBox.platesPerDevice[i][j] = 0;
        }
    }
    for(int i = 0; i < config.plateNum; i++){
        doubleBox.devicePlates[doubleBox.plateDevice[i]]++;
    }
    for(int i = 0; i < deviceCount; i++){
        id = 0;
        for(int j = 0; j < config.plateNum; j++){
            if(doubleBox.plateDevice[j] == i){
                doubleBox.platesPerDevice[i][id] = j;
                id++;
            }
        }
    }
    
    for(int i = 0; i < deviceCount; i++){
        printf(" device %d calculate %d plates \n ", i, doubleBox.devicePlates[i]);
        for(int j = 0; j < doubleBox.devicePlates[i]; j++){
            printf(" pl  %d   \n", doubleBox.platesPerDevice[i][j]);
        }
    }
    
    return 1;
}


