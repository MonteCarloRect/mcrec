#include "hip/hip_runtime.h"
//
#include <stdio.h>
#include "../mcrec.h"

int write_singlebox_log(FILE* logFile, gSingleBox &hData){
    float sumE;
    float sumP;
    //print on sreeen
    printf("%s Singlebox results %s\n", ANSI_COLOR_BLUE, ANSI_COLOR_RESET);
    printf("energy %f pressure %f \n", hData.avEnergy[0], hData.avPressure[0]);
    printf("energy correction %f pressure correction %f \n", hData.energyCorr[0], hData.pressureCorr[0]);
    sumE = 0.0f;
    sumP = 0.0f;
    for(int i=0; i<EQBLOCKS; i++){
        sumE += (hData.eqBlockEnergy[0 * EQBLOCKS +i] - hData.avEnergy[0]) * (hData.eqBlockEnergy[0 * EQBLOCKS +i] - hData.avEnergy[0]);
        sumP += (hData.eqBlockPressure[0 * EQBLOCKS +i] - hData.avPressure[0]) * (hData.eqBlockPressure[0 * EQBLOCKS +i] - hData.avPressure[0]);
    }
    sumE = sqrt(1.0 / (EQBLOCKS - 1.0) * sumE);
    sumP = sqrt(1.0 / (EQBLOCKS - 1.0) * sumP);
    printf("total energy correction %f +/- %f total pressure %f +/- %f \n", hData.avEnergy[0] + hData.energyCorr[0], sumE, hData.avPressure[0] + hData.pressureCorr[0], sumP);
    //write log
    
    return 0;
}

