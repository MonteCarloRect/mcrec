#include <stdio.h>
#include "../mcrec.h"

int write_prop_log(int deviceCount, hipDeviceProp_t* deviceProp, FILE* logFile){
	fprintf(logFile,"Device count %d:\n", deviceCount);
	for(int i=0;i<deviceCount;i++){
		fprintf(logFile, " -- %s\n",deviceProp[i].name);
	}
	return 0;
}

int write_config_log(options con,FILE* logFile){
	fprintf(logFile,"Number of substances %d, files:\n",con.subNum);
	for(int i=0;i<con.subNum;i++){
		fprintf(logFile,"%d -- %s\n",i, con.subFile[i]);
	}
	return 0;
}
