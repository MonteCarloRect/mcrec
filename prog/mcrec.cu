#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "global.h"
#include "mcrec.h"
#include "initial.h"
//#include <hiprand/hiprand_kernel.h>
//#include <hiprand.h>


int main (int argc, char * argv[]){
    //begin time
    time(&beginTime);
    printf("begin time %s", asctime(localtime(&beginTime))); 
//openlog file
    logFile=fopen("calculation.log","w");
    //
    get_device_prop(deviceCount, deviceProp);
    if(deviceCount< 1 ){
        printf("No CUDA device is detected\n");
        return 1;
    }
    write_prop_log(deviceCount, deviceProp, logFile);
    //
//    cuErr = hipMalloc(&gDBox, deviceCount * sizeof(gDoublebox));
//    if(cuErr != hipSuccess){
//        printf("Cannot allocate gDBox %s line %d, err: %s, device %d\n", __FILE__, __LINE__, hipGetErrorString(cuErr), deviceCount);
//    }

//    for(int i = 0; i < deviceCount; i++){
//        cuErr = hipMalloc((void**)&gConf, deviceCount * sizeof(gOptions));
//        if(cuErr != hipSuccess){
//            printf("Cannot allocate gConf %s line %d, err: %s, device %d\n", __FILE__, __LINE__, hipGetErrorString(cuErr), deviceCount);
//        }
//    }
//    printf("test 16.02\n");
//    cuErr = hipMalloc(&gTop, deviceCount * sizeof(gMolecula));
//    if(cuErr != hipSuccess){
//        printf("Cannot allocate gConf %s line %d, err: %s, device %d\n", __FILE__, __LINE__, hipGetErrorString(cuErr), deviceCount);
//    }
    
    gConf = (gOptions*) malloc(deviceCount * sizeof(gOptions));
    gTop = (gMolecula*) malloc(deviceCount * sizeof(gMolecula));
    gDBox = (gDoublebox*) malloc(deviceCount * sizeof(gDoublebox)); //allocate GPU data
    
    
    //read initial data
    read_options(config);
    write_config_log(config,logFile);
    //read gro data for each molecules
    initMol = (molecules *) malloc(config.subNum * sizeof(molecules));
    read_init_gro(config, initMol);
    //read topology of molecules
    read_top(allParams,paramsLines);
    for(int i=0;i<paramsLines;i++){
        printf("  %s \n",allParams[i].aName);
    }
    printf("number of top lines %d\n",paramsLines);
    //create initial simulation
    initial_flows(config, initFlows, initMol, gpuSingleBox, paramsLines, allParams, gpuParams, hostParams, gpuMixParams, hostMixParams, deviceProp);
    //copy data to GPU device
    data_to_device(gBox, initFlows, gConf, config, gTop, hostParams, initMol, hostData, hostTop, hostConf, deviceCount, gDBox);
    cuErr = hipGetLastError();
    printf("Cuda data2device last error: %s\n", hipGetErrorString(cuErr));
    //calculate initial flows
    dim3 singleThread(config.singleXDim);
    printf(" grid %d  - %d \n", singleThread.x, singleThread.y);
    
    cuErr = hipSetDevice(0);  //set to current device
    if(cuErr != hipSuccess){
        printf("Cannot swtich to device %s line %d, err: %s\n", __FILE__, __LINE__, hipGetErrorString(cuErr));
    }
    single_calc<<<config.flowNum, singleThread,0, gDBox[0].stream>>>(config.singleYDim, gConf[0], gTop[0], gBox);
    
    hipDeviceSynchronize();
    cuErr = hipGetLastError();
    printf("Cuda singlecalc last error: %s\n", hipGetErrorString(cuErr));
    //get single box data from gpu
    data_from_device(gBox, hostData, config);
    //calculate pressure enegry correction
    rcut(hostData, config, hostTop, initMol, hostConf);
    //print out results
    write_singlebox_log(logFile, hostData);
    
    //
    for(int i = 0; i < config.flowNum; i++){
        printf("flow %d avergae energy %f\n",i, hostData.avEnergy[i]);
    }
    //Double box
    //initial inserting of molecules
    
    //push data to devices
    
    //start cycle
    
    
//close log file
    freeAll(gpuSingleBox,initFlows,config);
    fclose(logFile);
    //end time
    time(&endTime);
    printf("end time %s\n", asctime(localtime(&endTime))); 
    printf("elapsed time %f sec", difftime(endTime, beginTime));
    
    //generate initial stata on plates
    plates_initial_state(config, doubleBox, hostData, initMol, deviceCount);
    //int double_box_init_allocate(options &config, hDoubleBox &doubleBox, int deviceCount);
    double_box_init_allocate(config, doubleBox, deviceCount);
    printf("double allocate done\n");
    
//    int double_box_host_to_device(options &config, hDoubleBox &doubleBox, gDoublebox gDBox, gDoublebox hDBox, gSingleBox &hostData, molecules* initMol, int deviceCount);
    double_box_host_to_device(config, doubleBox, gDBox, hDBox, hostData, initMol, deviceCount);
    printf("host to device done\n");
    
    double_equilibration(gDBox, doubleBox, gConf, gTop);
    printf("equlibration done");
    
    //printf("%f - %f\n", hDBox.xa[0], hDBox.xm[0]);
    printf("Done\n");
}
