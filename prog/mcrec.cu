#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "global.h"
#include "mcrec.h"
#include "initial.h"


int main (int argc, char * argv[]){

//openlog file
    logFile=fopen("calculation.log","w");
    //
    get_device_prop(deviceCount, deviceProp);
    write_prop_log(deviceCount,deviceProp,logFile);
    //read initial data
    read_options(config);
    write_config_log(config,logFile);
    //read gro data for each molecules
    initMol = (molecules *) malloc(config.subNum * sizeof(molecules));
    read_init_gro(config, initMol);
    //read topology of molecules
    read_top(allParams,paramsLines);
    for(int i=0;i<paramsLines;i++){
        printf("  %s \n",allParams[i].aName);
    }
    printf("number of top lines %d\n",paramsLines);
    //create initial simulation
    initial_flows(config, initFlows,initMol,gpuSingleBox,paramsLines,allParams,gpuParams,hostParams, gpuMixParams, hostMixParams,deviceProp);
    printf("\n test %d \n", initFlows[0].molNum);
    //put data to device
//    data_to_device(initFlows,gpuSingleBox,config);
    
    printf("\n test2 %d %d \n", initFlows[0].molNum, config.flowNum);
    ///calculate initial flows
    dim3 singleThread(config.singleXDim);
    printf(" grid %d  - %d \n", singleThread.x, singleThread.y);
    single_calc<<<config.flowNum,singleThread>>>(gpuSingleBox,gpuParams,config.singleYDim);
//close log file
    freeAll(gpuSingleBox,initFlows,config);
    fclose(logFile);


}
