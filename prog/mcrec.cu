#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "global.h"
#include "mcrec.h"
#include "initial.h"
//#include <hiprand/hiprand_kernel.h>
//#include <hiprand.h>


int main (int argc, char * argv[]){
    //begin time
    time(&beginTime);
    printf("begin time %s", asctime(localtime(&beginTime))); 
//openlog file
    logFile=fopen("calculation.log","w");
    //
    get_device_prop(deviceCount, deviceProp);
    if(deviceCount<1){
        printf("No CUDA device is detected\n");
        return 1;
    }
    write_prop_log(deviceCount, deviceProp, logFile);
    //read initial data
    read_options(config);
    write_config_log(config,logFile);
    //read gro data for each molecules
    initMol = (molecules *) malloc(config.subNum * sizeof(molecules));
    read_init_gro(config, initMol);
    //read topology of molecules
    read_top(allParams,paramsLines);
    for(int i=0;i<paramsLines;i++){
        printf("  %s \n",allParams[i].aName);
    }
    printf("number of top lines %d\n",paramsLines);
    //create initial simulation
    initial_flows(config, initFlows, initMol, gpuSingleBox, paramsLines, allParams, gpuParams, hostParams, gpuMixParams, hostMixParams, deviceProp);
    //copy data to GPU device
    data_to_device(gBox, initFlows, gConf, config, gTop, hostParams, initMol, hostData);
    cuErr = hipGetLastError();
    printf("Cuda data2device last error: %s\n", hipGetErrorString(cuErr));
    //calculate initial flows
    dim3 singleThread(config.singleXDim);
    printf(" grid %d  - %d \n", singleThread.x, singleThread.y);
    single_calc<<<config.flowNum, singleThread>>>(config.singleYDim, gConf, gTop, gBox);
    hipDeviceSynchronize();
    cuErr = hipGetLastError();
    printf("Cuda singlecalc last error: %s\n", hipGetErrorString(cuErr));
    //get single box data from gpu
    data_from_device(gBox, hostData, config);
    //calculate pressure enegry correction
    //!!!!!!!!!!!!!!!!
    //print out results
    write_singlebox_log(logFile, hostData);
//close log file
    freeAll(gpuSingleBox,initFlows,config);
    fclose(logFile);
    //end time
    time(&endTime);
    printf("end time %s\n", asctime(localtime(&endTime))); 
    printf("elapsed time %f sec", difftime(endTime, beginTime));
}
