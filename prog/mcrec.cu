#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "global.h"
#include "mcrec.h"
#include "initial.h"
//#include <hiprand/hiprand_kernel.h>
//#include <hiprand.h>


int main (int argc, char * argv[]){

//openlog file
    logFile=fopen("calculation.log","w");
    //
    get_device_prop(deviceCount, deviceProp);
    if(deviceCount<1){
        printf("No CUDA device is detected\n");
        return 1;
    }
    write_prop_log(deviceCount,deviceProp,logFile);
    //read initial data
    read_options(config);
    write_config_log(config,logFile);
    //read gro data for each molecules
    initMol = (molecules *) malloc(config.subNum * sizeof(molecules));
    read_init_gro(config, initMol);
    //read topology of molecules
    read_top(allParams,paramsLines);
    for(int i=0;i<paramsLines;i++){
        printf("  %s \n",allParams[i].aName);
    }
    printf("number of top lines %d\n",paramsLines);
    //create initial simulation
    initial_flows(config, initFlows, initMol, gpuSingleBox, paramsLines, allParams, gpuParams, hostParams, gpuMixParams, hostMixParams, deviceProp);
    //copy data to GPU device
    data_to_device(gBox, initFlows,gConf, config, gTop, hostParams, initMol);
    cuErr = hipGetLastError();
    printf("Cuda data2device last error: %s\n", hipGetErrorString(cuErr));
    //calculate initial flows
    dim3 singleThread(config.singleXDim);
    printf(" grid %d  - %d \n", singleThread.x, singleThread.y);
    single_calc<<<config.flowNum, singleThread>>>(config.singleYDim, gConf, gTop, gBox);
    hipDeviceSynchronize();
    cuErr = hipGetLastError();
    printf("Cuda singlecalc last error: %s\n", hipGetErrorString(cuErr));
    printf("ololo\n");
//close log file
    freeAll(gpuSingleBox,initFlows,config);
    fclose(logFile);


}
