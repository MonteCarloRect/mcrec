#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "global.h"
#include "mcrec.h"
#include "initial.h"


int main (int argc, char * argv[]){

//openlog file
    logFile=fopen("calculation.log","w");
    //
    get_device_prop(deviceCount, deviceProp);
    write_prop_log(deviceCount,deviceProp,logFile);
    //read initial data
    read_options(config);
    write_config_log(config,logFile);
    //read gro data for each molecules
    initMol = (molecules *) malloc(config.subNum * sizeof(molecules));
    read_init_gro(config, initMol);
//close log file
    fclose(logFile);


}
