
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SUBNUMMAX 10
#define BUFFER 255


int find_maximum(const int array[], int length) {
    int max = array[0];
    for (int i = 1; i < length; i++) {
        if (array[i] > max) {
            max = array[i];
        }
    }
    return max;
}

void get_string_section(const char string_input[], char string_output[], int begin, int length) {
    for (int i = 0; i < length; i++) {
        string_output[i] = string_input[begin + i];
    }
}

int get_integer_from_string(const char line[], int begin, int length) {
    char line_section[length];
    get_string_section(line, line_section, begin, length);
    return atoi(line_section);
}

float get_float(char a[], int begin, int length) {
    char b[length];
    for (int i = 0; i < length; i++) {
        b[i] = a[begin + i];
    }
    return atof(b);
}

int main(int argc, char *argv[]) {
    //----------------VAR
    int deviceCount;
    hipDeviceProp_t temppd;    //temp varaible
    hipDeviceProp_t *pd;    //array of device properties
    hipError_t currentError;    //current error
    FILE *fileId;    //input file ID
    FILE *file2Id;    //input file ID

    //substance
    int subNum;    //substance number
    char subFile[SUBNUMMAX][BUFFER];    //substance filenames
    int subAtomMax;    //maximum atom number in substances
    int *subAtomNum;    //atom numbers in molecules
    char **subName;    //substane residual name
    char ***subAtomName;    //attom names
    float3 **subAtomCoord;    //atom position in molecule
    float3 **subAtomVel;    //atom velocity


    //input flow
    int flowNum;    //flow numbers
    float **flowX;    //flow mole fractions
    float *flowT;    //flow temperatures
    float *flowN;    //flow number density
    int *flowIns;    //inserting molecules per

    //temp varaibles
    char tempString[BUFFER], tempString2[BUFFER];
    int tempInt, tempInt2;
    float tempFloat;
    char *tempString3, tempString4;



    //init vaaraibles
    int *initMolNum;

    //functions
    int find_maximum(int a[], int n);
    int get_integer_from_string(char a[], int n, int begin, int length);
    void get_string_section(char a[], char out[], int begin, int length);
    float get_float(char a[], int begin, int length);

    //----------------GET DEVICE PROPERTIES
    currentError = hipGetDeviceCount(&deviceCount);
    if (currentError != hipSuccess) {
        fprintf(stderr, "Cannot get CUDA device count: %s\n", hipGetErrorString(currentError));
        return 1;
    }
    if (!deviceCount) {
        fprintf(stderr, "No CUDA devices found\n");
        return 1;
    }
    pd = (hipDeviceProp_t *) malloc(deviceCount * sizeof(hipDeviceProp_t));
    for (int i = 0; i < deviceCount; i++) {
        hipGetDeviceProperties(&temppd, i);
//		priflowNlowT("Device name %s \n", pd.name);
        pd[i] = temppd;
        printf("Device name %s \n", pd[i].name);
        printf("Max Threads Dim: %d %d %d \n", pd[i].maxThreadsDim[0], pd[i].maxThreadsDim[1], pd[i].maxThreadsDim[2]);
        printf("Max Grid Size: %d %d %d \n", pd[i].maxGridSize[0], pd[i].maxGridSize[1], pd[i].maxGridSize[2]);
    }

    //------------------READ INPUT DATA
    fileId = fopen("data.mcr", "r");
    fscanf(fileId, "%d", &subNum);
//read molecules data
    printf("Substance number: %d\n", subNum);
    for (int i = 0; i < subNum; i++) {
        fscanf(fileId, "%s", subFile[i]);
    }
//read flow data
    fscanf(fileId, "%d", &flowNum);
    flowT = (float *) malloc(flowNum * sizeof(float));
    flowN = (float *) malloc(flowNum * sizeof(float));
    flowIns = (int *) malloc(flowNum * sizeof(int));
    for (int i = 0; i < flowNum; i++) {
        fscanf(fileId, "%f", &flowT[i]);
    }
    for (int i = 0; i < flowNum; i++) {
        fscanf(fileId, "%f", &flowN[i]);
    }
    for (int i = 0; i < flowNum; i++) {
        fscanf(fileId, "%d", &flowIns[i]);
    }
    flowX = (float **) malloc(flowNum * sizeof(float *));
    for (int i = 0; i < flowNum; i++) {
        flowX[i] = (float *) malloc(subNum * sizeof(float));
    }
    for (int i = 0; i < flowNum; i++) {
        for (int j = 0; j < subNum; j++) {
            fscanf(fileId, "%f", &flowX[i][j]);
        }
    }
    fclose(fileId);
//--------------------READ GRO FILES
//read molecules structure
    subAtomNum = (int *) malloc(subNum * sizeof(int));
    for (int i = 0; i < subNum; i++) {
        fileId = fopen(subFile[i], "r");
        if (fileId == NULL) {
            printf("Error opening file %s\n", subFile[i]);
            return 1;
        }
        fgets(tempString, BUFFER, fileId);
        fscanf(fileId, "%d", &subAtomNum[i]);
        fclose(fileId);
//			fscanf(file2Id,"%5d%-5s%5s%5d%8.3f%8.3f%8.3f%8.4f%8.4f%8.4f", )
    }
    //allocate gro varaibles
    subAtomMax = find_maximum(subAtomNum, subNum);
    printf("Maximum atoms numbers %d\n", subAtomMax);
    subName = (char **) malloc(subNum * sizeof(char *));    //allocate molecule names
    for (int i = 0; i < subNum; i++) {
        subName[i] = (char *) malloc(5 * sizeof(char));
    }
    subAtomCoord = (float3 **) malloc(subNum * sizeof(float3 * ));    //allocate coordinates
    for (int i = 0; i < subNum; i++) {
        subAtomCoord[i] = (float3 *) malloc(subAtomMax * sizeof(float3));
    }
    subAtomVel = (float3 **) malloc(subNum * sizeof(float3 * ));    //allocate velocities
    for (int i = 0; i < subNum; i++) {
        subAtomVel[i] = (float3 *) malloc(subAtomMax * sizeof(float3));
    }
    subAtomName = (char ***) malloc(subNum * sizeof(char **));    //allocate atoma names
    for (int i = 0; i < subNum; i++) {
        subAtomName[i] = (char **) malloc(subAtomMax * sizeof(char *));
        for (int j = 0; j < subAtomMax; j++) {
            subAtomName[i][j] = (char *) malloc(5 * sizeof(char));
        }
    }
    for (int i = 0; i < subNum; i++) {
        fileId = fopen(subFile[i], "r");
        fgets(tempString, BUFFER, fileId);    //skip 2 lines
        fgets(tempString, BUFFER, fileId);
        for (int j = 0; j < subAtomNum[i]; j++) {
            fgets(tempString2, BUFFER, fileId);    //read string to buffer
            get_string_section(tempString2, subName[i], 5, 5);    //parse string
            printf("sub %s\n", subName[i]);
            get_string_section(tempString2, subAtomName[i][j], 10, 5);
            printf("atom %s\n", subAtomName[i][j]);
            subAtomCoord[i][j].x = get_float(tempString2, 20, 8);
            subAtomCoord[i][j].y = get_float(tempString2, 28, 8);
            subAtomCoord[i][j].z = get_float(tempString2, 36, 8);
            printf("x %f y %f z %f  \n", subAtomCoord[i][j].x, subAtomCoord[i][j].y, subAtomCoord[i][j].z);
        }
        fclose(fileId);
    }
//---------------------READ TOPOLOGY



//------------Print initial data
    //print flow properties
    printf("#\tT,K\tn,mol/l\tinsert molecules\n");
    for (int i = 0; i < flowNum; i++) {
        printf("%d\t%f\t%f\t%d\n", i, flowT[i], flowN[i], flowIns[i]);
    }
    //molecule properties print
    printf("#\t molecule name\n");
    for (int i = 0; i < subNum; i++) {
        printf("%d\t%s\n", i, subFile[i]);
    }

    //------------------INITIAL SIMULATION
    initMolNum = (int *) malloc(flowNum * sizeof(int));
    for (int i = 0; i < flowNum; i++) {
        initMolNum[i] = flowIns[i] * 10;

    }




    //------------------FREE Arrays
    for (int i = 0; i < flowNum; i++) {
        free(flowX[i]);
    }
    free(flowX);
    free(flowT);
    free(flowN);
    free(pd);    //free properties of devices


//	priflowNlowT("Device count %d \n", deviceCount);
//	cudaGetDeviceProperties(&pd,0);
//	priflowNlowT("Device name %s \n", pd.name);
//	priflowNlowT("Registers per block: %d \n",pd.regsPerBlock);
//	priflowNlowT("Max Threads Dim: %d %d %d \n", pd.maxThreadsDim[0],pd.maxThreadsDim[1],pd.maxThreadsDim[2]);
//
    return 0;
}
