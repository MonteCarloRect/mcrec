
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SUBNUMMAX 10
#define BUFFER 255
#define INSTIMES 10
#define MAXMOL 5000


int find_maximum(int a[], int n) {
int c, max, index;
	max = a[0];
	index = 0;
	for (c = 1; c < n; c++) {
		if (a[c] > max) {
			index = c;
		max = a[c];
	}
}
return a[index];
}

int get_integer(char a[], int begin, int length){
char b[length];
	for(int i=0;i<length;i++){
		b[i]=a[begin+i];
	}
	return atoi(b);
}

void get_string(char a[], char out[], int begin, int length){
	for(int i=0;i<length;i++){
		out[i]=a[begin+i];
	}
}

float get_float(char a[], int begin, int length){
char b[length];
	for(int i=0;i<length;i++){
		b[i]=a[begin+i];
	}
	return atof(b);
}

int main (int argc, char * argv[]){
	//----------------VAR
	int deviceCount;
	hipDeviceProp_t temppd;	//temp varaible
	hipDeviceProp_t* pd;	//array of device properties
	hipError_t currentError;	//current error
	FILE* fileId;	//input file ID
	FILE* file2Id;	//input file ID

	//substance 
	int subNum;	//substance number
	char subFile[SUBNUMMAX][BUFFER];	//substance filenames
	int subAtomMax;	//maximum atom number in substances
	int* subAtomNum;	//atom numbers in molecules
	char** subName;	//substane residual name
	char*** subAtomName;	//attom names
	float3** subAtomCoord;	//atom position in molecule
	float3** subAtomVel;	//atom velocity
	

	//input flow
	int flowNum;	//flow numbers
	float** flowX;	//flow mole fractions
	int** flowNiIns;	//flow number of inserted molecules per cycle
	float* flowT;	//flow temperatures
	float* flowN;	//flow number density
	int* flowIns;	//inserting molecules per
	
	//temp varaibles
	char tempString[BUFFER],tempString2[BUFFER];
	int tempInt,tempInt2;
	float tempFloat;
	char* tempString3,tempString4;


	
	//init varaibles
	int* initMolNum;
	float3** initMolCoord_c;
	float3*** inttAtomCoord_c;
	int latice;
	int** initMolType_c;
	int** initBoxType_c;
	float3* tempCell;	//initial coordinates of 
	int* tempBusy;	//
	float Lbox;	//liquid cell length
	float Vbox;	//vapor cell length
	int id,id2;	//id of initial cell
	float liqFrac;
	int** initMolInsV;
	int** initMolInsL;
	int** initMolLiqList;	//list of molecules in liquid phase at plate
	int** initMolVapList;	//list of molecules in vapor phase at plate
	int* initMolLiqNum;	//numbers of molecules in liquid phase at plate
	int* initMolVapNum;	//number of molecules in vapor phase at plate


	//functions
	int find_maximum(int a[], int n);
	int get_integer(char a[], int n, int begin, int length);
	void get_string(char a[], char out[], int begin, int length);
	float get_float(char a[], int begin, int length);
	
	//random generator
	srand(time(NULL));
	//----------------GET DEVICE PROPERTIES
	currentError=hipGetDeviceCount(&deviceCount);
	if (currentError!=hipSuccess){
		fprintf(stderr,"Cannot get CUDA device count: %s\n", hipGetErrorString(currentError));
		return 1;
	}
	if (!deviceCount){
		fprintf(stderr, "No CUDA devices found\n");
		return 1;
	}
	pd=(hipDeviceProp_t*) malloc(deviceCount*sizeof(hipDeviceProp_t));
	for (int i=0;i<deviceCount;i++){
		hipGetDeviceProperties(&temppd,i);
//		priflowNlowT("Device name %s \n", pd.name);
		pd[i]=temppd;
		printf("Device name %s \n", pd[i].name);
		printf("Max Threads Dim: %d %d %d \n", pd[i].maxThreadsDim[0],pd[i].maxThreadsDim[1],pd[i].maxThreadsDim[2]);
		printf("Max Grid Size: %d %d %d \n", pd[i].maxGridSize[0], pd[i].maxGridSize[1], pd[i].maxGridSize[2]);
	}
	
	//------------------READ INPUT DATA
	fileId=fopen("data.mcr","r");
		fscanf(fileId,"%d",&subNum);
//read molecules data
		printf("Substance number: %d\n", subNum);
		for(int i=0;i<subNum;i++){
			fscanf(fileId,"%s",subFile[i]);
		}
//read flow data
		fscanf(fileId,"%d",&flowNum);
		flowT=(float*) malloc(flowNum*sizeof(float));
		flowN=(float*) malloc(flowNum*sizeof(float));
		flowIns=(int*) malloc(flowNum*sizeof(int));
		for(int i=0;i<flowNum;i++){
			fscanf(fileId,"%f",&flowT[i]);
		}
		for(int i=0;i<flowNum;i++){
			fscanf(fileId,"%f",&flowN[i]);
		}
		for(int i=0;i<flowNum;i++){
			fscanf(fileId,"%d",&flowIns[i]);
		}
		flowX=(float**) malloc(flowNum*sizeof(float*));
		for(int i=0;i<flowNum;i++){
			flowX[i]=(float*)malloc(subNum*sizeof(float));
		}
		flowNiIns=(int**) malloc(flowNum*sizeof(int*));
		for(int i=0; i<flowNum;i++){
			flowNiIns[i]=(int*) malloc(subNum*sizeof(int));
		}
		for(int i=0;i<flowNum;i++){
			for(int j=0;j<subNum;j++){
			fscanf(fileId,"%f", &flowX[i][j]);
			}
		}
		//calculate numbers of initial molecules
		for(int i=0;i<flowNum;i++){
			tempInt=0;
			for(int j=0;j<subNum-1;j++){
				flowNiIns[i][j]=floor(flowX[i][j]*flowIns[i]);
				tempInt+=flowNiIns[i][j];
			}
			flowNiIns[i][subNum-1]=flowIns[i]-tempInt;
		}
	fclose(fileId);
//--------------------READ GRO FILES
//read molecules structure 
	subAtomNum=(int*)malloc(subNum*sizeof(int));
	for(int i=0;i<subNum;i++){
		fileId=fopen(subFile[i],"r");
		 if (fileId == NULL){
			printf("Error opening file %s\n", subFile[i]);
			return 1;
		}
		fgets(tempString,BUFFER,fileId);
		fscanf(fileId,"%d",&subAtomNum[i]);
		fclose(fileId);
//			fscanf(file2Id,"%5d%-5s%5s%5d%8.3f%8.3f%8.3f%8.4f%8.4f%8.4f", )
	}
	//allocate gro varaibles
	subAtomMax=find_maximum(subAtomNum,subNum);
	printf("Maximum atoms numbers %d\n",subAtomMax);
	subName=(char**)malloc(subNum*sizeof(char*));	//allocate molecule names
	for(int i=0;i<subNum;i++){
		subName[i]=(char*)malloc(5*sizeof(char));
	}
	subAtomCoord=(float3**)malloc(subNum*sizeof(float3*));	//allocate coordinates
	for(int i=0;i<subNum;i++){
		subAtomCoord[i]=(float3*)malloc(subAtomMax*sizeof(float3));
	}
	subAtomVel=(float3**)malloc(subNum*sizeof(float3*));	//allocate velocities
	for(int i=0;i<subNum;i++){
		subAtomVel[i]=(float3*)malloc(subAtomMax*sizeof(float3));
	}
	subAtomName=(char***)malloc(subNum*sizeof(char**));	//allocate atoma names
	for(int i=0;i<subNum;i++){
		subAtomName[i]=(char**)malloc(subAtomMax*sizeof(char*));
		for(int j=0;j<subAtomMax;j++){
			subAtomName[i][j]=(char*)malloc(5*sizeof(char));
		}
	}
	for(int i=0;i<subNum;i++){
		fileId=fopen(subFile[i],"r");
		fgets(tempString,BUFFER,fileId);	//skip 2 lines
		fgets(tempString,BUFFER,fileId);
		for(int j=0;j<subAtomNum[i];j++){
			fgets(tempString2,BUFFER,fileId);	//read string to buffer
			get_string(tempString2,subName[i],5,5);	//parse string
			printf("sub %s\n",subName[i]);
			get_string(tempString2,subAtomName[i][j],10,5);
			printf("atom %s\n",subAtomName[i][j]);
			subAtomCoord[i][j].x=get_float(tempString2,20,8);
			subAtomCoord[i][j].y=get_float(tempString2,28,8);
			subAtomCoord[i][j].z=get_float(tempString2,36,8);
			printf("x %f y %f z %f  \n",subAtomCoord[i][j].x,subAtomCoord[i][j].y,subAtomCoord[i][j].z);
		}
		fclose(fileId);
	}
//---------------------READ TOPOLOGY



//------------Print initial data
	//print flow properties
	printf("#\tT,K\tn,mol/l\tinsert molecules\n");
	for(int i=0;i<flowNum;i++){
		printf("%d\t%f\t%f\t%d\n",i,flowT[i],flowN[i],flowIns[i]);
	}
	//molecule properties print
	printf("#\t molecule name\n");
	for(int i=0; i<subNum;i++){
		printf("%d\t%s\n",i,subFile[i]);
	}

	//------------------INITIAL SIMULATION
	initMolNum=(int*)malloc(flowNum*sizeof(int));
	initMolCoord_c=(float3**)malloc(flowNum*sizeof(float3*));
	initMolType_c=(int**)malloc(flowNum*sizeof(int*));
	initMolInsV=(int**)malloc(flowNum*sizeof(int*));
	initMolInsL=(int**)malloc(flowNum*sizeof(int*));
	initMolLiqList=(int**)malloc(flowNum*sizeof(int*));
	initMolVapList=(int**)malloc(flowNum*sizeof(int*));
	initMolLiqNum=(int*)malloc(flowNum*sizeof(int));
	initMolVapNum=(int*)malloc(flowNum*sizeof(int));
	//allocate initial arrays
	for(int i=0;i<flowNum;i++){
		initMolNum[i]=flowIns[i]*INSTIMES;	//numbers of molecules in initial plates 10 times ladger input flow
		initMolCoord_c[i]=(float3*)malloc(initMolNum[i]*sizeof(float3));	//allocate atom coords
		initMolType_c[i]=(int*)malloc(initMolNum[i]*sizeof(int));
		initMolInsV[i]=(int*)malloc(subNum*sizeof(int));
		initMolInsL[i]=(int*)malloc(subNum*sizeof(int));
		initMolLiqList[i]=(int*)malloc(initMolNum[i]*sizeof(int));
		initMolVapList[i]=(int*)malloc(initMolNum[i]*sizeof(int));
		for(int j=0;j<initMolNum[i];j++){
			initMolType_c[i][j]=-1;	//initial free space
		}
	}
	for (int i=0;i<flowNum;i++){
		//Insert in liquid box
		liqFrac=0.8;
		for(int j=0;j<subNum;j++){	//get liquid molecules per cell
			initMolInsL[i][j]=ceil(flowNiIns[i][j]*INSTIMES*liqFrac);
			initMolInsV[i][j]=flowNiIns[i][j]*INSTIMES-initMolInsL[i][j];
		}
		latice=ceil(pow(initMolNum[i]*liqFrac,1.0/3.0))+1;	//
		//insert in liquid
		printf("numbers %d\n",latice);
		printf("nmol %d \n",initMolNum[i]);
		tempCell=(float3*)malloc(latice*latice*latice*sizeof(float3));	//temporally cell array
		tempBusy=(int*)malloc(latice*latice*latice*sizeof(int));
		Lbox=10.0;	//написать выбор размеров начальных боксов
		id=0;
		for(int j=0;j<latice;j++){
			for(int k=0;k<latice;k++){
				for(int l=0;l<latice;l++){
					tempCell[id].x=(float)j*Lbox/(float)latice;
					tempCell[id].y=(float)k*Lbox/(float)latice;
					tempCell[id].z=(float)l*Lbox/(float)latice;
					tempBusy[id]=-1;
					id++;
				}
			}
		}
		id=0;
		for(int j=0;j<subNum;j++){
			tempInt=0;
			while(tempInt<initMolInsL[i][j]){
				tempInt2=rand() % (latice*latice*latice);
//				printf("random %d busy %d x %f \n",tempInt2,tempBusy[tempInt2],tempCell[tempInt2].x);
//				tempInt++;
				if(tempBusy[tempInt2]==-1){
//					initMolType_c[i]
					initMolType_c[i][id]=j;
					initMolCoord_c[i][id]=tempCell[tempInt2];
					initMolLiqList[i][id]=id;
					//liquid
					
					tempBusy[tempInt2]=1;
					tempInt++;
//					printf("%d sub %d id %d\n",id,j,initMolNum[i]);
//					printf(" x %f xl %f\n",initMolCoord_c[j][id].x,tempCell[tempInt2].x);
					id++;
				}
			}
		}
		free(tempCell);
		free(tempBusy);
		id2=id; //get the last element index
		//insert molecules in vapor=========================================
		latice=ceil(pow(initMolNum[i]*(1.0-liqFrac),1.0/3.0))+1;	//
		//insert in liquid
//		printf("numbers %d\n",latice);
//		printf("nmol %d \n",initMolNum[i]);
		tempCell=(float3*)malloc(latice*latice*latice*sizeof(float3));	//temporally cell array
		tempBusy=(int*)malloc(latice*latice*latice*sizeof(int));
		Lbox=40.0;	//написать выбор размеров начальных боксов
		id=0;
		for(int j=0;j<latice;j++){
			for(int k=0;k<latice;k++){
				for(int l=0;l<latice;l++){
					tempCell[id].x=(float)j*Lbox/(float)latice;
					tempCell[id].y=(float)k*Lbox/(float)latice;
					tempCell[id].z=(float)l*Lbox/(float)latice;
					tempBusy[id]=-1;
					id++;
				}
			}
		}
		initMolLiqNum[i]=id;	//numbers of molecules in liquid phase
		id=id2;
		id2=0;
		for(int j=0;j<subNum;j++){
			tempInt=0;
			while(tempInt<initMolInsV[i][j]){
				tempInt2=rand() % (latice*latice*latice);
//				printf("random %d busy %d x %f \n",tempInt2,tempBusy[tempInt2],tempCell[tempInt2].x);
//				tempInt++;
				if(tempBusy[tempInt2]==-1){
//					initMolType_c[i]
					initMolType_c[i][id]=j;	//set type of molecule
					initMolCoord_c[i][id]=tempCell[tempInt2];	//set coordinates from cubic latice
					initMolVapList[i][id2]=id;
					//vapor molecules list
						
					tempBusy[tempInt2]=1;
					tempInt++;
//					printf("%d sub %d id %d\n",id,j,initMolNum[i]);
//					printf(" x %f xl %f\n",initMolCoord_c[j][id].x,tempCell[tempInt2].x);
					id++;
					id2++;
				}
			}
		}
		initMolVapNum[i]=id2;
		free(tempCell);
		free(tempBusy);
	}
	//for initial calculates on first device
	
	
	
	

	//------------------FREE Arrays
//free
	
	free(flowT);
	free(flowN);
	free(pd);

	for(int i=0;i<flowNum;i++){
		free(flowX[i]);
		
		//initial arrays
		free(initMolCoord_c[i]);
	}
	free(flowX);

	free(initMolCoord_c);
	//free properties of devices


//	priflowNlowT("Device count %d \n", deviceCount);
//	cudaGetDeviceProperties(&pd,0);
//	priflowNlowT("Device name %s \n", pd.name);
//	priflowNlowT("Registers per block: %d \n",pd.regsPerBlock);
//	priflowNlowT("Max Threads Dim: %d %d %d \n", pd.maxThreadsDim[0],pd.maxThreadsDim[1],pd.maxThreadsDim[2]);
//	
	return 0;
}


