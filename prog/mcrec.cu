#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "global.h"
#include "mcrec.h"
#include "initial.h"


int main (int argc, char * argv[]){

//openlog file
    logFile=fopen("calculation.log","w");
    //
    get_device_prop(deviceCount, deviceProp);
    write_prop_log(deviceCount,deviceProp,logFile);
    //read initial data
    read_options(config);
    write_config_log(config,logFile);
    //read gro data for each molecules
    initMol = (molecules *) malloc(config.subNum * sizeof(molecules));
    read_init_gro(config, initMol);
    //read topology of molecules
    read_top(allParams,paramsLines);
    for(int i=0;i<paramsLines;i++){
        printf("  %s \n",allParams[i].aName);
    }
    printf("number of top lines %d\n",paramsLines);
    //create initial simulation
    initial_flows(config, initFlows,initMol,gpuSingleBox);
    printf("\n test %d \n", initFlows[0].molNum);
    //put data to device
//    data_to_device(initFlows,gpuSingleBox,config);
    
    printf("\n test2 %d \n", initFlows[0].molNum);
//close log file
    freeAll(gpuSingleBox,initFlows,config);
    fclose(logFile);


}
